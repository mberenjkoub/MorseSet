#include "hip/hip_runtime.h"


#include <cstdio>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_timer.h>


#include<stack>
#include<set>
#include<map>
#include<queue>
#include<math.h>

#include "streamline.h"
//#include "streamline_kernel.h"
#include "graph_generator.h"
#include "scan_common.h"
//#include "parallel_fwd.h"
//#include "hash_table.h"
# define M_PI           3.14159265358979323846  /* pi */
#define OUTBOUNDARY 0xFFFF
void
runTest(uint32_t* d_In, uint32_t* d_Out, uint32_t* h_out, int num_elements);
//void
//runTest(uint32_t* d_in, uint32_t* d_out, uint32_t* h_out, int num_elements);
__host__ __device__ uint32_t getDimRange(Dimension d, Dimension i){ return (i.x + i.y*d.x + i.z*d.x*d.y); }


__host__ __device__ double gauss_fun(double x, double sigma)
{

	return (exp(-pow(x, 2) / (2 * pow(sigma, 2)))) / (sigma*sqrt(2 * M_PI));

}


int iDivUp1(int a, int b) // Round a / b to nearest higher integer value

{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

__device__ __host__ void  Sort_Endpoints_C(ASF_vertex * m, Boundary b, Dimension d, Point step, uint32_t* output, uint32_t num_rows)
{


	ASF_vertex vertex = m[0];

	int index = 0;
	int index2 = 0;
	int x = 0;
	int y = 0;
	int z = 0;

	float xx = 0.;
	float yy = 0.;
	float zz = 0.;

	uint32_t* out1 = new uint32_t[num_rows];
	uint32_t* out2 = new uint32_t[num_rows];
	memset(out1, 0, num_rows*sizeof(uint32_t));
	memset(out2, 0, num_rows*sizeof(uint32_t));
	for (uint32_t row = 0; row < num_rows; row++)
	{
		vertex = m[row];

		if (!vertex.isInBoundary())
		{
			continue;

		}

		xx = (vertex.e.x - b.low.x) / step.x;
		yy = (vertex.e.y - b.low.y) / step.y;
		zz = (vertex.e.z - b.low.z) / step.z;

		x = (int)xx;
		y = (int)yy;
		z = (int)zz;

		index = z*(d.x*d.y) + y*d.x + x;
		index2 = zz*(d.x*d.y) + yy*d.x + xx;
		/*if (index2 != vertex.getRange())

		output[vertex.getRange()]++;;*/
		out1[index]++;;
		out2[index2]++;;
		output[vertex.getRange()]++;;



	}


}



__global__ void  Sort_Endpoints(ASF_vertex * m, Point* output, uint32_t num_rows)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= num_rows)
		return;


}


__global__ void  Assign_Ball(ASF_vertex * m, float radius, uint32_t num_rows)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= num_rows)
		return;
	dim3 grid(((num_rows + 510) / 512), 1, 1);  // (RSize-2) valid vertecis the first one is undefined 
	dim3 threads(512, 1, 1);
	float d_output[32678];

	ASF_vertex vertex = m[row];




}


__host__ __device__ void get_Lorenz_Field(float x, float y, float z, float &vxp, float &vyp, float 	&vzp)
{
	// 	float alpha = -1.28805;
	// 	float gamma = -0.502655;
	// 	float beta = 0.0314159;
	// 	vxp = gamma* cos(y) + alpha * sin(z);
	// 	vyp = alpha * cos(x) + beta* sin(x);
	// 	vzp = beta*cos(x) + gamma* sin(y);

	float sigma = 10.0;
	float ro = 28.0;
	float beta = 8.0 / 3.0;
	vxp = sigma * (y - x);
	vyp = (x*(ro - z)) - y;
	vzp = x*y - beta*z;
}



__host__ __device__ void trilinearInterpolation(ASF_vertex*m, Point p1, Point* v, uint32_t i, Dimension* _d, Boundary* _b, Point* step, float&vx, float& vy, float& vz, bool bForward)
{

	uint32_t xDim = _d->x;
	uint32_t yDim = _d->y;
	uint32_t zDim = _d->z;



	//if (!bForward)
	//	p1 = m[i].eb;

	Point highBoundary = _b->high;
	Point lowBoundary = _b->low;



	//	Point temp = (p1 - lowBoundary);
	//	Dimension index = temp.divide(*step);

	int indexx = (p1.x - lowBoundary.x) / step->x;
	int indexy = (p1.y - lowBoundary.y) / step->y;
	int indexz = (p1.z - lowBoundary.z) / step->z;


	uint32_t vi = indexx + indexy* _d->x + indexz*_d->x*_d->y; //getDimRange(*_d, index);
	//	float vx, vy, vz;
	get_Lorenz_Field(p1.x, p1.y, p1.z, vx, vy, vz);

	Point p0 = m[vi].p;


	double xd = (p1.x - p0.x) / step->x;//(p1[0]-step_x*int(p1[0]/step_x))/step_x;
	double yd = (p1.y - p0.y) / step->y;//(p1[1]-step_y*int(p1[1]/step_y))/step_y;
	double zd = (p1.z - p0.z) / step->z;//(p1[2]-step_z*int(p1[2]/step_z))/step_z;

	int kk = indexz;// (p1.z - lowBoundary.z) / step->z;;
	int jj = indexy;// (p1.y - lowBoundary.y) / step->y;;
	int ii = indexx;// (p1.x - lowBoundary.x) / step->x;;

	if (kk >= zDim - 1 || jj >= yDim - 1 || ii >= xDim - 1)
	{
		vx = v[((kk  * yDim + jj) * xDim + ii)].x;
		vy = v[((kk  * yDim + jj) * xDim + ii)].y;
		vz = v[((kk  * yDim + jj) * xDim + ii)].z;
		//return  v[((kk  * yDim + jj) * xDim + ii)];
	}

	if (p0 == p1)
	{
		vx = v[vi].x;
		vy = v[vi].y;
		vz = v[vi].z;

	}


	float v1[3];
	float v2[3];
	float v3[3];
	float v4[3];
	float v5[3];
	float v6[3];
	float v7[3];
	float v8[3];

	//vx = m_x[((kk * yDim + jj) * xDim + ii)];
	//vy = m_y[((kk * yDim + jj) * xDim + ii)];
	//vz = m_z[((kk * yDim + jj) * xDim + ii)];




	v1[0] = v[((kk * yDim + jj) * xDim + ii)].x;
	v1[1] = v[((kk * yDim + jj) * xDim + ii)].y;
	v1[2] = v[((kk * yDim + jj) * xDim + ii)].z;


	v2[0] = v[((kk * yDim + jj) * xDim + ii + 1)].x;
	v2[1] = v[((kk * yDim + jj) * xDim + ii + 1)].y;
	v2[2] = v[((kk * yDim + jj) * xDim + ii + 1)].z;

	v3[0] = v[((kk * yDim + (jj + 1)) * xDim + ii)].x;
	v3[1] = v[((kk * yDim + (jj + 1)) * xDim + ii)].y;
	v3[2] = v[((kk * yDim + (jj + 1)) * xDim + ii)].z;
	//

	v4[0] = v[((kk * yDim + (jj + 1)) * xDim + ii + 1)].x;
	v4[1] = v[((kk * yDim + (jj + 1)) * xDim + ii + 1)].y;
	v4[2] = v[((kk * yDim + (jj + 1)) * xDim + ii + 1)].z;

	//int idx2 = (k + 1)*(xDim*yDim) + (j + 1)*yDim + i + 1;

	v5[0] = v[(((kk + 1) * yDim + jj) * xDim + ii)].x;
	v5[1] = v[(((kk + 1) * yDim + jj) * xDim + ii)].y;
	v5[2] = v[(((kk + 1) * yDim + jj) * xDim + ii)].z;


	v6[0] = v[(((kk + 1) * yDim + jj) * xDim + ii + 1)].x;
	v6[1] = v[(((kk + 1) * yDim + jj) * xDim + ii + 1)].y;
	v6[2] = v[(((kk + 1) * yDim + jj) * xDim + ii + 1)].z;


	v7[0] = v[(((kk + 1) * yDim + (jj + 1)) * xDim + ii)].x;
	v7[1] = v[(((kk + 1) * yDim + (jj + 1)) * xDim + ii)].y;
	v7[2] = v[(((kk + 1) * yDim + (jj + 1)) * xDim + ii)].z;


	v8[0] = v[(((kk + 1) * yDim + (jj + 1)) * xDim + ii + 1)].x;
	v8[1] = v[(((kk + 1) * yDim + (jj + 1)) * xDim + ii + 1)].y;
	v8[2] = v[(((kk + 1) * yDim + (jj + 1)) * xDim + ii + 1)].z;


	double c00 = v1[0] * (1 - xd) + v2[0] * xd;
	double c10 = v3[0] * (1 - xd) + v4[0] * xd;
	double c01 = v5[0] * (1 - xd) + v6[0] * xd;
	double c11 = v7[0] * (1 - xd) + v8[0] * xd;

	double c0 = c00*(1 - yd) + c10*yd;
	double c1 = c01*(1 - yd) + c11*yd;

	//Point tempv = p1;
	vx = c0*(1 - zd) + c1*zd;


	c00 = v1[1] * (1 - xd) + v2[1] * xd;
	c10 = v3[1] * (1 - xd) + v4[1] * xd;
	c01 = v5[1] * (1 - xd) + v6[1] * xd;
	c11 = v7[1] * (1 - xd) + v8[1] * xd;

	c0 = c00*(1 - yd) + c10*yd;
	c1 = c01*(1 - yd) + c11*yd;

	vy = c0*(1 - zd) + c1*zd;

	c00 = v1[2] * (1 - xd) + v2[2] * xd;
	c10 = v3[2] * (1 - xd) + v4[2] * xd;
	c01 = v5[2] * (1 - xd) + v6[2] * xd;
	c11 = v7[2] * (1 - xd) + v8[2] * xd;

	c0 = c00*(1 - yd) + c10*yd;
	c1 = c01*(1 - yd) + c11*yd;

	vz = c0*(1 - zd) + c1*zd;
	float vxx, vyy, vzz;
	//	get_Lorenz_Field(p1.x, p1.y, p1.z, vxx, vyy, vzz);
	if (abs(vx - vxx) > exp(-6.0) || abs(vy - vyy) > exp(-6.0) || abs(vz - vzz) > exp(-6.0))
		printf("");
	//tempv.x = vx;
	//tempv.y = vy;
	//tempv.z = vz;
	//	return tempv;

	return;



}



void _Tracing_c(ASF_vertex * m, Point*v, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t whichData, uint32_t originalnum_rows, uint32_t num_rows, uint32_t level)
{
	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	for (uint32_t row = 0; row < num_rows; row++)
	{


		if (level % 2 == 0)
		{
			if (row >= num_rows)
				continue;
		}
		else
		{
			if (row >= num_rows || row < originalnum_rows)
				continue;
		}
		/*if (row == 32818)
		printf("");*/
		ASF_vertex vertex = m[row];

		if (!vertex.isInBoundary())
		{
			continue;
		}

		Point e = vertex.e;
		float vx, vy, vz;
		trilinearInterpolation(m, e, v, vertex.getOldRange(), d, b, step, vx, vy, vz, bForward);

		/*if (row == 4591)
		printf("v = %f,%f,%f \n", vx, vy, vz);*/
		float dist = sqrt(vx*vx + vy*vy + vz*vz);

		vx = (vx / (dist*4.0))*step->x;
		vy = (vy / (dist*4.0))*step->y;
		vz = (vz / (dist*4.0))*step->z;
		/*_v = _v / 4.0;
		_v *= *step;*/
		//glColor3f(rgb[0], rgb[1], rgb[2]);    
		//		Point e = vertex.e;
		if (bForward)
		{
			vertex.e.x += vx;
			vertex.e.y += vy;
			vertex.e.z += vz;
		}
		//vertex.e += _v;
		else
		{
			vertex.e.x -= vx;
			vertex.e.y -= vy;
			vertex.e.z -= vz;
		}
		//vertex.eb -= _v;

		float ep[3];
		//generalstreamlineTracing_single(p1, bForward, ep, false);
	//	if (bForward)
		{
			if (ep[0] != vertex.e.x && ep[1] != vertex.e.y && ep[2] != vertex.e.z)
				printf("");
		}
		/*else
		{
			if (ep[0] != vertex.eb.x && ep[1] != vertex.eb.y && ep[2] != vertex.eb.z)
				printf("");

		}*/




		bool xy = false;
		bool yz = false;
		bool xz = false;
		//if (bForward)
		{
			if (vertex.checkInBoundary(b))
			{
				uint32_t range = vertex.getRange(b, step, d);
				if (vertex.isInNextLevel_yz())
					yz = true;
				if (vertex.isInNextLevel_xy())xy = true;
				if (vertex.isInNextLevel_xz())xz = true;
				vertex.setRange(range);
				if (xy)	vertex.setInNextLevel_xy();
				if (xz)vertex.setInNextLevel_xz();
				if (yz)vertex.setInNextLevel_yz();

			}
			else
			{
				vertex.unsetInBoundary();
			}
		}
		/*else
		{
			if (vertex.checkInBoundaryBackward(b))
			{
				if (vertex.isInNextLevel_yz())
					yz = true;
				if (vertex.isInNextLevel_xy())xy = true;
				if (vertex.isInNextLevel_xz())xz = true;
				uint32_t range = vertex.getRangeBackward(b, step, d);
				vertex.setRangeBackward(range);
				if (xy)	vertex.setInNextLevel_xy();
				if (xz)vertex.setInNextLevel_xz();
				if (yz)vertex.setInNextLevel_yz();

			}
			else
			{
				vertex.unsetInBoundary();
			}
		}*/




		m[row] = vertex;
	}
	return;
}


__host__ __device__ ASF_vertex*  DivideEdges(ASF_vertex* v1, ASF_vertex* v2, uint32_t range, Dimension*d, Point* step, Boundary*b, uint32_t faceNum, bool bForward)
{

	Point p_left_c = v1->p;
	ASF_vertex _tv = *v1;
	//return &_tv;
	p_left_c.x = (v1->p.x + v2->p.x) / 2;
	p_left_c.y = (v1->p.y + v2->p.y) / 2;
	p_left_c.z = (v1->p.z + v2->p.z) / 2;
	_tv.p = p_left_c;


	p_left_c.x = (v1->e.x + v2->e.x) / 2;
	p_left_c.y = (v1->e.y + v2->e.y) / 2;
	p_left_c.z = (v1->e.z + v2->e.z) / 2;

	//_tv.p = p_left_c;
	/*if (!bForward)
	{
		p_left_c.x = (v1->eb.x + v2->eb.x) / 2;
		p_left_c.y = (v1->eb.y + v2->eb.y) / 2;
		p_left_c.z = (v1->eb.z + v2->eb.z) / 2;
		_tv.eb = p_left_c;
		uint32_t range = _tv.getRangeBackward(b, step, d);

		_tv.setRangeBackward(range);
	}
	else*/
	{
		_tv.e = p_left_c;

		uint32_t range = _tv.getRange(b, step, d);

		_tv.setRange(range);
	}
	uint32_t oldrange = v1->getOldRange();   // _tv.getOldRange(&_b, &step, &_d);
	_tv.setOldRange(oldrange);
	_tv.setInBoundary();


	return &_tv;

}


__global__ void CheckRangeSetKernel(const ASF_vertex * m, uint32_t * Fr, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	ASF_vertex vertex = m[row];

	if (vertex.isInBoundary())
		Fr[vertex.getOldRange()] = 1;
	else
		Fr[vertex.getOldRange()] = 0;
	return;

}

__global__ void CheckConnectivityKernel(const ASF_vertex * m, uint32_t * Fr, Dimension*d, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	int iz = row / (d->x*d->y);
	int iy = (row - iz*(d->x*d->y)) / d->x;
	int ix = (row - iz*(d->x*d->y)) % d->x;

	Fr[row] = 0;

	if (ix == d->x - 1 || iy == d->y - 1 || iz == d->z - 1)
		return;
	ASF_vertex vertex = m[row];
	if (vertex.isInNextLevel_xy())
		Fr[row] += 1;
	if (vertex.isInNextLevel_xz())
		Fr[row] += 1;
	if (vertex.isInNextLevel_yz())
		Fr[row] += 1;

}



__host__ __device__ void CheckConnectivityKernel_c(const ASF_vertex * m, uint32_t * Fr, Dimension*d, const uint32_t num_rows)
{
	//	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	for (uint32_t row = 0; row < num_rows; row++)
	{
		if (row >= num_rows)
			return;
		int iz = row / (d->x*d->y);
		int iy = (row - iz*(d->x*d->y)) / d->x;
		int ix = (row - iz*(d->x*d->y)) % d->x;

		Fr[row] = 0;

		if (ix == d->x - 1 || iy == d->y - 1 || iz == d->z - 1)
			continue;
		ASF_vertex vertex = m[row];
		if (vertex.isInNextLevel_xy())
			Fr[row] += 1;
		if (vertex.isInNextLevel_xz())
			Fr[row] += 1;
		if (vertex.isInNextLevel_yz())
			Fr[row] += 1;

	}

	return;

}

//__global__ void _CheckFacing(ASF_vertex*m, ASF_vertex*d_a, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t num_rows)
//{
//	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (row >= num_rows)
//		return;
//
//	ASF_vertex vertex1 = m[row];
//
//	if (!vertex1.isInSCC())
//	{
//		return;
//
//	}
//	ASF_vertex vertex2 = m[vertex1.Fr[0]];
//	ASF_vertex vertex3 = m[vertex1.Fr[1]];
//	ASF_vertex vertex4 = m[vertex1.Fr[2]];
//	ASF_vertex pa = d_a[row];
//	//Point* p = DivideFaces(&vertex1, &vertex2, &vertex3, &vertex4);
//	/* a[row * 5].p = p[0];
//	a[row * 5].p = p[0];
//	a[row * 5].p = p[0];
//	a[row * 5].p = p[0];*/
//	d_a[row * 2] = vertex1;
////	pa.p = p[0];
//	pa.setInBoundary();
//	pa.setOldRange(vertex1.getOldRange());
//	d_a[row * 2 + 1] = pa;
//
//
//
//
//	return;
//}


//void _CheckFacing_c(ASF_vertex*m, ASF_vertex*d_a, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t num_rows)
//{
//	// uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	for (uint32_t row = 0; row < num_rows; row++)
//	{
//
//		if (row >= num_rows)
//			return;
//
//		ASF_vertex vertex1 = m[row];
//		bool b = m[14831].isInSCC();
//
//		if (!vertex1.isInSCC())
//		{
//			continue;
//
//		}
//		ASF_vertex vertex2 = m[vertex1.Fr[0]];
//		ASF_vertex vertex3 = m[vertex1.Fr[1]];
//		ASF_vertex vertex4 = m[vertex1.Fr[2]];
//		ASF_vertex pa = d_a[row];
//		//Point* p = DivideFaces(&vertex1, &vertex2, &vertex3, &vertex4);
//		/* a[row * 5].p = p[0];
//		a[row * 5].p = p[0];
//		a[row * 5].p = p[0];
//		a[row * 5].p = p[0];*/
//		d_a[row * 2] = vertex1;
////		pa.p = p[0];
//		pa.setInBoundary();
//		pa.setOldRange(vertex1.getOldRange());
//		d_a[row * 2 + 1] = pa;
//
//
//	}
//
//	return;
//}


__global__ void Initialize(ASF_vertex * m, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= num_rows)
		return;

	ASF_vertex vertex = m[row];
	vertex.setInBoundary();

	/*if (row < 10)
	printf("%d \n", row);*/

	//vertex.Fr_xy.x = row + 1;
	//vertex.Fr_xy.y = row + 1 + d->x;
	//vertex.Fr_xy.z = row + d->x;

	//vertex.Fr_xz.x = row + 1;
	//vertex.Fr_xz.y = row + d->y * d->x + 1;
	//vertex.Fr_xz.z = row + d->x*d->y;

	//vertex.Fr_yz.x = row + d->x;
	//vertex.Fr_yz.y = row + (1 + d->y) * d->x;
	//vertex.Fr_yz.z = row + d->x*d->y;


	//vertex.setInNextLevel_xy();
	//vertex.setInNextLevel_xz();
	//vertex.setInNextLevel_yz();
	m[row] = vertex;


}

__global__ void Initialize2(ASF_vertex * m, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= num_rows)
		return;

	ASF_vertex vertex = m[row];
	if (vertex.isInBoundary())
	{
		vertex.setInNextLevel_xy();
		vertex.setInNextLevel_xz();
		vertex.setInNextLevel_yz();
	}

	m[row] = vertex;


}



__global__ void _TracingMGPU(ASF_vertex * m, Point*v, Boundary* b, Dimension* d, Point* step, bool bForward, uint32_t num_rows)//Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t whichdata, uint32_t originalnum_rows, uint32_t num_rows, uint32_t level)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;



	if (row >= num_rows)
		printf("");
	ASF_vertex vertex = m[row];

	if (!vertex.isInBoundary())
	{
		return;
	}

	float p1[3];
	//if (bForward)
	{
		p1[0] = vertex.e.x;
		p1[1] = vertex.e.y;
		p1[2] = vertex.e.z;
	}
	/*else
	{
		p1[0] = vertex.eb.x;
		p1[1] = vertex.eb.y;
		p1[2] = vertex.eb.z;
	}*/
	if (row == 191)
		printf("");
	Point e = vertex.e;
	float vx, vy, vz;
	trilinearInterpolation(m, e, v, vertex.getOldRange(), d, b, step, vx, vy, vz, bForward);

	/*if (row == 4591)
	printf("v = %f,%f,%f \n", vx, vy, vz);*/
	float dist = sqrt(vx*vx + vy*vy + vz*vz);

	vx = (vx / (dist*4.0))*step->x;
	vy = (vy / (dist*4.0))*step->y;
	vz = (vz / (dist*4.0))*step->z;
	/*_v = _v / 4.0;
	_v *= *step;*/
	//glColor3f(rgb[0], rgb[1], rgb[2]);    
	//	Point e = vertex.e;
	//if (bForward)
	{
		vertex.e.x += vx;
		vertex.e.y += vy;
		vertex.e.z += vz;
	}
	//vertex.e += _v;
	/*else
	{
		vertex.eb.x -= vx;
		vertex.eb.y -= vy;
		vertex.eb.z -= vz;
	}*/
	//vertex.eb -= _v;

	float ep[3];
	//generalstreamlineTracing_single(p1, bForward, ep, false);
	


	bool xy = false;
	bool yz = false;
	bool xz = false;
//	if (bForward)
	{
		if (vertex.checkInBoundary(b))
		{
			uint32_t range = vertex.getRange(b, step, d);
			if (vertex.isInNextLevel_yz())
				yz = true;
			if (vertex.isInNextLevel_xy())xy = true;
			if (vertex.isInNextLevel_xz())xz = true;
			vertex.setRange(range);
			if (xy)	vertex.setInNextLevel_xy();
			if (xz)vertex.setInNextLevel_xz();
			if (yz)vertex.setInNextLevel_yz();

		}
		else
		{
			vertex.unsetInBoundary();
		}
	}
	/*else
	{
		if (vertex.checkInBoundaryBackward(b))
		{
			if (vertex.isInNextLevel_yz())
				yz = true;
			if (vertex.isInNextLevel_xy())xy = true;
			if (vertex.isInNextLevel_xz())xz = true;
			uint32_t range = vertex.getRangeBackward(b, step, d);
			vertex.setRangeBackward(range);
			if (xy)	vertex.setInNextLevel_xy();
			if (xz)vertex.setInNextLevel_xz();
			if (yz)vertex.setInNextLevel_yz();

		}
		else
		{
			vertex.unsetInBoundary();
		}
	}*/




	m[row] = vertex;

	return;
}





__global__ void ADPSynchKernel_X(ASF_vertex * m, const uint32_t OldRange, const Edge * Bc, const uint32_t * Br, uint32_t * pivot,
	const uint32_t num_rows, uint32_t * COL_pivot = NULL)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	ASF_vertex vertex = m[row];


	if (!vertex.isInFWD() || vertex.getOldRange() != OldRange)
		return;

	if (vertex.isFWDVisited()) {
		if (row == vertex.getRange()) {
			if (COL_pivot)
				*COL_pivot = row;

			uint32_t row_begin = Br[row];
			uint32_t row_end = Br[row + 1];

			bool skip = false;
			for (uint32_t column = row_begin; column < row_end; column++) {
				uint32_t index = Bc[column].getValue();
				ASF_vertex p_vertex = m[index];

				if (p_vertex.isInFWD() && p_vertex.getOldRange() == vertex.getOldRange() && p_vertex.isFWDVisited()) {
					skip = true;
					break;
				}
			}
			if (skip) {
				vertex.setInBWD();
				vertex.setBWDVisited();
			}
			else {
				vertex.setInOWCTY();
				vertex.setReached();
			}
			vertex.setOldRange(vertex.getRange());
			vertex.setDone2();

			m[row] = vertex;
		}
	}
	else {
		if (!*pivot)
			*pivot = row;
	}
}


__global__ void cuReduceEdges1(uint32_t *g_idata, uint32_t * g_odata)
{
	__shared__ uint32_t sdata[48];
	volatile uint32_t * stest = sdata;
	uint32_t tid = threadIdx.x;
	stest[tid] = g_idata[tid] + g_idata[tid + 32];
	//	stest[ tid ] = g_idata[ tid ];
	// 	sdata[ tid ] += sdata[ tid + 16 ];
	// 	sdata[ tid ] += sdata[ tid + 8 ];
	// 	sdata[ tid ] += sdata[ tid + 4 ];
	// 	sdata[ tid ] += sdata[ tid + 2 ];
	// 	sdata[ tid ] += sdata[ tid + 1 ];
	//
	// 	if ( tid == 0 )
	// 		*g_odata = sdata[ 0 ];

	stest[tid] += stest[tid + 16];
	stest[tid] += stest[tid + 8];
	stest[tid] += stest[tid + 4];
	stest[tid] += stest[tid + 2];
	stest[tid] += stest[tid + 1];

	if (tid == 0)
		*g_odata = sdata[0];
}


__host__ __device__ void cuReduceEndpoints_C(ASF_vertex* m, uint32_t *Fr, Edge *Fc, bool bForward, const uint32_t n)
{
	/*uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > n)
	return;*/
	ASF_vertex vertex = m[0];

	for (uint32_t row = 0; row < n; row++)
	{
		vertex = m[row];

		//Edge edge ;
		if (vertex.isInBoundary())
		{
		//	if (bForward)
				Fc[Fr[row]].setValue(vertex.getRange());
			/*else
				Fc[Fr[row]].setValue(vertex.getRangeBackward());*/


			Fc[Fr[row]].setValidBit();

			//Fc[Fr[row]] = edge;
		}

	}

}

__global__ void cuReduceEndpoints(ASF_vertex* m, uint32_t *Fr, Edge *Fc, bool bForward, const uint32_t n)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > n)
		return;
	ASF_vertex vertex = m[row];
	//Edge edge ;
	if (vertex.isInBoundary())
	{
		//if (bForward)
			Fc[Fr[row]].setValue(vertex.getRange());
		/*else
			Fc[Fr[row]].setValue(vertex.getRangeBackward());*/


		Fc[Fr[row]].setValidBit();

		//Fc[Fr[row]] = edge;
	}



}


__global__ void cuReduceEdges_shared(ASF_vertex* m, uint32_t *Fr, Edge *Fc, bool bForward, const uint32_t n)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > n)
		return;
	__shared__ unsigned char _sa[32786];
	ASF_vertex vertex = m[row];
	//Edge edge ;
	if (vertex.isInBoundary())
	{
		//if (bForward)
			Fc[Fr[row]].setValue(vertex.getRange());
		/*else
			Fc[Fr[row]].setValue(vertex.getRangeBackward());*/


		Fc[Fr[row]].setValidBit();

		//Fc[Fr[row]] = edge;
	}




}



__global__ void cuReduceEdges(ASF_vertex* m, uint32_t *Fr, Edge *Fc, bool bForward, const uint32_t n)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > n)
		return;
	ASF_vertex vertex = m[row];
	//Edge edge ;
	if (vertex.isInBoundary())
	{
	//	if (bForward)
			Fc[Fr[row]].setValue(vertex.getRange());
	/*	else
			Fc[Fr[row]].setValue(vertex.getRangeBackward());*/


		Fc[Fr[row]].setValidBit();

		//Fc[Fr[row]] = edge;
	}




}



__host__ __device__ bool checkEdge(ASF_vertex vertex1, ASF_vertex vertex2, Dimension* d, bool bForward)
{
	int range1 = vertex1.getRange();
	int range2 = vertex2.getRange();


	/*if (!bForward)
	{
		range1 = vertex1.getRangeBackward();
		range2 = vertex2.getRangeBackward();
	}
*/


	int iz1 = range1 / (d->x*d->y);
	int iy1 = (range1 - iz1*(d->x*d->y)) / d->x;
	int ix1 = (range1 - iz1*(d->x*d->y)) % d->x;

	int iz2 = range2 / (d->x*d->y);
	int iy2 = (range2 - iz2*(d->x*d->y)) / d->x;
	int ix2 = (range2 - iz2*(d->x*d->y)) % d->x;

	float dist = 0.;
	dist = sqrt((float)((iz2 - iz1)*(iz2 - iz1) + (iy2 - iy1)*(iy2 - iy1) + (ix2 - ix1)*(ix2 - ix1)));

	/*if (iz1 == d->z || iz1 == d->z - 1 || iy1 == d->y || iy1 == d->y - 1 || ix1 == d->x || ix1 == d->x - 1)
	printf("");*/

	/*if (iz2 == d->z || iz2 == d->z - 1 || iy2 == d->y || iy2 == d->y - 1 || ix2 == d->x || ix2 == d->x - 1)
	printf("");*/


	//printf("%f --", dist);

	//return false;

	if (dist <= 2) //abs(iz1 - iz2) <= 2 && abs(iy2 - iy1) <= 2 && abs(ix2 - ix1) <= 2)

		return true;
	//if (!bForward)
	//	printf("%d ... %d,%d,..., %d,%d,%d,...,%d,%d,%d \n",vertex1.getOldRange() ,range1,range2, iz1, iy1, ix1,iz2,iy2,ix2);

	return false;




}





//pair <uint32_t, float> Flow_Combinatorialization(ASF_vertex* _a,Point* _v, Boundary* _b, Dimension* _d, uint32_t  _tau, Edge ** oFc, uint32_t ** oFr, uint32_t * oRSize,uint32_t whichData, bool bForward)
//{
//
//	//-----------GPU initialization---------------------------->
//	uint32_t  * d_Fr;
//	uint32_t  * d_To;
//
//	Edge * d_Fc;
//
//	ASF_vertex * d_m;
//	Boundary *d_b;
//	Dimension* d_d;
//	Point* d_step;
//	Point* d_v;
//	float temp;
//
//	
//	uint32_t terminate = 1;
//	int interruptions = 0;
//	uint32_t CSize = _d->x*_d->y*_d->z;
//
//	uint32_t currentSize = _d->x*_d->y*_d->z;
//#ifdef _DEBUG
//	int FWD_ints = 0;
//	int OWCTY_ints = 0;
//	int BWD_ints = 0;
//	StopWatchInterface* KernelTime = 0;
//	StopWatchInterface* IntTime = 0;
//	(sdkCreateTimer(&KernelTime));
//	(sdkCreateTimer(&IntTime));
//#endif
//
//	//if (!_DeviceSet) {
//	//	_DeviceSet = true;
//	//	checkCudaErrors(hipSetDevice(1));
//	//}
//
//
//	hipError_t e1, e2, e3, e4, e5, e6, e7;
//	checkCudaErrors(e1 = hipMalloc((void**)&d_b, sizeof(Boundary)));
//	checkCudaErrors(e2 = hipMalloc((void**)&d_d, sizeof(Dimension)));
//	checkCudaErrors(e3 = hipMalloc((void**)&d_v, currentSize* sizeof(Point)));
//	checkCudaErrors(e4 = hipMalloc((void**)&d_step, sizeof(Point)));
//
//
//	//checkCudaErrors(e4 = hipMalloc((void**)&d_Br, RSize * sizeof(uint32_t)));
//	checkCudaErrors(e4 = hipMalloc((void**)&d_Fr, CSize * sizeof(uint32_t)));
//	checkCudaErrors(e4 = hipMalloc((void**)&d_To, CSize * sizeof(uint32_t)));
//
//
//	checkCudaErrors(e1 = hipMalloc((void**)&d_b, sizeof(Boundary)));
//
//
//
//
//	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory ||
//		e3 == hipErrorOutOfMemory || e4 == hipErrorOutOfMemory ||
//		e5 == hipErrorOutOfMemory || e6 == hipErrorOutOfMemory ||
//		e7 == hipErrorOutOfMemory) {
//		throw "Error: Not enough memory on GPU\n";
//	}
//
//	uint32_t* To = new uint32_t[CSize];
//	uint32_t* From = new uint32_t[CSize];
//	uint32_t* Fr;// = new uint32_t[CSize];
//	//To2 = new uint32_t[CSize];
//	//col
//	//unsigned int COLTime = 0;
//	StopWatchInterface* COLTime = 0;
//	(sdkCreateTimer(&COLTime));
//	uint32_t * d_temp_COL;
//	uint32_t * d_temp_COL2;
//	uint32_t * d_COL_OldRange;
//	bool COL_used = false;
//	COL_vertex * d_cm;
//
//
//
//	Point step = (_b->high - _b->low);
//	step /= (*_d);;
//
//	float radius = step.x;
//	checkCudaErrors(hipMemcpy(d_b, _b, sizeof(Boundary), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_d, _d, sizeof(Dimension), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_step, &step, sizeof(Dimension), hipMemcpyHostToDevice));
//
//	//checkCudaErrors(hipMemcpy(d_m, _a, (CSize)* sizeof(ASF_vertex), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_v, _v, (currentSize)* sizeof(Point), hipMemcpyHostToDevice));
//
//	//unsigned int SCCTime = 0;
//	StopWatchInterface* SCCTime = 0;
//	(sdkCreateTimer(&SCCTime));
//	(sdkStartTimer(&SCCTime));
//
//	dim3 grid(((CSize + 510) / 512), 1, 1);  // (RSize-2) valid vertecis the first one is undefined 
//	dim3 threads(512, 1, 1);
//	dim3 grid1(((CSize + 255) / 256), 1, 1);
//	dim3 threads1(blockSize, 1, 1);
//	dim3 grid2(1, 1, 1);
//	dim3 threads2(32, 1, 1);
//
//	{
//
//		printf("Vertices: %u\n", CSize - 2);
//		//printf("Edges: %u\n", CSize);
//	}
//	//-----------Main algorithm-------------------------------->
//
//	//-----------Trimming-------------------------------------->
//	//if (trimm) 
//
//	
//	//Point _v;
//	uint32_t currTau = _tau;
//	uint32_t tits = 1;
//	uint32_t currSize = CSize;
//	uint32_t icycle = currTau / tits;
//	ASF_vertex * currA;
//	currA = _a;
//	//checkCudaErrors(e5 = hipMalloc((void**)&d_m, (CSize)* sizeof(ASF_vertex)));
//	
//	
//	checkCudaErrors(e5 = hipMalloc((void**)&d_To, (currSize)* 30*sizeof(uint32_t)));
//	checkCudaErrors(e6 = hipMalloc((void**)&d_Fr, (currSize)* 30*sizeof(uint32_t)));
//	checkCudaErrors(e5 = hipMalloc((void**)&d_m, (currSize)*30* sizeof(ASF_vertex)));
//	checkCudaErrors(hipMemcpy(d_m, _a, (currSize)* sizeof(ASF_vertex), hipMemcpyHostToDevice));
//	Initialize << <grid, threads >> >(d_m, d_d, d_b, d_step, bForward, currSize);
//	//checkCudaErrors(hipMemcpy(_a, d_m, (CSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
//
//
//	uint32_t* Fr2 = new uint32_t[CSize * 30];
//	Fr = new uint32_t[CSize * 30];
//	uint32_t* Fi = new uint32_t[CSize*30];
//	ASF_vertex* _aa;
//	//ASF_vertex* tempa = new ASF_vertex[41898];;
//	_aa = new ASF_vertex[CSize*30];
//	currA = _aa;
//	memcpy(currA, _a, currSize*sizeof(ASF_vertex));
//
//	uint32_t oldsize = CSize;
//	int level = 0;
//	//To2 = To;
//	//Fr2 = Fi ;
//
//	do{
//		 tits = 1;
//
//		 dim3 grid(((currSize + 510) / 512), 1, 1);  // (RSize-2) valid vertecis the first one is undefined 
//
//		do {
//
//			//checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
//
//			//if (!bForward)
//			//	_Tracing_c (currA,_v, _d, _b, &step, bForward, whichData, CSize, currSize,level );
//			Tracing << <grid, threads >> >(d_m, d_v, d_d, d_b, d_step, bForward, whichData,oldsize, currSize,level );
//			
//			
//
//
//			tits--;
//		} while (tits > 0);// && terminate);
//		tits = 1;
//
//		//if (Fr[CSize - 1] > 0)
//		{
//		//	checkCudaErrors(hipMemcpy(currA, d_m, (currSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
//			/*if (icycle == 70)
//				_CheckNeighborhood_c3(currA, Fr, Fi, _d, _b, &step, bForward, oldsize, currSize, level);*/
//			//_CheckNeighborhood_c3(currA, Fr, Fi, _d, _b, &step, bForward, oldsize, currSize, level);
////			_CheckNeighborhood << <grid, threads >> >(d_m, d_To,  d_d, d_b, d_step, bForward, oldsize, currSize, level);
//
//		}
//
//		//_CheckNeighborhood_c(currA, Fr, _d, _b, &step, bForward, CSize, currSize);
//	
//		
//		//CheckConnectivityKernel_c(currA, To, _d, CSize);
//
//		//checkCudaErrors(hipMemcpy(d_m, currA, (currSize)* sizeof(ASF_vertex), hipMemcpyHostToDevice));
//		//checkCudaErrors(hipMemcpy(d_To, Fi, (currSize)* sizeof(uint32_t), hipMemcpyHostToDevice));
//
//	/*	if (currA[32772].Fr_xy.z > currSize)
//			printf("");*/
//		//CheckConnectivityKernel << <grid, threads >> >(d_m, d_To,d_d, currSize );
//
//		//checkCudaErrors(hipMemcpy(currA, d_m, (currSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
//		
//		/*CheckConnectivityKernel_c (_a, To, CSize - 1);*/
//		checkCudaErrors(hipMemcpy(Fi, d_To, (currSize) * sizeof(uint32_t), hipMemcpyDeviceToHost));
////		checkCudaErrors(hipMemcpy(d_To, To, (CSize) * sizeof(uint32_t), hipMemcpyHostToDevice));
//		//To2 = Fi;
////		reduction(d_To, d_Fr, Fi, Fr2, currSize);
//		checkCudaErrors(hipMemcpy(d_Fr, Fr2, (currSize)* sizeof(uint32_t), hipMemcpyHostToDevice));
//		//if (icycle == 5)
//		//	printf("");
//		if (Fr2[currSize - 1] > 0)
//		{
//				
//			printf("%d \n", Fr2[currSize - 1]);
//			checkCudaErrors(hipMemcpy(currA, d_m, (currSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
////			InsertSeedPoint << <grid, threads >> > (d_m, d_Fr, d_d, d_step, (currTau / tits) - icycle, currSize, bForward);
//			
//			oldsize = currSize;
//			currSize = currSize + Fr2[currSize - 1] * 5;
//
//			//checkCudaErrors(hipMemcpy(d_m, currA, (currSize)* sizeof(ASF_vertex), hipMemcpyHostToDevice));
//			//memset(Fi, 0, currSize*sizeof(uint32_t));
//
//			icycle++;
//			level = 1;
//
//
//		}
//		else
//			level = 0;
//
//	/*	else if (currSize != oldsize)
//			currA = _aa;*/
//		//	Initialize2 << <grid, threads >> >(d_m, d_d, d_b, d_step, bForward, currSize);*/
//		printf(" \n %d \n", icycle);
//		icycle--;
//
//
//	} while (icycle > 0 );// && terminate);
//
//	dim3 grid_2(((currSize + 510) / 512), 1, 1);  // (RSize-2) valid vertecis the first one is undefined 
//
//		//checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
//		checkCudaErrors(e3 = hipMalloc((void**)&d_Fc, currSize * sizeof(Edge)));
//		Edge* Fc = new Edge[currSize];
//		checkCudaErrors(hipMemcpy(d_m, currA, (currSize)* sizeof(ASF_vertex), hipMemcpyHostToDevice));
//
//		checkCudaErrors(hipFree(d_To));
//		checkCudaErrors(hipFree(d_Fr));
//		checkCudaErrors(e5 = hipMalloc((void**)&d_To, (currSize)* sizeof(uint32_t)));
//		checkCudaErrors(e6 = hipMalloc((void**)&d_Fr, (currSize)* sizeof(uint32_t)));
//		checkCudaErrors(hipMemcpy(currA, d_m, (currSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
//
//		memset(Fi, 0, currSize*sizeof(uint32_t));
//		for (int i = 0; i < currSize; i++)
//		{
//			ASF_vertex vertex = currA[i];
//			if (vertex.isInBoundary())
//				Fi[vertex.getOldRange()]++;
//		}
//		//CheckRangeSetKernel << <grid_2, threads >> >(d_m, d_To, currSize - 1);
//		To = new uint32_t[currSize];
//		//checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
//	
//
//		//checkCudaErrors(hipMemcpy(To, d_To, currSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
//
//		//checkCudaErrors(hipMemcpy(d_To, Fi, (currSize)* sizeof(uint32_t), hipMemcpyHostToDevice));
//
////		reduction(d_To, d_Fr, Fi, Fr, CSize);
//		memset(Fi, 0, CSize*sizeof(uint32_t));
//		checkCudaErrors(hipMemcpy(d_Fr, Fr, currSize * sizeof(uint32_t), hipMemcpyHostToDevice));
//		for (int row = 0; row < currSize;row++)
//		{
//			ASF_vertex vertex = currA[row];
//			if (vertex.isInBoundary())
//			{
//				uint32_t i = Fr[vertex.getOldRange()] + Fi[vertex.getOldRange()] - 1;
//				if (bForward)
//					Fc[i].setValue(vertex.getRange());
//				else
//					Fc[i].setValue(vertex.getRangeBackward());
//
//				Fc[i].setValidBit();
//				Fi[vertex.getOldRange()]++;
//			}
//					
//		}
//	//	cuReduceEdges << <grid_2, threads >> >(d_m, d_Fr, d_Fc, bForward, currSize);
//		//cuReduceEdges_shared << <grid_2, threads >> >(d_m, d_Fr, d_Fc, bForward, currSize);
//
//	//checkCudaErrors(hipMemcpy(Fc, d_Fc, CSize * sizeof(Edge), hipMemcpyDeviceToHost));
//
//
//	
//
//	//===============================================================================
//	//	checkCudaErrors(hipMemcpy(_a, d_m, (currSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
//
//
//	//	checkCudaErrors(hipMemcpy(To, d_To, CSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
//	//	checkCudaErrors(hipMemcpy(From, d_From, CSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
//	//	checkCudaErrors(hipMemcpy(Fr, d_Fr, currSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
//	//	checkCudaErrors(hipMemcpy(Fc, d_Fc, currSize * sizeof(Edge), hipMemcpyDeviceToHost));
//
//		uint32_t r = currA[14731].getRange();
//
//		if (!bForward)
//			r = currA[14731].getRangeBackward();
//	
////	checkCudaErrors(hipMemcpy(To, d_To, (1) * sizeof(uint32_t), hipMemcpyDeviceToHost));
////	checkCudaErrors(hipMemcpy(_a, d_m, (CSize)* sizeof(ASF_vertex), hipMemcpyDeviceToHost));
//
//
//
//
//	*oFc = Fc;
//	*oFr = Fr;
//	*oRSize = Fr[CSize - 1];
//
//	//<----------Trimming---------------------------------------
//
//
//	pair <uint32_t, float> result;
//	return result;
//}
////{


//__global__ InsertSeedPoint(ASF_vertex* m, fEdge*eg, uint32_t * Fr, Dimension*d, Point* step,  uint32_t num_vertexes, uint32_t numEdges, bool bForward)
//{
//
//
//	ASF_vertex vertex1 = m[0];
//	//	om[0] = vertex1;
//
//	//for (uint32_t row = 1; row < num_rows; row++)
//	{
//
//		/* if (row < 2*originalnumrows)
//		om[row] = vertex1;*/
//		if ((row >= 1 && (Fr[row] - Fr[row - 1]) == 0))
//			continue;
//
//		//if ((Fr[row] - Fr[row - 1]) == 0)
//		{
//
//			int curVertexIdx = originalnumrows + Fr[row];
//			int curEdgeIdx = num_rows + Fr[row] * 3;
//			int curtriangleIdx = trianglenum + Fr[row] * 2;
//
//			fEdge edge1 = eg[row];
//			fEdge edge2 = eg[row];
//			fEdge edge3 = eg[row];
//			fEdge edge4 = eg[row];
//			ASF_vertex vertex1 = m[edge1.v1];
//			ASF_vertex vertex2 = m[edge1.v2];
//			ASF_vertex vedge = DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), 0, bForward);
//			m[curVertexIdx] = vedge;
//
//			int triangleIdx1 = edge1.E2T[0];
//			int triangleIdx2 = edge1.E2T[1];
//
//			if (vertex1.getOldRange() == 4529 || vertex2.getOldRange() == 4529)
//				printf("");
//
//			fTriangle t1 = tr[triangleIdx1];
//			if (row == 179098)
//				printf("");
//			if (curtriangleIdx + 1 == 131306 || curtriangleIdx == 131306)// || triangleIdx2 == 14393 || curtriangleIdx == 14393 || curtriangleIdx == 14392)
//				printf("");
//			fTriangle to = tr[triangleIdx1];
//			fTriangle t2 = tr[triangleIdx2];
//			fTriangle to2 = tr[triangleIdx2];
//			fTriangle t3 = tr[triangleIdx1];
//			fTriangle t4 = tr[triangleIdx2];
//
//
//			if (to.edge[0] != row && to.edge[1] != row && to.edge[2] != row)
//				printf("");
//
//
//			if (t1.edge[2] == row)
//			{
//				t1.edge[0] = to.edge[2];
//				t1.edge[1] = curEdgeIdx + 1;
//
//				if (eg[to.edge[1]].v1 == edge1.v1 || eg[to.edge[1]].v2 == edge1.v1)
//				{
//					t1.edge[2] = to.edge[1];
//					t3.edge[1] = to.edge[0];
//
//				}
//				else if (eg[to.edge[0]].v1 == edge1.v1 || eg[to.edge[0]].v2 == edge1.v1)
//				{
//					t1.edge[2] = to.edge[0];
//					t3.edge[1] = to.edge[1];
//
//				}
//
//				t3.edge[0] = curEdgeIdx;
//				t3.edge[2] = curEdgeIdx + 1;
//
//
//			}
//
//			else if (t1.edge[1] == row)
//			{
//				t1.edge[0] = to.edge[1];
//				t1.edge[1] = curEdgeIdx + 1;
//				//t1.edge[2] = to.edge[0];
//
//				if (eg[to.edge[0]].v1 == edge1.v1 || eg[to.edge[0]].v2 == edge1.v1)
//				{
//					t1.edge[2] = to.edge[0];
//					t3.edge[1] = to.edge[2];
//
//				}
//				else if (eg[to.edge[2]].v1 == edge1.v1 || eg[to.edge[2]].v2 == edge1.v1)
//				{
//					t1.edge[2] = to.edge[2];
//					t3.edge[1] = to.edge[0];
//
//				}
//
//				t3.edge[0] = curEdgeIdx;
//				//t3.edge[1] = to.edge[2];
//				t3.edge[2] = curEdgeIdx + 1;
//
//			}
//
//			else if (t1.edge[0] == row)
//			{
//				t1.edge[0] = to.edge[0];
//				t1.edge[1] = curEdgeIdx + 1;
//				//	t1.edge[2] = to.edge[2];
//
//
//				if (eg[to.edge[1]].v1 == edge1.v1 || eg[to.edge[1]].v2 == edge1.v1)
//				{
//					t1.edge[2] = to.edge[1];
//					t3.edge[1] = to.edge[2];
//
//				}
//				else if (eg[to.edge[2]].v1 == edge1.v1 || eg[to.edge[2]].v2 == edge1.v1)
//				{
//					t1.edge[2] = to.edge[2];
//					t3.edge[1] = to.edge[1];
//
//				}
//
//				t3.edge[0] = curEdgeIdx;
//				//	t3.edge[1] = to.edge[1];
//				t3.edge[2] = curEdgeIdx + 1;
//
//
//
//			}
//
//			//=====================================
//
//			edge1.v2 = curVertexIdx;
//			edge2.v1 = curVertexIdx;
//
//			edge3.v1 = curVertexIdx;
//			edge4.v1 = curVertexIdx;
//
//			if (row == 178834)
//				printf("");
//
//			if (t2.edge[2] == row)
//			{
//				t2.edge[0] = to2.edge[2];
//				t2.edge[1] = curEdgeIdx + 2;
//				if (eg[to2.edge[0]].v1 == edge1.v1 || eg[to2.edge[0]].v2 == edge1.v1)
//				{
//					t2.edge[2] = to2.edge[0];
//					t4.edge[1] = to2.edge[1];
//				}
//				else if (eg[to2.edge[1]].v1 == edge1.v1 || eg[to2.edge[1]].v2 == edge1.v1)
//				{
//					t2.edge[0] = to2.edge[1];
//					t4.edge[1] = to2.edge[0];
//
//				}
//
//
//				t4.edge[0] = curEdgeIdx;
//				t4.edge[2] = curEdgeIdx + 2;
//
//
//			}
//
//			else if (t2.edge[1] == row)
//			{
//				t2.edge[0] = to2.edge[1];
//				t2.edge[1] = curEdgeIdx + 2;
//				//t2.edge[2] = to2.edge[0];
//
//				if (eg[to2.edge[0]].v1 == edge1.v1 || eg[to2.edge[0]].v2 == edge1.v1)
//				{
//					t2.edge[2] = to2.edge[0];
//					t4.edge[1] = to2.edge[2];
//				}
//				else if (eg[to2.edge[2]].v1 == edge1.v1 || eg[to2.edge[2]].v2 == edge1.v1)
//				{
//					t2.edge[2] = to2.edge[2];
//					t4.edge[1] = to2.edge[0];
//
//				}
//
//				t4.edge[0] = curEdgeIdx;
//				//t4.edge[1] = to2.edge[2];
//				t4.edge[2] = curEdgeIdx + 2;
//
//			}
//
//			else if (t2.edge[0] == row)
//			{
//				t2.edge[0] = to2.edge[0];
//				t2.edge[1] = curEdgeIdx + 2;
//
//				if (eg[to2.edge[1]].v1 == edge1.v1 || eg[to2.edge[1]].v2 == edge1.v1)
//				{
//					t2.edge[2] = to2.edge[1];
//					t4.edge[1] = to2.edge[2];
//				}
//				else if (eg[to2.edge[2]].v1 == edge1.v1 || eg[to2.edge[2]].v2 == edge1.v1)
//				{
//					t2.edge[2] = to2.edge[2];
//					t4.edge[1] = to2.edge[1];
//
//				}
//				//t2.edge[2] = to2.edge[0];
//
//				t4.edge[0] = curEdgeIdx;
//				//t4.edge[1] = to2.edge[2];
//				t4.edge[2] = curEdgeIdx + 2;
//
//			}
//
//
//
//
//			if (to.edge[0] == row)
//			{
//				if ((eg[to.edge[2]].v1 == edge1.v1) /*|| (eg[to.edge[2]].v2 == edge2.v2)*/)
//					edge3.v2 = eg[to.edge[2]].v2;
//				else if ((eg[to.edge[2]].v2 == edge1.v1))
//					edge3.v2 = eg[to.edge[2]].v1;
//
//
//				else if ((eg[to.edge[1]].v1 == edge1.v1) /*|| (eg[to.edge[2]].v2 == edge2.v2)*/)
//					edge3.v2 = eg[to.edge[1]].v2;
//				else if ((eg[to.edge[1]].v2 == edge1.v1))
//					edge3.v2 = eg[to.edge[1]].v1;
//
//				else if (t1.edge[0] == row)
//					printf("");
//
//			}
//
//
//			else if (to.edge[1] == row)
//			{
//				if (eg[to.edge[2]].v2 == edge1.v1 /*|| eg[to.edge[2]].v2 == edge1.v2*/)
//					edge3.v2 = eg[to.edge[2]].v1;
//				else if ((eg[to.edge[2]].v1 == edge1.v1 /*|| eg[to.edge[2]].v1 == edge1.v2*/))
//					edge3.v2 = eg[to.edge[2]].v2;
//
//				else if (eg[to.edge[0]].v2 == edge1.v1 /*|| eg[to.edge[2]].v2 == edge1.v2*/)
//					edge3.v2 = eg[to.edge[0]].v1;
//				else if ((eg[to.edge[0]].v1 == edge1.v1 /*|| eg[to.edge[2]].v1 == edge1.v2*/))
//					edge3.v2 = eg[to.edge[0]].v2;
//
//
//				else if (t1.edge[1] == row)
//					printf("");
//			}
//
//
//
//			else if (to.edge[2] == row)
//			{
//				if (eg[to.edge[0]].v2 == edge1.v1/* || eg[to.edge[0]].v2 == edge1.v2*/)
//					edge3.v2 = eg[to.edge[0]].v1;
//				else if (eg[to.edge[0]].v1 == edge1.v1 /*|| eg[to.edge[0]].v1 == edge1.v2)*/)
//					edge3.v2 = eg[to.edge[0]].v2;
//
//				else if (eg[to.edge[1]].v2 == edge1.v1/* || eg[to.edge[0]].v2 == edge1.v2*/)
//					edge3.v2 = eg[to.edge[1]].v1;
//				else if (eg[to.edge[1]].v1 == edge1.v1 /*|| eg[to.edge[0]].v1 == edge1.v2)*/)
//					edge3.v2 = eg[to.edge[1]].v2;
//
//				else if (t1.edge[2] == row)
//					printf("");
//
//			}
//
//			//==========================================================================================
//
//
//
//			if (to2.edge[0] == row)
//			{
//				if (eg[to2.edge[1]].v2 == edge1.v1 /*|| eg[to2.edge[1]].v2 == edge1.v2*/)
//					edge4.v2 = eg[to2.edge[1]].v1;
//				else if (eg[to2.edge[1]].v1 == edge1.v1 /*|| eg[to2.edge[1]].v1 == edge1.v2)*/)
//					edge4.v2 = eg[to2.edge[1]].v2;
//
//				else if (eg[to2.edge[2]].v2 == edge1.v1 /*|| eg[to2.edge[1]].v2 == edge1.v2*/)
//					edge4.v2 = eg[to2.edge[2]].v1;
//				else if (eg[to2.edge[2]].v1 == edge1.v1 /*|| eg[to2.edge[1]].v1 == edge1.v2)*/)
//					edge4.v2 = eg[to2.edge[2]].v2;
//
//
//				else if (t2.edge[0] == row)
//					printf("");
//			}
//
//
//
//			else if (to2.edge[1] == row)
//			{
//				if (eg[to2.edge[2]].v2 == edge1.v1 /*|| eg[to2.edge[2]].v1 == edge1.v1*/)
//					edge4.v2 = eg[to2.edge[2]].v1;
//				else if (eg[to2.edge[2]].v1 == edge1.v1 /*|| eg[to2.edge[2]].v1 == edge1.v2)*/)
//					edge4.v2 = eg[to2.edge[2]].v2;
//
//				else if (eg[to2.edge[0]].v2 == edge1.v1 /*|| eg[to2.edge[2]].v1 == edge1.v1*/)
//					edge4.v2 = eg[to2.edge[0]].v1;
//				else if (eg[to2.edge[0]].v1 == edge1.v1 /*|| eg[to2.edge[2]].v1 == edge1.v2)*/)
//					edge4.v2 = eg[to2.edge[0]].v2;
//
//				else if (t2.edge[1] == row)
//					printf("");
//			}
//
//
//			else if (to2.edge[2] == row)
//			{
//				if (eg[to2.edge[1]].v2 == edge1.v1 /*|| eg[to2.edge[1]].v2 == edge1.v1*/)
//					edge4.v2 = eg[to2.edge[1]].v1;
//				else if (eg[to2.edge[1]].v1 == edge1.v1/* || eg[to2.edge[0]].v2 == edge1.v1*/)
//					edge4.v2 = eg[to2.edge[1]].v2;
//
//				else if (eg[to2.edge[0]].v2 == edge1.v1 /*|| eg[to2.edge[1]].v2 == edge1.v1*/)
//					edge4.v2 = eg[to2.edge[0]].v1;
//				else if (eg[to2.edge[0]].v1 == edge1.v1/* || eg[to2.edge[0]].v2 == edge1.v1*/)
//					edge4.v2 = eg[to2.edge[0]].v2;
//
//
//				else if (t2.edge[2] == row)
//					printf("");
//			}
//
//			edge1.v2 = curVertexIdx;
//			edge2.v1 = curVertexIdx;
//
//			edge3.v1 = curVertexIdx;
//			edge4.v1 = curVertexIdx;
//
//			edge2.E2T[0] = curtriangleIdx;
//			edge2.E2T[1] = curtriangleIdx + 1;
//
//			edge3.E2T[0] = triangleIdx1;;
//			edge3.E2T[1] = curtriangleIdx;
//
//			edge4.E2T[0] = triangleIdx2;;
//			edge4.E2T[1] = curtriangleIdx + 1;
//			eg[row] = edge1;
//			eg[curEdgeIdx] = edge2;
//			eg[curEdgeIdx + 1] = edge3;
//			eg[curEdgeIdx + 2] = edge4;
//
//			t3.T2F = to.T2F;
//			t4.T2F = to2.T2F;
//
//
//			if (eg[t4.edge[1]].E2T[0] == triangleIdx2)
//				eg[t4.edge[1]].E2T[0] = curtriangleIdx + 1;
//			else if (eg[t4.edge[1]].E2T[1] == triangleIdx2)
//				eg[t4.edge[1]].E2T[1] = curtriangleIdx + 1;
//
//			if (eg[t3.edge[1]].E2T[0] == triangleIdx1)
//				eg[t3.edge[1]].E2T[0] = curtriangleIdx;
//			else if (eg[t3.edge[1]].E2T[1] == triangleIdx1)
//				eg[t3.edge[1]].E2T[1] = curtriangleIdx;
//			if (curtriangleIdx == 131272 || curtriangleIdx + 1 == 131272)
//				printf("");
//
//			if (to.T2F != to2.T2F)
//				printf("");
//			tr[triangleIdx1] = t1;
//			tr[triangleIdx2] = t2;
//
//			tr[curtriangleIdx] = t3;
//			tr[curtriangleIdx + 1] = t4;
//
//			float rgb[3];
//			rgb[0] = 1.0;
//			rgb[0] = 0.0;
//			rgb[0] = 0.0;
//
//			if (t1.edge[0] != row)
//				printf("");
//
//			if (t1.edge[0] == t1.edge[1] || t1.edge[0] == t1.edge[2] || t1.edge[2] == t1.edge[1])
//				printf("");
//
//			if (t2.edge[0] == t2.edge[1] || t2.edge[0] == t2.edge[2] || t2.edge[2] == t2.edge[1])
//				printf("");
//
//			if (t3.edge[0] == t3.edge[1] || t3.edge[0] == t3.edge[2] || t3.edge[2] == t3.edge[1])
//				printf("");
//
//
//			if (t4.edge[0] == t4.edge[1] || t4.edge[0] == t4.edge[2] || t4.edge[2] == t4.edge[1])
//				printf("");
//
//
//
//			if (eg[t1.edge[0]].v2 != eg[t1.edge[1]].v1 && eg[t1.edge[0]].v2 != eg[t1.edge[1]].v2)
//			{
//				printf("");
//				if (eg[t1.edge[0]].v1 != eg[t1.edge[1]].v1 && eg[t1.edge[0]].v1 != eg[t1.edge[1]].v2)
//					printf("");
//			}
//
//
//			if (eg[t1.edge[1]].v2 != eg[t1.edge[2]].v1 && eg[t1.edge[1]].v2 != eg[t1.edge[2]].v2)
//			{
//
//				printf("");
//				if (eg[t1.edge[1]].v1 != eg[t1.edge[2]].v1 && eg[t1.edge[1]].v1 != eg[t1.edge[2]].v2)
//					printf("");
//			}
//
//
//			if (eg[t1.edge[2]].v2 != eg[t1.edge[0]].v1 && eg[t1.edge[2]].v2 != eg[t1.edge[0]].v2)
//			{
//				printf("");
//				if (eg[t1.edge[2]].v1 != eg[t1.edge[0]].v1 && eg[t1.edge[2]].v1 != eg[t1.edge[0]].v2)
//					printf("");
//			}
//
//
//			//====================================================================================
//
//			if (eg[t2.edge[0]].v2 != eg[t2.edge[1]].v1 && eg[t2.edge[0]].v2 != eg[t2.edge[1]].v2)
//			{
//
//				printf("");
//				if (eg[t2.edge[0]].v1 != eg[t2.edge[1]].v1 && eg[t2.edge[0]].v1 != eg[t2.edge[1]].v2)
//					printf("");
//			}
//
//			if (eg[t2.edge[1]].v2 != eg[t2.edge[2]].v1 && eg[t2.edge[1]].v2 != eg[t2.edge[2]].v2)
//			{
//				printf("");
//				if (eg[t2.edge[1]].v1 != eg[t2.edge[2]].v1 && eg[t2.edge[1]].v1 != eg[t2.edge[2]].v2)
//					printf("");
//			}
//
//
//			if (eg[t2.edge[2]].v2 != eg[t2.edge[0]].v1 && eg[t2.edge[2]].v2 != eg[t2.edge[0]].v2)
//			{
//				printf("");
//				if (eg[t2.edge[2]].v1 != eg[t2.edge[0]].v1 && eg[t2.edge[2]].v1 != eg[t2.edge[0]].v2)
//					printf("");
//			}
//
//
//			//======================================================================================
//
//			if (eg[t3.edge[0]].v2 != eg[t3.edge[1]].v1 && eg[t3.edge[0]].v2 != eg[t3.edge[1]].v2)
//			{
//
//				printf("");
//				if (eg[t3.edge[0]].v1 != eg[t3.edge[1]].v1 && eg[t3.edge[0]].v1 != eg[t3.edge[1]].v2)
//					printf("");
//			}
//			if (eg[t3.edge[1]].v2 != eg[t3.edge[2]].v1 && eg[t3.edge[1]].v2 != eg[t3.edge[2]].v2)
//			{
//				printf("");
//				if (eg[t3.edge[1]].v1 != eg[t3.edge[2]].v1 && eg[t3.edge[1]].v1 != eg[t3.edge[2]].v2)
//					printf("");
//			}
//
//
//			if (eg[t3.edge[2]].v2 != eg[t3.edge[0]].v1 && eg[t3.edge[2]].v2 != eg[t3.edge[0]].v2)
//			{
//				printf("");
//				if (eg[t3.edge[2]].v1 != eg[t3.edge[0]].v1 && eg[t3.edge[2]].v1 != eg[t3.edge[0]].v2)
//					printf("");
//			}
//
//
//			//===========================================================================================
//
//			if (eg[t4.edge[0]].v2 != eg[t4.edge[1]].v1 && eg[t4.edge[0]].v2 != eg[t4.edge[1]].v2)
//			{
//
//				printf("");
//				if (eg[t4.edge[0]].v1 != eg[t4.edge[1]].v1 && eg[t4.edge[0]].v1 != eg[t4.edge[1]].v2)
//					printf("");
//			}
//
//			if (eg[t4.edge[1]].v2 != eg[t4.edge[2]].v1 && eg[t4.edge[1]].v2 != eg[t4.edge[2]].v2)
//			{
//				printf("");
//				if (eg[t4.edge[1]].v1 != eg[t4.edge[2]].v1 && eg[t4.edge[1]].v1 != eg[t4.edge[2]].v2)
//					printf("");
//			}
//
//
//			if (eg[t4.edge[2]].v2 != eg[t4.edge[0]].v1 && eg[t4.edge[2]].v2 != eg[t4.edge[0]].v2)
//			{
//				printf("");
//				if (eg[t4.edge[2]].v1 != eg[t4.edge[0]].v1 && eg[t4.edge[2]].v1 != eg[t4.edge[0]].v2)
//					printf("");
//			}
//
//
//			//============================================================================================
//			continue;
//			if (eg[t1.edge[0]].v2 != eg[t1.edge[1]].v1 || eg[t1.edge[1]].v2 != eg[t1.edge[2]].v1 || eg[t1.edge[2]].v2 != eg[t1.edge[0]].v1)
//				printf("%d,%d,   %d,%d,    %d,%d \n", eg[t1.edge[0]].v2, eg[t1.edge[0]].v1, eg[t1.edge[1]].v1, eg[t1.edge[1]].v2, eg[t1.edge[2]].v2, eg[t1.edge[2]].v1);
//
//
//			if (eg[t2.edge[0]].v2 != eg[t2.edge[1]].v1 || eg[t2.edge[1]].v2 != eg[t2.edge[2]].v1 || eg[t2.edge[2]].v2 != eg[t2.edge[0]].v1)
//				printf("%d,%d,   %d,%d,    %d,%d \n", eg[t2.edge[0]].v2, eg[t2.edge[0]].v1, eg[t2.edge[1]].v1, eg[t2.edge[1]].v2, eg[t2.edge[2]].v2, eg[t2.edge[2]].v1);
//
//
//
//			if (eg[t3.edge[0]].v2 != eg[t3.edge[1]].v2 || eg[t3.edge[1]].v1 != eg[t3.edge[2]].v2 || eg[t3.edge[2]].v1 != eg[t3.edge[0]].v1)
//				printf("%d,%d,   %d,%d,    %d,%d \n", eg[t3.edge[0]].v2, eg[t3.edge[0]].v1, eg[t3.edge[1]].v1, eg[t3.edge[1]].v2, eg[t3.edge[2]].v2, eg[t3.edge[2]].v1);
//
//
//			if (eg[t4.edge[0]].v2 != eg[t4.edge[1]].v1 || eg[t4.edge[1]].v2 != eg[t4.edge[2]].v1 || eg[t4.edge[2]].v2 != eg[t4.edge[0]].v1)
//				printf("%d,%d,   %d,%d,    %d,%d \n", eg[t4.edge[0]].v2, eg[t4.edge[0]].v1, eg[t4.edge[1]].v1, eg[t4.edge[1]].v2, eg[t4.edge[2]].v2, eg[t4.edge[2]].v1);
//
//			continue;
//			if (t1.edge[0] == row && eg[t1.edge[1]].v1 == edge1.v2)
//				edge3.v2 = eg[t1.edge[1]].v2;
//			else if (t1.edge[0] == row && eg[t1.edge[1]].v2 == edge1.v2)
//				edge3.v2 = eg[t1.edge[1]].v1;
//
//
//			if (t1.edge[1] == row && eg[t1.edge[2]].v1 == edge1.v2)
//				edge3.v2 = eg[t1.edge[2]].v2;
//			else if (t1.edge[1] == row && eg[t1.edge[2]].v2 == edge1.v2)
//				edge3.v2 = eg[t1.edge[2]].v1;
//
//			if (t1.edge[2] == row && eg[t1.edge[0]].v1 == edge1.v2)
//				edge3.v2 = eg[t1.edge[0]].v2;
//			else if (t1.edge[2] == row && eg[t1.edge[0]].v2 == edge2.v2)
//				edge3.v2 = eg[t1.edge[1]].v1;
//
//
//			edge1.v2 = curVertexIdx;
//			edge2.v1 = curVertexIdx;
//
//			edge3.v1 = curVertexIdx;
//
//			t3.edge[0] = curVertexIdx;
//			if (t1.edge[0] == row)
//			{
//				if (eg[t1.edge[1]].v2 == edge3.v2)
//					t1.edge[1] = curEdgeIdx + 1;
//
//				if (eg[t3.edge[1]].v1 == edge3.v2)
//					t3.edge[2] = curEdgeIdx + 1;
//
//
//			}
//
//			if (t1.edge[1] == row)
//			{
//				if (eg[t1.edge[2]].v1 == curVertexIdx)
//					t1.edge[1] = curEdgeIdx + 1;
//
//				if (eg[t3.edge[1]].v2 == edge3.v2)
//					t1.edge[2] = curEdgeIdx + 1;
//
//
//			}
//
//			if (t1.edge[2] == row)
//			{
//				if (eg[t1.edge[2]].v1 == curVertexIdx)
//					t1.edge[1] = curEdgeIdx + 1;
//
//				if (eg[t3.edge[1]].v2 == edge3.v2)
//					t1.edge[2] = curEdgeIdx + 1;
//
//
//			}
//
//
//
//			if (t1.edge[0] == row)
//			{
//				t3.edge[0] = curEdgeIdx;
//				t3.edge[2] = curEdgeIdx + 1;
//
//				if (eg[t1.edge[2]].v1 == edge1.v1)
//					t3.edge[1] = t1.edge[1];
//				else
//					t3.edge[1] = t1.edge[2];
//
//
//				edge3.v2 = eg[t1.edge[1]].v2;
//				t1.edge[1] = curEdgeIdx + 1;
//			}
//
//			else if (t1.edge[1] == row)
//			{
//				t3.edge[0] = curEdgeIdx;
//				if (eg[t1.edge[2]].v1 == edge1.v1)
//					t3.edge[1] = t1.edge[0];
//				else
//					t3.edge[1] = t1.edge[2];
//
//				t3.edge[2] = curEdgeIdx + 1;
//
//				edge3.v2 = eg[t1.edge[2]].v2;
//				t1.edge[2] = curEdgeIdx + 1;
//
//			}
//
//			else if (t1.edge[2] == row)
//			{
//				t3.edge[0] = curEdgeIdx;
//				//t3.edge[1] = t1.edge[0];
//				t3.edge[2] = curEdgeIdx + 1;
//				if (eg[t1.edge[1]].v1 == edge1.v2)
//					t3.edge[1] = t1.edge[0];
//				else
//					t3.edge[1] = t1.edge[1];
//
//				edge3.v2 = eg[t1.edge[0]].v2;
//				t1.edge[1] = curEdgeIdx + 1;
//
//			}
//
//			edge4.v1 = curVertexIdx;
//
//			if (t2.edge[0] == row)
//			{
//				t4.edge[0] = curEdgeIdx;
//				t4.edge[2] = curEdgeIdx + 2;
//
//				if (eg[t2.edge[2]].v1 == edge1.v1)
//					t4.edge[1] = t2.edge[1];
//				else
//					t4.edge[1] = t2.edge[0];
//				edge4.v2 = eg[t2.edge[1]].v2;
//				t2.edge[1] = curEdgeIdx + 2;
//			}
//			else if (t2.edge[1] == row)
//			{
//				t4.edge[0] = curEdgeIdx;
//				t4.edge[1] = t2.edge[2];
//				t4.edge[2] = curEdgeIdx + 2;
//
//				if (eg[t4.edge[2]].v1 == edge2.v1)
//					t4.edge[1] = t2.edge[0];
//				else
//					t4.edge[1] = t2.edge[2];
//
//				edge4.v2 = eg[t2.edge[2]].v2;
//				t2.edge[2] = curEdgeIdx + 2;
//
//			}
//			else if (t2.edge[2] == row)
//			{
//				t4.edge[0] = curEdgeIdx;
//				t4.edge[1] = t2.edge[0];
//				t4.edge[2] = curEdgeIdx + 2;
//
//				if (eg[t2.edge[1]].v2 == edge2.v2 || eg[t2.edge[1]].v2 == edge2.v1)
//					t4.edge[1] = t2.edge[1];
//				else
//					t4.edge[1] = t2.edge[0];
//
//				edge4.v2 = eg[t2.edge[0]].v2;
//				t2.edge[1] = curEdgeIdx + 2;
//
//			}
//
//
//			edge2.E2T[0] = curtriangleIdx;
//			edge2.E2T[1] = curtriangleIdx + 1;
//
//			edge3.E2T[0] = triangleIdx1;;
//			edge3.E2T[1] = curtriangleIdx;
//
//			edge4.E2T[0] = triangleIdx2;;
//			edge4.E2T[1] = curtriangleIdx + 1;
//			eg[row] = edge1;
//			eg[curEdgeIdx] = edge2;
//			eg[curEdgeIdx + 1] = edge3;
//			eg[curEdgeIdx + 2] = edge4;
//
//			tr[triangleIdx1] = t1;
//			tr[triangleIdx2] = t2;
//
//			tr[curtriangleIdx] = t3;
//			tr[curtriangleIdx + 1] = t4;
//
//		}
//		continue;
//		co = 0;
//		//fTriangle _t = tr[row];
//
//		int curidx = num_rows;
//		if (vertex1.getOldRange() == 6671)
//			printf("");
//		if (vertex1.isInNextLevel_xy())
//		{
//			ASF_vertex vertex2 = m[vertex1.fx[1]];
//			ASF_vertex vedge = DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), 0, bForward);
//			curidx = num_rows + (Fr[row - 1] + co);
//			vedge.fy[1] = vertex1.fy[1];
//
//			if (vedge.getOldRange() != vertex1.getOldRange())
//				printf("");
//			vedge.fx[1] = vertex1.fx[1];
//			vedge.fcxy[1] = vertex1.fcxy[1];
//			vertex1.fx[1] = curidx;
//
//			m[curidx] = vedge;
//			vertex1.unsetInFace_xy();
//			co++;
//		}
//
//		if (curidx == 65549)
//			printf("");
//
//		if (vertex1.isInNextLevel_yz())
//		{
//			ASF_vertex vertex2 = m[vertex1.fy[1]];
//			ASF_vertex vedge = DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), 0, bForward);
//			if (vedge.getOldRange() != vertex1.getOldRange())
//				printf("");
//			curidx = num_rows + (Fr[row - 1] + co);
//			vedge.fx[1] = vertex1.fx[1];
//
//			vedge.fy[1] = vertex1.fy[1];
//			vedge.fcxy[1] = vertex1.fcxy[1];
//
//			vertex1.fy[1] = curidx;
//			m[curidx] = vedge;
//			vertex1.unsetInFace_yz();
//			co++;
//
//		}
//
//
//		if (vertex1.isInNextLevel_xz())
//		{
//			ASF_vertex vertex2 = m[vertex1.fcxy[1]];
//			ASF_vertex vedge = DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), 0, bForward);
//			curidx = num_rows + (Fr[row - 1] + co);
//			vedge.fcxy[1] = vertex1.fcxy[1];
//
//			if (vedge.getOldRange() != vertex1.getOldRange())
//				printf("");
//			vedge.fy[1] = vertex1.fy[1];
//			vedge.fx[1] = vertex1.fx[1];
//			vertex1.fcxy[1] = curidx;
//			m[curidx] = vedge;
//			vertex1.unsetInFace_xz();
//			co++;
//
//		}
//
//
//		////m[row] = vertex1;
//		//if ((Fr[row] - Fr[row - 1]) < 0)
//		//	printf("");
//		//else
//		m[row] = vertex1;
//	}
//
//}



//=====================================================================================================
__global__ void CheckNeighborhood_c3(fEdge* eg, ASF_vertex*m, uint32_t* Fr, Dimension* d, Boundary* b, Point* step, int i, bool bForward, uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;


	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	//	for (uint32_t row = 0; row < num_rows; row++)

	{

		if (row >= num_rows)
			return;

		Fr[row] = 0;


		int co = 0;
		fEdge edge = eg[row];
		// if (row == 0)


		if (!edge.isInBoundary())
			return;
//		edge.bsplit = false;
		//  printf("%d--", row);
		ASF_vertex vertex1 = m[edge.v1];
		ASF_vertex vertex2 = m[edge.v2];
		if (!vertex1.isInBoundary() || !vertex2.isInBoundary())
		{
			edge.unsetInBoundary();
			eg[row] = edge;

			return;

		}
		/*if (abs(vertex1.p.x - vertex2.p.x) > step->x + exp(-6.0) || abs(vertex1.p.y - vertex2.p.y) > step->y + exp(-6.0) || abs(vertex1.p.z - vertex2.p.z) > step->z + exp(-6.0))
		{

		continue;
		}*/

		if (abs((int)vertex1.getOldRange() - (int)vertex2.getOldRange()) > 1 && abs((int)vertex1.getOldRange() - (int)vertex2.getOldRange()) != d->x && abs((int)vertex1.getOldRange() - (int)vertex2.getOldRange()) != (d->x*d->y))
			printf("");


		float dist = sqrt((vertex1.e.x - vertex2.e.x)*(vertex1.e.x - vertex2.e.x) + (vertex1.e.y - vertex2.e.y)*(vertex1.e.y - vertex2.e.y) + (vertex1.e.z - vertex2.e.z)*(vertex1.e.z - vertex2.e.z));

		//if (abs(vertex1.p.x - vertex2.p.x)  > step->x + exp(-6.0) || abs(vertex1.p.y - vertex2.p.y)  > step->y + exp(-6.0) || abs(vertex1.p.z - vertex2.p.z)  > step->z + exp(-6.0))
		//	printf(" %d-- ", row);
		//// printf("");

		//if (dist > 2 * step->x && (row == 98304 || row == 13778 || row == 98328 || row == 98327))
		//	printf(" %d-- ", row);
		// printf("");
		// printf(" %d-- ", row);

		if (!checkEdge(vertex1, vertex2, d, bForward))
		{
			/*if (row == 98312)
			printf("\n v1 = %d,%d --- range = %d,%d \n",edge.v1,edge.v2, vertex1.getRange(), vertex2.getRange());*/
			Fr[row] = 1;
			/*if (i == 1)
			printf(" %d-- ", row);*/
		}

		eg[row] = edge;



	}





	//if (row > num_rows)
	//	return;
	//fEdge edge = eg[row];
	//Fr[row] = 0;
	//if (!eg[row].isInBoundary())
	//	return;
	//	ASF_vertex vertex1 = m[edge.v1];
	//	ASF_vertex vertex2 = m[edge.v2];
	//	if (!vertex1.isInBoundary() || !vertex2.isInBoundary())
	//	{
	//		edge.unsetInBoundary();
	//		eg[row] = edge;

	//		return;

	//	}
	//	//printf("%d,%d -- ", d->x, d->y);
	//	if (!checkEdge(vertex1, vertex2, d, bForward))
	//	{

	//		Fr[row] = 1;
	//		//printf(" %d-- ", row);
	//	}
	//	//Fr[row] = 10;

	//	//printf("%d -- ", row);

}


//__global__ void CheckNeighborhood_c3(ASF_vertex*m, fEdge* eg, uint32_t* Fr, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t num_rows)
//{
//
//
//	 uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
////	for (uint32_t row = 0; row < num_rows; row++)
//	
//	{
//
//		 if (row >= num_rows)
//			 return;
//
//		 Fr[row] = 0;
//
//
//		 int co = 0;
//		 fEdge edge = eg[row];
//		// if (row == 0)
//		 if (row>32768)
//		 printf("%d--", row);
//
//		 if (!edge.isInBoundary())
//			 return;
//		 edge.bsplit = false;
//		//  printf("%d--", row);
//		 ASF_vertex vertex1 = m[edge.v1];
//		 ASF_vertex vertex2 = m[edge.v2];
//		 if (!vertex1.isInBoundary() || !vertex2.isInBoundary())
//		 {
//			 edge.unsetInBoundary();
//			 eg[row] = edge;
//
//			 return;
//
//		 }
//		 /*if (abs(vertex1.p.x - vertex2.p.x) > step->x + exp(-6.0) || abs(vertex1.p.y - vertex2.p.y) > step->y + exp(-6.0) || abs(vertex1.p.z - vertex2.p.z) > step->z + exp(-6.0))
//		 {
//
//		 continue;
//		 }*/
//
//		 if (abs((int)vertex1.getOldRange() - (int)vertex2.getOldRange()) > 1 && abs((int)vertex1.getOldRange() - (int)vertex2.getOldRange()) != d->x && abs((int)vertex1.getOldRange() - (int)vertex2.getOldRange()) != (d->x*d->y))
//			 printf("");
//		
//		 
//		 float dist = sqrt((vertex1.e.x - vertex2.e.x)*(vertex1.e.x - vertex2.e.x) + (vertex1.e.y - vertex2.e.y)*(vertex1.e.y - vertex2.e.y) + (vertex1.e.z - vertex2.e.z)*(vertex1.e.z - vertex2.e.z));
//
//		 if (abs(vertex1.p.x - vertex2.p.x)  > step->x + exp(-6.0) || abs(vertex1.p.y - vertex2.p.y)  > step->y + exp(-6.0) || abs(vertex1.p.z - vertex2.p.z)  > step->z + exp(-6.0))
//			 printf(" %d-- ", row);
//		 // printf("");
//
//		 if (dist > 2 * step->x && (row == 98304 || row == 13778 || row == 98328 || row == 98327))
//			 printf(" %d-- ", row);
//		 // printf("");
//		// printf(" %d-- ", row);
//
//		 if (!checkEdge(vertex1, vertex2, d, bForward))
//		 {
//
//			 Fr[row] = 1;
//			 printf(" %d-- ", row);
//		 }
//
//		 eg[row] = edge;
//
//
//
//	}
//	
//}


__global__ void EdgeReduction(fEdge*eg, ASF_vertex*m, fFace* Ff, uint32_t* Fe_Face, uint32_t* Fe_Edge, uint32_t num_edges)
{


	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	{

		if (row >= num_edges)
			return;

		if (Fe_Edge[row] == 0)
		{

			fEdge edge = eg[row];
			if (!edge.isInBoundary())
				return;
			//face.bSplit = false;
			for (int i = 0; i < 4; i++)
			if (Fe_Face[edge.E2F[i]] == 1 && edge.level <= Ff[edge.E2F[i]].level)// && edge.length >= Ff[edge.E2F[i]].dx2)
			{
				if (row > 98304)
					printf("");
				Fe_Edge[row] = 1;
				break;
			}
		}

	}

}


__global__ void CheckFace_c3(fFace* fc, ASF_vertex*m, fEdge* eg, uint32_t*Fe_Edge, uint32_t* Fe_Face, uint32_t num_face)
{


	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	//	for (uint32_t row = 0; row < num_rows; row++)

	{
		if (row >= num_face)
			return;
		if (row >= num_face)
			return;
		if (row == 98317)
			printf("");


		fFace face = fc[row];
//		face.bSplit = false;





		//}

		if (!face.isInBoundary())
			return;
		if (!eg[face.edge[0]].isInBoundary() || !eg[face.edge[1]].isInBoundary() || !eg[face.edge[2]].isInBoundary() || !eg[face.edge[3]].isInBoundary())
			return;
		if (Fe_Edge[face.edge[0]] == 1 || Fe_Edge[face.edge[1]] == 1 || Fe_Edge[face.edge[2]] == 1 || Fe_Edge[face.edge[3]] == 1)
		{
			if (!m[eg[face.edge[0]].v1].isInBoundary() || !m[eg[face.edge[0]].v2].isInBoundary() || !m[eg[face.edge[1]].v2].isInBoundary() || !m[eg[face.edge[3]].v2].isInBoundary())

				return;



			Fe_Face[row] = 1;
			fc[row].level = fc[row].level + 1;
			//face.bSplit = true;
		}
		fc[row] = face;

	}




	return;

}




__global__ void SplitEdge(fEdge* eg, ASF_vertex*m, fFace* fc, uint32_t* Fe_Edge, uint32_t* Fr_Edge, Dimension* d, Boundary* b, Point* step, bool bForward, uint32_t num_vertex, uint32_t num_edges)
{


	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	//	for (uint32_t row = 0; row < num_rows; row++)

	{

		if (row > num_edges)
			return;

		if (Fe_Edge[row] == 1)
		{
			int curVertexId = num_vertex + Fr_Edge[row];
			int curEdgeId = num_edges + Fr_Edge[row];

			int co = 0;



			fEdge edge1 = eg[row];
			fEdge edge2 = eg[row];
			ASF_vertex vertex1 = m[edge1.v1];
			ASF_vertex vertex2 = m[edge1.v2];


			if ((vertex1.p.x - vertex2.p.x) > step->x || (vertex1.p.y - vertex2.p.y) > step->y || (vertex1.p.z - vertex2.p.z) > step->z)
				printf("");
			ASF_vertex* vedge = DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), d, step, b, 0, bForward);

			//return;;

	//		edge1.length = edge1.length / 2;
			edge1.v2 = curVertexId;
			edge2.v1 = curVertexId;
//			edge1.bsplit = true;
			edge1.next = curEdgeId;
			edge1.level = edge1.level * 2;
			edge2.level = edge2.level * 2;
			edge2.Prev = row;
			vedge->level = edge2.level;

			float rgb[3];
			rgb[0] = 0.;
			rgb[1] = 0.;
			rgb[2] = 0.;

			//	display_voxel(vertex1.getOldRange(), rgb, d->x, d->y, d->z);

			eg[row] = edge1;
			eg[curEdgeId] = edge2;;
			m[curVertexId] = *vedge;
			//printf("%d,%d,%d range  = %d,%d \n", edge1.v1, edge1.v2, edge2.v2, vertex1.getRange(), vertex2.getRange());



			/*	if (CheckEdgeCondition(m, edge1) == false)
			printf("");*/
			/*if (m[edge1.v1].getOldRange() == 1551)
			drawLine(edge1.v1, edge1.v2, rgb);*/
			rgb[0] = 1.;
			rgb[1] = 0.;
			rgb[2] = 0.;
			/*	if (m[edge2.v1].getOldRange() == 1551)
			drawLine(edge2.v1, edge2.v2, rgb);*/

		}

	}
}

#define BLOCKSIZE 512

template<class T>
__global__ void kernelFunction1(T * __restrict__ d_data, const unsigned int NperGPU) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	ASF_vertex vertex = d_data[tid];
	if (tid < NperGPU)
	{
		uint32_t rng = vertex.range;
		for (int k = 0; k < 1000; k++) vertex.range = 2 * rng;// *3.0 *d_data[tid];
		d_data[tid] = vertex;
	}


}

/*******************/
/* KERNEL FUNCTION */
/*******************/
template<class T>
__global__ void Initialize_(T * __restrict__ m, fEdge* eg, fFace* fr, Boundary* b, Dimension* d, Point* step, bool bForward, const unsigned int num_rows)
{

	const int row = threadIdx.x + blockIdx.x * blockDim.x;

	if (row >= num_rows)
		return;
	//printf(" %d --", row);
	ASF_vertex vertex = m[row];
	int xdim = d->x;
	int ydim = d->y;
	int zdim = d->z;

	//if (row + d->x < num_rows /*&& row + d->x*d->y < num_rows*/)
	{

		/*if ((row % d->x - 1)==0 || (row % ((d->x*d->y) - 1))==0)
		continue;*/

		vertex.setInBoundary();

		int curEdgeidx = row * 3;


		/*vertex.fxy = curEdgeidx;
		vertex.fyz = curEdgeidx + 1;
		vertex.fxz = curEdgeidx + 2;*/
		m[row] = vertex;
		int iz = row / (d->x*d->y);
		int iy = (row - iz*(d->x*d->y)) / (d->x);
		int ix = (row - iz*(d->x*d->y)) % (d->x);
		/*if (iz == d->z - 1 || iy == d->y - 1 || ix == d->x - 1)
		{
		eg[curEdgeidx].unsetInBoundary();
		eg[curEdgeidx + 1].unsetInBoundary();
		eg[curEdgeidx + 2].unsetInBoundary();
		fr[curEdgeidx].unsetInBoundary();
		fr[curEdgeidx + 1].unsetInBoundary();
		fr[curEdgeidx + 2].unsetInBoundary();
		continue;
		}*/

		if (curEdgeidx == 2979)
			printf("");

		if (row + 1 < num_rows && ix < d->x - 1)
		{

			ASF_vertex vertex2 = m[row + 1];
			if (abs(vertex.p.x - vertex2.p.x) > step->x + exp(-6.0) || abs(vertex.p.y - vertex2.p.y) > step->y + exp(-6.0) || abs(vertex.p.z - vertex2.p.z) > step->z + exp(-6.0))
				eg[curEdgeidx].unsetInBoundary();
			else
			{
				eg[curEdgeidx].v1 = row;
				eg[curEdgeidx].v2 = row + 1;
//				eg[curEdgeidx].bsplit = false;
				eg[curEdgeidx].E2V = row;
				eg[curEdgeidx].E2F[0] = curEdgeidx;
				eg[curEdgeidx].E2F[1] = curEdgeidx + 1;
		//		eg[curEdgeidx].length = step->x;;
				eg[curEdgeidx].level = 1;
				eg[curEdgeidx].Prev = 0;
				eg[curEdgeidx].setInBoundary();
			}



			if (curEdgeidx >= d->x * 3)
			{
				eg[curEdgeidx].E2F[2] = (row - d->x) * 3;

			}
			else
				eg[curEdgeidx].E2F[2] = OUTBOUNDARY;

			if (row >= d->x*d->y)
				eg[curEdgeidx].E2F[3] = (row - d->x*d->y) * 3 + 1;
			else
				eg[curEdgeidx].E2F[3] = OUTBOUNDARY;

		}


		//eg[curEdgeidx ].E2T[0] = (row ) * 4;

		/*if (row > d->x)
		eg[curEdgeidx ].E2T[1] = (row - d->x) * 4 + 2;
		else
		eg[curEdgeidx ].E2T[1] = 0;*/
		if (row + d->x < num_rows && iy < d->y - 1)
		{

			ASF_vertex vertex2 = m[row + d->x];
			if (abs(vertex.p.x - vertex2.p.x) > step->x + exp(-6.0) || abs(vertex.p.y - vertex2.p.y) > step->y + exp(-6.0) || abs(vertex.p.z - vertex2.p.z) > step->z + exp(-6.0))
				eg[curEdgeidx + 1].unsetInBoundary();
			else
			{
				eg[curEdgeidx + 1].v1 = row;
				eg[curEdgeidx + 1].v2 = row + d->x;
//				eg[curEdgeidx + 1].bsplit = false;
				eg[curEdgeidx + 1].E2V = row;

				eg[curEdgeidx + 1].E2F[0] = curEdgeidx;
				eg[curEdgeidx + 1].E2F[1] = curEdgeidx + 2;
//				eg[curEdgeidx + 1].length = step->y;;
				eg[curEdgeidx + 1].level = 1;
				eg[curEdgeidx + 1].Prev = 0;
				eg[curEdgeidx + 1].setInBoundary();





				if (curEdgeidx >= 3)
				{
					eg[curEdgeidx + 1].E2F[2] = (row - 1) * 3;

				}

				else
					eg[curEdgeidx + 1].E2F[2] = OUTBOUNDARY;

				if (curEdgeidx >= d->x*d->y * 3)
					eg[curEdgeidx + 1].E2F[3] = (row - d->x*d->y) * 3 + 2;
				else
					eg[curEdgeidx + 1].E2F[3] = OUTBOUNDARY;
			}
		}



		if (row + d->x*d->y < num_rows && iz < d->z - 1)
		{

			ASF_vertex vertex2 = m[row + d->x*d->y];
			if (abs(vertex.p.x - vertex2.p.x) > step->x + exp(-6.0) || abs(vertex.p.y - vertex2.p.y) > step->y + exp(-6.0) || abs(vertex.p.z - vertex2.p.z) > step->z + exp(-6.0))
				eg[curEdgeidx + 2].unsetInBoundary();
			else
			{

				eg[curEdgeidx + 2].v1 = row;
				eg[curEdgeidx + 2].v2 = row + (d->x*d->y);
//				eg[curEdgeidx + 2].bsplit = false;
				eg[curEdgeidx + 2].E2V = curEdgeidx;

				eg[curEdgeidx + 2].E2F[0] = curEdgeidx + 1;
				eg[curEdgeidx + 2].E2F[1] = curEdgeidx + 2;

//				eg[curEdgeidx + 2].length = step->z;;
				eg[curEdgeidx + 2].level = 1;
				eg[curEdgeidx + 2].Prev = 0;
				eg[curEdgeidx + 2].setInBoundary();


				if (curEdgeidx >= 3 && row%d->x != 0)
					eg[curEdgeidx + 2].E2F[2] = (row - 1) * 3 + 1;
				else
					eg[curEdgeidx + 2].E2F[2] = OUTBOUNDARY;


				if (row >= d->x)
				{
					eg[curEdgeidx + 2].E2F[3] = (row - d->x) * 3 + 2;

				}
				else
					eg[curEdgeidx + 2].E2F[3] = OUTBOUNDARY;
			}

		}
		//eg[curEdgeidx + 1].E2T[0] = (row)* 4 +3;

		/*if (row > 1)
		eg[curEdgeidx + 1].E2T[1] = (row - 1) * 4 +1;
		else
		eg[curEdgeidx + 1].E2T[1] = 0;*/



		if (eg[curEdgeidx].isInBoundary() && eg[curEdgeidx + 1].isInBoundary())
		{


			if ((row + 1) % d->x != 0 && row + (d->x) < num_rows)
			{
				fr[curEdgeidx].edge[0] = curEdgeidx;
				fr[curEdgeidx].edge[1] = (row + 1) * 3 + 1;
				fr[curEdgeidx].edge[2] = (row + d->x) * 3;
				fr[curEdgeidx].edge[3] = curEdgeidx + 1;
				fr[curEdgeidx].F2V = row;
				fr[curEdgeidx].cornerId = row + d->x + 1;
//				fr[curEdgeidx].bSplit = false;
//				fr[curEdgeidx].dx1 = step->x;
//				fr[curEdgeidx].dx2 = step->y;
				fr[curEdgeidx].setInBoundary();
				fr[curEdgeidx].level = 1;;

				if (row + (d->x*d->y) < num_rows)
				{
					fr[curEdgeidx + 1].edge[0] = curEdgeidx;
					fr[curEdgeidx + 1].edge[1] = (row + 1) * 3 + 2;
					fr[curEdgeidx + 1].edge[2] = (row + (d->x*d->y)) * 3;
					fr[curEdgeidx + 1].edge[3] = curEdgeidx + 2;
					fr[curEdgeidx + 1].cornerId = row + (d->x*d->y) + 1;
					fr[curEdgeidx + 1].F2V = row;
//					fr[curEdgeidx + 1].bSplit = false;
//					fr[curEdgeidx + 1].dx1 = step->x;
//					fr[curEdgeidx + 1].dx2 = step->z;
					fr[curEdgeidx + 1].setInBoundary();
					fr[curEdgeidx + 1].level = 1;;


				}
				else
				{
					fr[curEdgeidx + 1].unsetInBoundary();

				}
			}

		}
		else
		{
			fr[curEdgeidx].unsetInBoundary();
			fr[curEdgeidx + 1].unsetInBoundary();

		}


		if (eg[curEdgeidx + 2].isInBoundary() && eg[curEdgeidx + 1].isInBoundary() && (row == 0 || ((row) % d->x*d->y != 0 && row + (d->x*d->y) < num_rows)))
		{
			fr[curEdgeidx + 2].edge[0] = curEdgeidx + 1;
			fr[curEdgeidx + 2].edge[1] = (row + (d->x)) * 3 + 2;
			fr[curEdgeidx + 2].edge[2] = (row + (d->x*d->y)) * 3 + 1;
			fr[curEdgeidx + 2].edge[3] = curEdgeidx + 2;
			fr[curEdgeidx + 2].F2V = row;
			fr[curEdgeidx + 2].cornerId = row + (d->x*(d->y + 1));

//			fr[curEdgeidx + 2].bSplit = false;
//			fr[curEdgeidx + 2].dx1 = step->y;
//			fr[curEdgeidx + 2].dx2 = step->z;
			fr[curEdgeidx + 2].setInBoundary();
			fr[curEdgeidx + 2].level = 1;;


		}
		else
		{
			fr[curEdgeidx + 2].unsetInBoundary();
		}

		m[row] = vertex;
		/*if (curEdgeidx > num_rows && curEdgeidx < num_rows + 10)
		printf("%d,%d --", eg[curEdgeidx].v1, eg[curEdgeidx].v2);*/


	}





	//if (tid < NperGPU) for (int k = 0; k < 1000; k++) d_data[tid].range =2;// d_data[tid] * 3.0 *d_data[tid];

}
/******************/
/* PLAN STRUCTURE */
/******************/
// --- Async
template<class T>
struct plan {
	T               *d_data;
	Boundary* b;
	Dimension*d;
	Point* s;
	Point* v;
	fEdge* eg;
	fFace* fc;
	uint32_t* Fe_Edge;
	uint32_t* Fr_Edge;
	uint32_t* Fe_Face;
	uint32_t* Fr_Face;
	//Stream for asynchronous command execution
	hipStream_t stream;
};

#define sample_seeds 50
/*********************/
/* SVD PLAN CREATION */
/*********************/
template<class T>
void createPlan(plan<T>& plan, unsigned int NperGPU, unsigned int NEdgeperGPU, unsigned int N, unsigned int gpuID) {

	// --- Device allocation
	(hipSetDevice(gpuID));
	checkCudaErrors(hipStreamCreate(&plan.stream));
	checkCudaErrors(hipMalloc(&(plan.d_data), NperGPU * sample_seeds* sizeof(T)));
	checkCudaErrors(hipMalloc(&(plan.v), N * sizeof(Point)));
	checkCudaErrors(hipMalloc(&(plan.Fe_Edge), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
	checkCudaErrors(hipMalloc(&(plan.Fr_Edge), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));

	checkCudaErrors(hipMalloc(&(plan.Fe_Face), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
	checkCudaErrors(hipMalloc(&(plan.Fr_Face), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));

	checkCudaErrors(hipMalloc(&(plan.eg), NEdgeperGPU * sample_seeds * sizeof(fEdge)));
	checkCudaErrors(hipMalloc(&(plan.fc), NEdgeperGPU * sample_seeds * sizeof(fFace)));
	checkCudaErrors(hipMalloc(&(plan.b), 1 * sizeof(Boundary)));
	checkCudaErrors(hipMalloc(&(plan.d), 1 * sizeof(Dimension)));
	checkCudaErrors(hipMalloc(&(plan.s), 1 * sizeof(Point)));

}

template<class T>
__global__ void kernelFunction(T * __restrict__ d_data, int i, const unsigned int NperGPU) {

	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < NperGPU) /*for (int k = 0; k < 10; k++)*/
	{
		ASF_vertex d = d_data[tid];


	}

}

//template<class T>
//__global__ void Tracing(T * __restrict__ d_data, Point*v, Boundary* b, Dimension* _d, Point* step, bool bForward,int i, uint32_t NperGPU) {
//
//	const int tid = threadIdx.x + blockIdx.x * blockDim.x;
//
//	/*if (tid == 1)
//		printf(" num rows = %d  ", i);
//*/
//	if (tid < NperGPU) /*for (int k = 0; k < 10; k++)*/
//	{
//	
//
//		ASF_vertex vertex = d_data[tid];
//
//		if (!vertex.isInBoundary())
//		{
//			return;
//
//		}
//
//		/*if (i == 0)
//			printf("%f,%f \n",v[32000].x, v[32000].y);
//		
//		d_data[tid].e.x =  v[vertex.getOldRange()].x;
//		d_data[tid].e.y = vertex.e.y + 0.1;
//		d_data[tid].e.z = vertex.e.z + 0.1;
//
//		return;*/
//		Point e = vertex.e;
//		Point _v = trilinearInterpolation(e, v, vertex.getOldRange(), _d, b, step, bForward);
//		/*vertex.e.x = b->high.x;
//		vertex.e.y = b->high.y;
//		vertex.e.z = b->high.z;
//		if (row < 10)
//		printf("%f,%f \n", _v.x, _v.y);*/
//
//		float dist = _v.getDist();
//		_v.x = (_v.x / (dist*4.0));
//		_v.y = (_v.y / (dist*4.0));
//		_v.z = (_v.z / (dist*4.0));
//
//		_v.x = _v.x*step->x;
//		_v.y = _v.y*step->y;
//		_v.z = _v.z*step->z;
//
//		if (bForward)
//		{
//			vertex.e.x = vertex.e.x + _v.x;
//			vertex.e.y = vertex.e.y + _v.y;
//			vertex.e.z = vertex.e.z + _v.z;
//
//		}
//		else
//		{
//			vertex.eb.x = vertex.eb.x - _v.x;
//			vertex.eb.y = vertex.eb.y - _v.y;
//			vertex.eb.z = vertex.eb.z - _v.z;
//		}
//		if (tid == 4591 )
//			printf(" \n i = %d, %d, %f,%f,%f \n",i,tid, vertex.e.x, vertex.e.y, vertex.e.z);
//	
//		
//
//		bool xy = false;
//		bool yz = false;
//		bool xz = false;
//
//
//		if (bForward)
//		{
//			if (vertex.checkInBoundary(b))
//			{
//				uint32_t range = vertex.getRange(b, step, _d);
//			/*	if (tid == 4591 || tid == 4592)
//					printf("%d raaaaaaaaaaaaangeeeeeeeeeeeee\n", range);*/
//				/*if (vertex.isInNextLevel_yz())
//					yz = true;
//				if (vertex.isInNextLevel_xy())
//					xy = true;
//				if (vertex.isInNextLevel_xz())
//					xz = true;*/
//
//				if (bForward)
//					vertex.setRange(range);
//
//
//				/*if (xy)
//					vertex.setInNextLevel_xy();
//				if (xz)
//					vertex.setInNextLevel_xz();
//				if (yz)
//					vertex.setInNextLevel_yz();*/
//
//			}
//			else
//			{
//				vertex.unsetInBoundary();
//				vertex.unsetInFace_xy();
//				vertex.unsetInFace_yz();
//				vertex.unsetInFace_xz();
//			}
//		}
//		else
//		{
//			if (vertex.checkInBoundaryBackward(b))
//			{
//				uint32_t range = vertex.getRangeBackward(b, step, _d);
//				vertex.setRangeBackward(range);
//
//
//				/*if (xy)
//					vertex.setInNextLevel_xy();
//				if (xz)
//					vertex.setInNextLevel_xz();
//				if (yz)
//					vertex.setInNextLevel_yz();*/
//
//			}
//			else
//			{
//				vertex.unsetInBoundary();
//			/*	vertex.unsetInFace_xy();
//				vertex.unsetInFace_yz();
//				vertex.unsetInFace_xz();*/
//			}
//		}
//		
//		d_data[tid] = vertex;
//	}
//
//}


int isPowerOfTwo(unsigned int x, unsigned int& ipower)
{
	unsigned int powerOfTwo = 1;
	ipower = 0;
	while (powerOfTwo < x && powerOfTwo < 2147483648)
	{
		powerOfTwo *= 2;
		ipower++;

	}
	return (x == powerOfTwo);
}


void scanExclusiveHost(
	uint *dst,
	uint *src,
	uint batchSize,
	uint arrayLength
	)
{
	//for (uint i = 0; i < batchSize; i++, src += arrayLength, dst += arrayLength)
	{
		dst[0] = 0;

		for (uint j = 1; j < arrayLength; j++)
			dst[j] = src[j - 1] + dst[j - 1];
	}
}



__global__ void Tracing(ASF_vertex * m, Point*v, Boundary* b, Dimension* d, Point* step, bool bForward, uint32_t i, uint32_t num_rows)

{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > num_rows)
		return;

	ASF_vertex vertex = m[row];

	if (!vertex.isInBoundary())
	{
		return;

	}

	Point e = vertex.e;
	//if (!bForward)
	//	e = vertex.eb;
	float vx, vy, vz;
	trilinearInterpolation(m, e, v, vertex.getOldRange(), d, b, step, vx, vy, vz, bForward);

	float dist = sqrt(vx*vx + vy*vy + vz*vz);

	vx = (vx / (dist*4.0))*step->x;
	vy = (vy / (dist*4.0))*step->y;
	vz = (vz / (dist*4.0))*step->z;
	/*_v = _v / 4.0;
	_v *= *step;*/
	//glColor3f(rgb[0], rgb[1], rgb[2]);    
	//	Point e = vertex.e;
//	if (bForward)
	{
		vertex.e.x += vx;
		vertex.e.y += vy;
		vertex.e.z += vz;
	}
	//vertex.e += _v;
	//else
	//{
	//	vertex.eb.x -= vx;
	//	vertex.eb.y -= vy;
	//	vertex.eb.z -= vz;
	//}
	//vertex.eb -= _v;
	/*if (row == 4591)
	printf("%f,%f,%f  \n", vertex.e.x, vertex.e.y, vertex.e.z);*/
	float ep[3];
	//generalstreamlineTracing_single(p1, bForward, ep, false);





	bool xy = false;
	bool yz = false;
	bool xz = false;
//	if (bForward)
	{
		if (vertex.checkInBoundary(b))
		{
			uint32_t range = vertex.getRange(b, step, d);
			if (vertex.isInNextLevel_yz())
				yz = true;
			if (vertex.isInNextLevel_xy())xy = true;
			if (vertex.isInNextLevel_xz())xz = true;
			vertex.setRange(range);
			if (xy)	vertex.setInNextLevel_xy();
			if (xz)vertex.setInNextLevel_xz();
			if (yz)vertex.setInNextLevel_yz();

		}
		else
		{
			vertex.unsetInBoundary();
		}
	}
	/*else
	{
		if (vertex.checkInBoundaryBackward(b))
		{
			if (vertex.isInNextLevel_yz())
				yz = true;
			if (vertex.isInNextLevel_xy())xy = true;
			if (vertex.isInNextLevel_xz())xz = true;
			uint32_t range = vertex.getRangeBackward(b, step, d);
			vertex.setRangeBackward(range);
			if (xy)	vertex.setInNextLevel_xy();
			if (xz)vertex.setInNextLevel_xz();
			if (yz)vertex.setInNextLevel_yz();

		}
		else
		{
			vertex.unsetInBoundary();
		}
	}
*/



	m[row] = vertex;


	return;
}



