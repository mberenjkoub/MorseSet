#include "hip/hip_runtime.h"


#include <cstdio>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_timer.h>
#include<stack>
#include<set>
#include<map>
#include<queue>
#include<math.h>

#include "SCC.h"
#include "scc_kernel.h"
//#include "graph_generator.h"
//#include "parallel_fwd.h"
//#include "hash_table.h"

#ifdef _DEBUG
void bbin_printf(uint32_t elem, int N = 32, int end = 0)
{
	for ( int i = N - 1; i >= end; i-- )
		printf("%d", (bool)(elem & ((uint32_t)1 << i)));
}
#endif

bool _DeviceSet;
//
__global__ void checkTerminateAndSetOldMap(COL_vertex * m, const uint32_t num_rows, uint32_t * propagate)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (row >= num_rows)
		return;
	COL_vertex vertex = m[row];
	if (vertex.isPropagate())
		return;

	*propagate = 1;
	vertex.setOldMap(vertex.getMap());
	vertex.setMap(0);

	m[row] = vertex;
}
//
//
__global__ void COL_FWD(const Edge * Fc, const uint32_t * Fr, COL_vertex * m, const uint32_t num_rows, uint32_t * propagate)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x + 1; // + 1 we dont work with undefined vertex
	if (row >= num_rows)
		return;
	COL_vertex vertex = m[row];
	if (vertex.isPropagate() || !vertex.isBackwardVisited())
		return;

	uint32_t row_begin = Fr[row];
	uint32_t row_end = Fr[row + 1];

	int prop = 0;
	for (uint32_t column = row_begin; column < row_end; column++) {
		uint32_t index = Fc[column].getValue();
		COL_vertex p_vertex = m[index];

		if (p_vertex.isBackwardVisited() || p_vertex.getMap() != vertex.getMap())
			continue;
		p_vertex.setBackwardVisitedBit();
		m[index] = p_vertex;
		prop = 1;
	}
	if (prop)
		*propagate = prop;
	vertex.setPropagateBit();
	m[row] = vertex;
}
//
//
__global__ void OBFcomputeSCCs(const OBF_vertex * m, uint32_t * Fr, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows || row == 0)
		return;

	OBF_vertex vertex = m[row];

	Fr[vertex.getOldRange() - 1] = 1;
}

 void OBFcomputeSCCs_c(const OBF_vertex * m,ASF_vertex*a,Dimension*d,  uint32_t * Fr, const uint32_t num_rows)
{
	 uint32_t oldrange[100];
	 uint32_t oldrangeCounter[100];

	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	 int cc = 1;
	 uint32_t range = 0;
	 for (uint32_t row = 0; row < num_rows; row++)
	 {
		 if (row >= num_rows || row == 0)
			 continue;
		 int  i = 1;
		 OBF_vertex vertex = m[row];


		 if (vertex.getOldRange() != row)
		 {
			 for ( i = 1; i < cc; i++)
			 {

				 if (vertex.getOldRange() == oldrange[i])
				 {
					 a[row].setInSCC();
					 range = a[row].getOldRange();
			/*		 a[row].Fr[0] = row + 1;
					 a[row].Fr[1] = row + d->x + 1;
					 a[row].Fr[2] = row + d->x;*/

					 oldrangeCounter[i]++;


					 break;
					 printf(" %d,%d ", row, vertex.getOldRange());

				 }
			 }
			 if (i == cc)
			 {
				 oldrange[cc] = vertex.getOldRange();
				 cc++;
			 }




		 }
		 if(vertex.getOldRange() > 0)
		 Fr[vertex.getOldRange() - 1] = 1;
	 }


}
//
//
__global__ void COLcomputeSCCs(const COL_vertex * m, uint32_t * Fr, const uint32_t num_rows, const ordering o)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (row >= num_rows)
		return;

	COL_vertex vertex = m[row];
	uint32_t temp = MAX_VERTEX;
	temp -= vertex.getMap() + 1;
	if (temp < num_rows)
		Fr[temp] = 1;

	//	Fr[ ( o == MAX ) ? (vertex.getMap() - 1) : ((MAX_VERTEX - vertex.getMap()) - 1) ] = 1;
	//	Fr[ 0 ] = 1;
	//	Fr[ (MAX_VERTEX - vertex.getMap()) - 1 ] = 1;
}
//
//
__global__ void setBWDseed(COL_vertex * m, const uint32_t num_rows, const ordering o)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x + 1;
	if (row >= num_rows)
		return;
	COL_vertex vertex = m[row];
	if (vertex.isPropagate())
		return;

	if (o == MAX) {
		if (row == vertex.getMap())
			vertex.setBackwardVisitedBit();
	}
	else {
		if ((MAX_VERTEX - row) == vertex.getMap())
			vertex.setBackwardVisitedBit();
	}
	m[row] = vertex;
}
//
//
__global__ void p_COL_MAP(const Edge * Fc, const uint32_t * Fr, COL_vertex * m, const uint32_t num_rows, const ordering o,
	uint32_t * propagate)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x + 1; // + 1 we dont work with undefined vertex
	if (row >= num_rows)
		return;
	COL_vertex vertex = m[row];
	if (vertex.isPropagate())
		return;

	uint32_t row_begin = Fr[row];
	uint32_t row_end = Fr[row + 1];
	uint32_t max_candidate;

	if (o == MAX)
		max_candidate = row;
	else
		max_candidate = MAX_VERTEX - row;

	for (uint32_t column = row_begin; column < row_end; column++) {
		uint32_t index = Fc[column].getValue();
		COL_vertex p_vertex = m[index];
		if (!p_vertex.isPropagate() && p_vertex.getOldMap() == vertex.getOldMap())
			max_candidate = max(max_candidate, max(vertex.getMap(), p_vertex.getMap()));
	}

	if (max_candidate > vertex.getMap()) {
		vertex.setMap(max_candidate);
		*propagate = 1;
	}
	m[row] = vertex;
}
//
__global__ void OBFtoCOLKernel(const OBF_vertex * m, COL_vertex * cm, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;

	cm[row] = m[row];
}
//
__global__ void SetCOL(OBF_vertex * m, const uint32_t * OldRange, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;

	OBF_vertex vertex = m[row];
	if (vertex.getOldRange() == *OldRange && !vertex.isInSCC()) {
		vertex.setInCOL();
		m[row] = vertex;
	}
}

//
//
// void OBFCOLcompute_c(const OBF_vertex * m, uint32_t * out_field, const uint32_t * OldRange, const uint32_t num_rows)
//{
//	 for (uint32_t row = 0; row <= num_rows; row++)
//	 {
//		 //uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//		 if (row >= num_rows)
//			 return;
//
//		 OBF_vertex vertex = m[row];
//		 if (vertex.getOldRange() == *OldRange && !vertex.isInSCC())
//			 out_field[row - 1] = 1;
//	 }
//}
//
//
__global__ void OBFCOLcompute(const OBF_vertex * m, uint32_t * out_field, const uint32_t * OldRange, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	if (row > 100)
		return;
	OBF_vertex vertex = m[row];
	//printf("old range = %d, %d", vertex.getOldRange(), OldRange);

	if (vertex.getOldRange() == *OldRange && !vertex.isInSCC())
		out_field[row - 1] = 1;
}
//
//
__global__ void FWDOWCTY_pivot2(OBF_vertex * m, const uint32_t * FWDOldRange, const uint32_t OWCTYOldRange, const uint32_t * FWDpivot,
	const uint32_t * OWCTYpivot, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (vertex.isInFWD() && vertex.getOldRange() == *FWDOldRange && *FWDpivot == row) {
		vertex.setFWDVisited();
		vertex.setRange(row);
		vertex.setDone2();
		m[row] = vertex;
	}
	else {
		if (vertex.isInOWCTY() && vertex.getOldRange() == OWCTYOldRange && vertex.isReached()) {
			if (row == *OWCTYpivot)
				vertex.setDone2();
			vertex.setRange(*OWCTYpivot);
			m[row] = vertex;
		}
	}
}
//
//
// void cuReduce1_c(uint32_t *g_idata, uint32_t * g_odata)
//{
//	 uint32_t sdata[48];
//	volatile uint32_t * stest = sdata;
//	uint32_t tid = threadIdx.x;
//	stest[tid] = g_idata[tid] + g_idata[tid + 32];
//	//	stest[ tid ] = g_idata[ tid ];
//	// 	sdata[ tid ] += sdata[ tid + 16 ];
//	// 	sdata[ tid ] += sdata[ tid + 8 ];
//	// 	sdata[ tid ] += sdata[ tid + 4 ];
//	// 	sdata[ tid ] += sdata[ tid + 2 ];
//	// 	sdata[ tid ] += sdata[ tid + 1 ];
//
//	// 	if ( tid == 0 )
//	// 		*g_odata = sdata[ 0 ];
//
//	stest[tid] += stest[tid + 16];
//	stest[tid] += stest[tid + 8];
//	stest[tid] += stest[tid + 4];
//	stest[tid] += stest[tid + 2];
//	stest[tid] += stest[tid + 1];
//
//	if (tid == 0)
//		*g_odata = sdata[0];
//}
//
//
__global__ void cuReduce1(uint32_t *g_idata, uint32_t * g_odata)
{
	__shared__ uint32_t sdata[48];
	volatile uint32_t * stest = sdata;
	uint32_t tid = threadIdx.x;
	stest[tid] = g_idata[tid] + g_idata[tid + 32];
	//	stest[ tid ] = g_idata[ tid ];
	// 	sdata[ tid ] += sdata[ tid + 16 ];
	// 	sdata[ tid ] += sdata[ tid + 8 ];
	// 	sdata[ tid ] += sdata[ tid + 4 ];
	// 	sdata[ tid ] += sdata[ tid + 2 ];
	// 	sdata[ tid ] += sdata[ tid + 1 ];
//
	// 	if ( tid == 0 )
	// 		*g_odata = sdata[ 0 ];

	stest[tid] += stest[tid + 16];
	stest[tid] += stest[tid + 8];
	stest[tid] += stest[tid + 4];
	stest[tid] += stest[tid + 2];
	stest[tid] += stest[tid + 1];

	if (tid == 0)
		*g_odata = sdata[0];
}


__global__ void cuReduce(const uint32_t *g_idata, uint32_t *g_odata, const uint32_t n)
{
	__shared__ uint32_t sdata[blockSize];

	uint32_t tid = threadIdx.x;
	if (tid > 128)
	printf("threadidx = %d \t", threadIdx.x);



	uint32_t i = blockIdx.x * (blockSize * 2) + threadIdx.x;
	//printf("i = %d \n", i);
	uint32_t gridSize = blockSize * 2 * gridDim.x;
	//printf("grid size = %d \n", gridSize);

	sdata[tid] = 0;

	while (i < n) {
		sdata[tid] += g_idata[i];
		if (i + blockSize < n)
			sdata[tid] += g_idata[i + blockSize];
		else
			break;
		i += gridSize;
	}
	__syncthreads();

	if (tid < 64)
		sdata[tid] += sdata[tid + 64];
	__syncthreads();

	// 	if ( tid < 32 ) {
	// 		sdata[ tid ] += sdata[ tid + 32 ];
	// 		sdata[ tid ] += sdata[ tid + 16 ];
	// 		sdata[ tid ] += sdata[ tid + 8 ];
	// 		sdata[ tid ] += sdata[ tid + 4 ];
	// 		sdata[ tid ] += sdata[ tid + 2 ];
	// 		sdata[ tid ] += sdata[ tid + 1 ];
	// 	}
	//
	// 	if ( tid == 0 )
	// 		g_odata[ blockIdx.x ] = sdata[ 0 ];
	volatile uint32_t * stest = sdata;
	if (tid < 32) {
		stest[tid] += stest[tid + 32];
		stest[tid] += stest[tid + 16];
		stest[tid] += stest[tid + 8];
		stest[tid] += stest[tid + 4];
		stest[tid] += stest[tid + 2];
		stest[tid] += stest[tid + 1];
	}

	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}


//
//
__global__ void BWDSynchKernel_FWDOWCTYpivot1(OBF_vertex * m, const uint32_t OldRange, uint32_t * NewRange, uint32_t * FWDpivot,
	uint32_t * OWCTYpivot, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInBWD() || vertex.getOldRange() != OldRange)
		return;

	if (row == vertex.getRange())
		*NewRange = row;
	if (vertex.isBWDVisited()) {
		vertex.setOldRange(vertex.getRange());
		vertex.setRange(0);
		vertex.setInFWD();
		if (!*FWDpivot)
			*FWDpivot = row;
	}
	else {
		if (vertex.isBWDPropagate()) {
			*OWCTYpivot = row;
			vertex.setInOWCTY();
			vertex.setReached();
		}
		else
			vertex.setInOWCTY();
	}
	m[row] = vertex;
}
//
//
//
__global__ void BWDTerminateKernel_OWCTY1(OBF_vertex * m, const Edge * Fc, const uint32_t * Fr, const uint32_t OldRange,
	const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInBWD() || vertex.getOldRange() != OldRange)
		return;

	if (vertex.isBWDVisited()) {
		uint32_t row_begin = Fr[row];
		uint32_t row_end = Fr[row + 1];

		for (uint32_t column = row_begin; column < row_end; column++) {
			uint32_t index = Fc[column].getValue();
			OBF_vertex p_vertex = m[index];

			if (p_vertex.getOldRange() != OldRange || !p_vertex.isInBWD() || p_vertex.isBWDVisited())
				continue;
//
			p_vertex.setBWDPropagate();
			m[index] = p_vertex;
		}

		if (m[OldRange].getRange() == vertex.getRange())
			m[row].setInSCC();
	}
}
//
//
__global__ void CheckTerminateKernel(const OBF_vertex * m, uint32_t * terminate, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInSCC())
		*terminate = 0;
}
//
//
__global__ void BWD_pivot2(OBF_vertex * m, const uint32_t OldRange, uint32_t * pivot, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInBWD() || vertex.getOldRange() != OldRange)
		return;
//
	if (vertex.isBWDVisited())
		vertex.setRange(*pivot);

	if (row == *pivot) {
		vertex.setDone2();
	}
	m[row] = vertex;
}
//
//
__global__ void OWCTYSynchKernel_BWDpivot1(OBF_vertex * m, const Edge * Bc, const uint32_t * Br, const uint32_t OldRange, uint32_t * pivot,
	const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInOWCTY() || vertex.getOldRange() != OldRange)
		return;

	if (vertex.isElim()) {
		vertex.setOldRange(row);
		vertex.setInSCC();
		//testing	vertex.setRange( 0 );
	}
	else {
		if (vertex.isReached()) {
			vertex.setInBWD();
			vertex.setBWDVisited();
			*pivot = row;
		}
		else
			vertex.setInBWD();
	}
	m[row] = vertex;
}
//
//
//
__global__ void FWD_pivot2(OBF_vertex * m, const uint32_t OldRange, uint32_t * pivot, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInFWD() || vertex.getOldRange() != OldRange)
		return;
//
	if (vertex.isFWDVisited()) {
		vertex.setOldRange(vertex.getRange());
		OBF_vertex old_pivot = m[vertex.getRange()];
		if (old_pivot.isInBWD())
			vertex.setInBWD();
		else
			vertex.setInOWCTY();
	}
	else {
		if (*pivot == row) {
			vertex.setFWDVisited();
			vertex.setRange(row);
			vertex.setDone2();
		}
	}
	m[row] = vertex;
}

// void FWDSynchKernel_FWDpivot1_c(OBF_vertex * m, const uint32_t OldRange, const Edge * Bc, const uint32_t * Br, uint32_t * pivot,
//	const uint32_t num_rows, uint32_t * COL_pivot = NULL)
//{
//	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	 for (uint32_t row = 0; row <= num_rows; row++)
//	 {
//		 if (row >= num_rows)
//			 return;
//		 OBF_vertex vertex = m[row];
//		 if (!vertex.isInFWD() || vertex.getOldRange() != OldRange)
//			 return;
//
//		 if (vertex.isFWDVisited()) {
//			 if (row == vertex.getRange()) {
//				 if (COL_pivot)
//					 *COL_pivot = row;
//
//				 uint32_t row_begin = Br[row];
//				 uint32_t row_end = Br[row + 1];
//
//				 bool skip = false;
//				 for (uint32_t column = row_begin; column < row_end; column++) {
//					 uint32_t index = Bc[column].getValue();
//					 OBF_vertex p_vertex = m[index];
//
//					 if (p_vertex.isInFWD() && p_vertex.getOldRange() == vertex.getOldRange() && p_vertex.isFWDVisited()) {
//						 skip = true;
//						 break;
//					 }
//				 }
//				 if (skip) {
//					 vertex.setInBWD();
//					 vertex.setBWDVisited();
//				 }
//				 else {
//					 vertex.setInOWCTY();
//					 vertex.setReached();
//				 }
//				 vertex.setOldRange(vertex.getRange());
//				 vertex.setDone2();
//
//				 m[row] = vertex;
//			 }
//		 }
//		 else {
//			 if (!*pivot)
//				 *pivot = row;
//		 }
//	 }
//}
//
//
__global__ void FWDSynchKernel_FWDpivot1(OBF_vertex * m, const uint32_t OldRange, const Edge * Bc, const uint32_t * Br, uint32_t * pivot,
	const uint32_t num_rows, uint32_t * COL_pivot = NULL)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInFWD() || vertex.getOldRange() != OldRange)
		return;

	if (vertex.isFWDVisited()) {
		if (row == vertex.getRange()) {
			if (COL_pivot)
				*COL_pivot = row;

			uint32_t row_begin = Br[row];
			uint32_t row_end = Br[row + 1];

			bool skip = false;
			for (uint32_t column = row_begin; column < row_end; column++) {
				uint32_t index = Bc[column].getValue();
				OBF_vertex p_vertex = m[index];

				if (p_vertex.isInFWD() && p_vertex.getOldRange() == vertex.getOldRange() && p_vertex.isFWDVisited()) {
					skip = true;
					break;
				}
			}
			if (skip) {
				vertex.setInBWD();
				vertex.setBWDVisited();
			}
			else {
				vertex.setInOWCTY();
				vertex.setReached();
			}
			vertex.setOldRange(vertex.getRange());
			vertex.setDone2();

			m[row] = vertex;
		}
	}
	else {
		if (!*pivot)
			*pivot = row;
	}
}


//
//
// void OBFKernel_c(const Edge * Fc, const uint32_t * Fr, const Edge * Bc, const uint32_t * Br, OBF_vertex * m, uint32_t * Temps,
//	const uint32_t num_rows)
//{
//	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	 for (uint32_t row = 0; row <= num_rows; row++)
//	 {
//
//
//		 if (row >= num_rows)
//			 return;
//		 OBF_vertex vertex = m[row];
//
//		 int phase = vertex.getPhase();
//		 if (row == 32734)
//			 printf("%d", vertex.getRange());
//		 if (phase != 3 && vertex.getRange() == row) {
//			 if (!vertex.isDone1() && !vertex.isDone2())
//				 Temps[phase] = vertex.getOldRange() + (phase == 0);
//			 else {
//				 if (vertex.isDone1()) {
//					 vertex.unsetDone1();
//					 vertex.setDone2();
//					 m[row] = vertex;
//				 }
//				 else
//					 m[row].unsetDone2();
//			 }
//		 }
//
//		 uint32_t row_begin;
//		 uint32_t row_end;
//		 bool prop;
//		 switch (phase) {
//		 case 0: //FWD
//			 if (!vertex.isFWDVisited() || vertex.isFWDPropagate())
//				 continue;
//
//			 row_begin = Fr[row];
//			 row_end = Fr[row + 1];
//
//			 prop = false;
//			 for (uint32_t column = row_begin; column < row_end; column++) {
//				 uint32_t index = Fc[column].getValue();
//				 OBF_vertex p_vertex = m[index];
//
//				 if (!p_vertex.isInFWD() || p_vertex.getOldRange() != vertex.getOldRange() || p_vertex.isFWDVisited())
//					 continue;
//
//				 p_vertex.setFWDVisited();
//				 p_vertex.setRange(vertex.getRange());
//				 m[index] = p_vertex;
//				 prop = true;
//			 }
//			 vertex.setFWDPropagate();
//			 m[row] = vertex;
//			 if (prop)
//				 m[vertex.getRange()].setDone1();
//
//			 break;
//		 case 1: //OWCTY
//			 if (!vertex.isReached() || vertex.isElim())
//				 continue;
//
//			 row_begin = Br[row];
//			 row_end = Br[row + 1];
//
//			 prop = true;
//			 for (uint32_t column = row_begin; column < row_end; column++) {
//				 uint32_t index = Bc[column].getValue();
//				 OBF_vertex p_vertex = m[index];
//
//				 if (p_vertex.isInOWCTY() && p_vertex.getOldRange() == vertex.getOldRange() && !p_vertex.isElim())// && index != row )
//					 prop = false;
//			 }
//			 if (prop) {
//				 row_begin = Fr[row];
//				 row_end = Fr[row + 1];
//
//				 for (uint32_t column = row_begin; column < row_end; column++) {
//					 uint32_t index = Fc[column].getValue();
//					 OBF_vertex p_vertex = m[index];
//
//					 if (!p_vertex.isInOWCTY() || p_vertex.getOldRange() != vertex.getOldRange() || p_vertex.isReached())
//						 continue;
//
//					 p_vertex.setReached();
//					 p_vertex.setRange(vertex.getRange());
//					 m[index] = p_vertex;
//				 }
//				 vertex.setElim();
//				 m[row] = vertex;
//				 m[vertex.getRange()].setDone1();
//			 }
//
//			 break;
//		 case 2: //BWD
//			 if (!vertex.isBWDVisited() || vertex.isBWDPropagate())
//				 continue;
//
//			 row_begin = Br[row];
//			 row_end = Br[row + 1];
//
//			 prop = false;
//			 for (uint32_t column = row_begin; column < row_end; column++) {
//				 uint32_t index = Bc[column].getValue();
//				 OBF_vertex p_vertex = m[index];
//
//				 if (!p_vertex.isInBWD() || p_vertex.getOldRange() != vertex.getOldRange() || p_vertex.isBWDVisited())
//					 continue;
//
//				 p_vertex.setBWDVisited();
//				 p_vertex.setRange(vertex.getRange());
//				 m[index] = p_vertex;
//				 prop = true;
//			 }
//			 vertex.setBWDPropagate();
//			 m[row] = vertex;
//			 if (prop)
//				 m[vertex.getRange()].setDone1();
//
//			 break;
//		 }
//	 }
//}
//
//
__global__ void OBFKernel(const Edge * Fc, const uint32_t * Fr, const Edge * Bc, const uint32_t * Br, OBF_vertex * m, uint32_t * Temps,
	const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];

	int phase = vertex.getPhase();

	if (phase != 3 && vertex.getRange() == row) {
		if (!vertex.isDone1() && !vertex.isDone2())
			Temps[phase] = vertex.getOldRange() + (phase == 0);
		else {
			if (vertex.isDone1()) {
				vertex.unsetDone1();
				vertex.setDone2();
				m[row] = vertex;
			}
			else
				m[row].unsetDone2();
		}
	}
//
	uint32_t row_begin;
	uint32_t row_end;
	bool prop;
	switch (phase) {
	case 0: //FWD
		if (!vertex.isFWDVisited() || vertex.isFWDPropagate())
			return;

		row_begin = Fr[row];
		row_end = Fr[row + 1];
//
		prop = false;
		for (uint32_t column = row_begin; column < row_end; column++) {
			uint32_t index = Fc[column].getValue();
			OBF_vertex p_vertex = m[index];

			if (!p_vertex.isInFWD() || p_vertex.getOldRange() != vertex.getOldRange() || p_vertex.isFWDVisited())
				continue;

			p_vertex.setFWDVisited();
			p_vertex.setRange(vertex.getRange());
			m[index] = p_vertex;
			prop = true;
		}
		vertex.setFWDPropagate();
		m[row] = vertex;
		if (prop)
			m[vertex.getRange()].setDone1();

		break;
	case 1: //OWCTY
		if (!vertex.isReached() || vertex.isElim())
			return;

		row_begin = Br[row];
		row_end = Br[row + 1];
//
		prop = true;
		for (uint32_t column = row_begin; column < row_end; column++) {
			uint32_t index = Bc[column].getValue();
			OBF_vertex p_vertex = m[index];

			if (p_vertex.isInOWCTY() && p_vertex.getOldRange() == vertex.getOldRange() && !p_vertex.isElim())// && index != row )
				prop = false;
		}
		if (prop) {
			row_begin = Fr[row];
			row_end = Fr[row + 1];
//
			for (uint32_t column = row_begin; column < row_end; column++) {
				uint32_t index = Fc[column].getValue();
				OBF_vertex p_vertex = m[index];

				if (!p_vertex.isInOWCTY() || p_vertex.getOldRange() != vertex.getOldRange() || p_vertex.isReached())
					continue;

				p_vertex.setReached();
				p_vertex.setRange(vertex.getRange());
				m[index] = p_vertex;
			}
			vertex.setElim();
			m[row] = vertex;
			m[vertex.getRange()].setDone1();
		}
//
		break;
	case 2: //BWD
		if (!vertex.isBWDVisited() || vertex.isBWDPropagate())
			return;

		row_begin = Br[row];
		row_end = Br[row + 1];

		prop = false;
		for (uint32_t column = row_begin; column < row_end; column++) {
			uint32_t index = Bc[column].getValue();
			OBF_vertex p_vertex = m[index];

			if (!p_vertex.isInBWD() || p_vertex.getOldRange() != vertex.getOldRange() || p_vertex.isBWDVisited())
				continue;

			p_vertex.setBWDVisited();
			p_vertex.setRange(vertex.getRange());
			m[index] = p_vertex;
			prop = true;
		}
		vertex.setBWDPropagate();
		m[row] = vertex;
		if (prop)
			m[vertex.getRange()].setDone1();

		break;
	}
}
//
//
// void _Trimming_c(OBF_vertex * m, const Edge * Bc, const uint32_t * Br, const uint32_t num_rows,
//	uint32_t * terminate)
//{
//	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	 for (uint32_t row = 0; row <= num_rows; row++)
//	 {
//
//		 if (row >= num_rows)
//			 return;
//		 OBF_vertex vertex = m[row];
//
//		 if (vertex.isInSCC())
//		 {
//			 continue;
//
//		 }
//
//
//		 uint32_t row_begin = Br[row];
//		 uint32_t row_end = Br[row + 1];
//
//		 bool eliminate = true;
//		 for (uint32_t column = row_begin; column < row_end; column++) {
//			 //printf("col = %d ,old range = %d \n", column, vertex.getOldRange());
//
//			 uint32_t index = Bc[column].getValue();
//			 OBF_vertex p_vertex = m[index];
//
//			 if (!p_vertex.isInSCC())
//			 {
//				 eliminate = false;
//			 }
//
//		 }
//		 /*	if ( !eliminate ) {
//		 eliminate = true;
//		 row_begin = Fr[ row ];
//		 row_end = Fr[ row + 1 ];
//
//		 for ( uint32_t column = row_begin; column < row_end; column++ ) {
//		 uint32_t index = Fc[ column ].getValue();
//		 OBF_vertex p_vertex = m[ index ];
//
//		 if ( !p_vertex.isInSCC() )
//		 eliminate = false;
//		 }
//		 }*/
//		 if (eliminate) {
//
//			 vertex.setOldRange(row);
//			 vertex.setInSCC();
//			 m[row] = vertex;
//			 *terminate = 1;
//		 }
//	 }
//	return;
//}
//
//


__global__ void _Trimming(OBF_vertex * m, const Edge * Bc, const uint32_t * Br, const uint32_t num_rows,
	uint32_t * terminate)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];

	if (vertex.isInSCC())
	{
		return;

	}


	uint32_t row_begin = Br[row];
	uint32_t row_end = Br[row + 1];

	bool eliminate = true;
	for (uint32_t column = row_begin; column < row_end; column++) {
		//printf("col = %d ,old range = %d \n", column, vertex.getOldRange());

		uint32_t index = Bc[column].getValue();
		OBF_vertex p_vertex = m[index];

		if (!p_vertex.isInSCC())
		{
			eliminate = false;
		}

	}
	/*	if ( !eliminate ) {
	eliminate = true;
	row_begin = Fr[ row ];
	row_end = Fr[ row + 1 ];

	for ( uint32_t column = row_begin; column < row_end; column++ ) {
	uint32_t index = Fc[ column ].getValue();
	OBF_vertex p_vertex = m[ index ];

	if ( !p_vertex.isInSCC() )
	eliminate = false;
	}
	}*/
	if (eliminate) {

		vertex.setOldRange(row);
		vertex.setInSCC();
		m[row] = vertex;
		*terminate = 1;
	}
	return;
}



//
// void CheckTerminateSetKernel_c(const OBF_vertex * m, uint32_t * terminate, const uint32_t num_rows)
//{
//	//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	 for (uint32_t row = 0; row <= num_rows; row++)
//	 {
//		 if (row >= num_rows)
//			 return;
//		 OBF_vertex vertex = m[row];
//		 if (!vertex.isInSCC())
//			 *terminate = row;
//		 else
//			 printf(" %d ", row);
//	 }
//}
//
__global__ void CheckTerminateSetKernel(const OBF_vertex * m, uint32_t * terminate, const uint32_t num_rows)
{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row >= num_rows)
		return;
	OBF_vertex vertex = m[row];
	if (!vertex.isInSCC())
		*terminate = row;
}




pair <uint32_t, float> OBF_Decomposition(uint32_t CSize, uint32_t RSize, Edge *Fc, uint32_t *Fr, Edge * Bc, uint32_t * Br, OBF_vertex * m,ASF_vertex* o_m,Dimension*d,
	bool COL_enforced, uint32_t COL_limit, bool trimm, bool quiet)
{
	//-----------GPU initialization---------------------------->
	uint32_t  * d_Fr, *d_Br;
	Edge * d_Fc, *d_Bc;
	OBF_vertex * d_m;
	float temp;

	uint32_t * d_pivot;
	uint32_t * d_Temps;//[0]-d_FWD, [1]-d_OWCTY, [2]-d_BWD
	uint32_t Temps[Temp_count];
	uint32_t terminate = 1;
	int interruptions = 0;

#ifdef _DEBUG
	int FWD_ints = 0;
	int OWCTY_ints = 0;
	int BWD_ints = 0;
	StopWatchInterface* KernelTime = 0;
	StopWatchInterface* IntTime = 0;
	(sdkCreateTimer(&KernelTime));
	(sdkCreateTimer(&IntTime));
#endif

	if (!_DeviceSet) {
		_DeviceSet = true;
		checkCudaErrors(hipSetDevice(2));
	}

	if (COL_enforced)
		COL_limit = RSize / 100;

	hipError_t e1, e2, e3, e4, e5, e6, e7;
	checkCudaErrors(e1 = hipMalloc((void**)&d_Fc, CSize * sizeof(Edge)));
	checkCudaErrors(e2 = hipMalloc((void**)&d_Fr, RSize * sizeof(uint32_t)));
	checkCudaErrors(e3 = hipMalloc((void**)&d_Bc, CSize * sizeof(Edge)));
	checkCudaErrors(e4 = hipMalloc((void**)&d_Br, RSize * sizeof(uint32_t)));
	checkCudaErrors(e5 = hipMalloc((void**)&d_m, (RSize - 1) * sizeof(OBF_vertex)));
	checkCudaErrors(e6 = hipMalloc((void**)&d_Temps, Temp_count * sizeof(uint32_t)));
	checkCudaErrors(e7 = hipMalloc((void**)&d_pivot, sizeof(uint32_t)));

	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory ||
		e3 == hipErrorOutOfMemory || e4 == hipErrorOutOfMemory ||
		e5 == hipErrorOutOfMemory || e6 == hipErrorOutOfMemory ||
		e7 == hipErrorOutOfMemory) {
		throw "Error: Not enough memory on GPU\n";
	}

	//col
	//unsigned int COLTime = 0;
	StopWatchInterface* COLTime = 0;
	(sdkCreateTimer(&COLTime));
	uint32_t * d_temp_COL;
	uint32_t * d_temp_COL2;
	uint32_t * d_COL_OldRange;
	bool COL_used = false;
	COL_vertex * d_cm;

	checkCudaErrors(e1 = hipMalloc((void**)&d_temp_COL, (RSize - 2) * sizeof(uint32_t)));
	checkCudaErrors(e2 = hipMalloc((void**)&d_temp_COL2, 64 * sizeof(uint32_t)));
	checkCudaErrors(e3 = hipMalloc((void**)&d_COL_OldRange, sizeof(uint32_t)));
	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory || e3 == hipErrorOutOfMemory) {
		throw "Error: Not enough memory on GPU\n";
	}

	checkCudaErrors(hipMemcpy(d_Fc, Fc, CSize * sizeof(Edge), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Fr, Fr, RSize * sizeof(uint32_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Bc, Bc, CSize * sizeof(Edge), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_Br, Br, RSize * sizeof(uint32_t), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_m, m, (RSize - 1) * sizeof(OBF_vertex), hipMemcpyHostToDevice));

	//unsigned int SCCTime = 0;
	StopWatchInterface* SCCTime = 0;
	(sdkCreateTimer(&SCCTime));
	(sdkStartTimer(&SCCTime));

	dim3 grid(((RSize + 510) / 512), 1, 1);  // (RSize-2) valid vertecis the first one is undefined
	dim3 threads(512, 1, 1);
	dim3 grid1(64, 1, 1);
	dim3 threads1(blockSize, 1, 1);
	dim3 grid2(1, 1, 1);
	dim3 threads2(32, 1, 1);
	//<----------GPU initialization-----------------------------
	//if (!quiet)
	{
		// 		printf("Computing OBF decomposition \n");
		// 		printf("Size: %u, blockdim.x: %u, threaddim.x: %u\n", RSize - 1, grid.x, threads.x);
		printf("Vertices: %u\n", RSize - 2);
		printf("Edges: %u\n", CSize);
	}
	//-----------Main algorithm-------------------------------->
	OBF_vertex initial;
	initial.setInSCC();
	checkCudaErrors(hipMemcpy(&(d_m[0]), &initial, sizeof(OBF_vertex), hipMemcpyHostToDevice));
	//-----------Trimming-------------------------------------->
	if (trimm) {
		uint32_t tits = 1;
		do {
			if (!(tits++ & 0x7FF)) {
				temp = sdkGetTimerValue(&SCCTime);
				if (temp > Time_limit)
					break;
			}
			checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
			_Trimming << <grid, threads >> >(d_m, d_Bc, d_Br, RSize - 1, d_pivot);
			checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
		} while (terminate);
		checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
		CheckTerminateSetKernel << <grid, threads >> >(d_m, d_pivot, RSize - 1);
		checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
	}
	//<----------Trimming---------------------------------------

	//-----------Initial FWD pivot setup----------------------->
	if (terminate) {
		initial.setInFWD();
		initial.setRange(terminate);
		initial.setFWDVisited();
		initial.setDone2();
		checkCudaErrors(hipMemcpy(&(d_m[terminate]), &initial, sizeof(OBF_vertex), hipMemcpyHostToDevice));
		terminate = 0;
	}
	else
		terminate = 1;
	//<----------Initial FWD pivot setup------------------------

#ifdef _DEBUG
	long int its = 0;
#endif

	int i;
	while (!terminate) {

#ifdef _DEBUG
		its++;
		//printf("its: %d\n", its);
		hipDeviceSynchronize();
		(sdkStartTimer(&KernelTime));
#endif


		checkCudaErrors(hipMemset(d_Temps, 0, Temp_count * sizeof(uint32_t)));
		OBFKernel << <grid, threads >> >(d_Fc, d_Fr, d_Bc, d_Br, d_m, d_Temps, RSize - 1);
		checkCudaErrors(hipMemcpy(Temps, d_Temps, Temp_count * sizeof(uint32_t), hipMemcpyDeviceToHost));

#ifdef _DEBUG
		hipDeviceSynchronize();
		(sdkStopTimer(&KernelTime));
		(sdkStartTimer(&IntTime));
#endif

		//-----------Interrupt handling---------------------------->
		for (i = 0; i < Temp_count; i++)
		if (Temps[i])
			break;

		if (i == Temp_count) {
			continue;
		}

		interruptions++;
		//printf("inters: %d\n", i);

		//time_limit
		if (!(interruptions & 0x7FF)) {
			temp = sdkGetTimerValue(&SCCTime);
			if (temp > Time_limit)
				break;
		}

		checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
		switch (i) {
		case 0:

#ifdef _DEBUG
			FWD_ints++;
#endif

			Temps[0]--;

			if (COL_enforced)
				FWDSynchKernel_FWDpivot1 << <grid, threads >> >(d_m, Temps[0], d_Bc, d_Br, d_pivot, RSize - 1, d_COL_OldRange);
			else
				FWDSynchKernel_FWDpivot1 << <grid, threads >> >(d_m, Temps[0], d_Bc, d_Br, d_pivot, RSize - 1);
			FWD_pivot2 << <grid, threads >> >(d_m, Temps[0], d_pivot, RSize - 1);
			break;
		case 1:

#ifdef _DEBUG
			OWCTY_ints++;
#endif
			OWCTYSynchKernel_BWDpivot1 << <grid, threads >> >(d_m, d_Bc, d_Br, Temps[1], d_pivot, RSize - 1);
			BWD_pivot2 << <grid, threads >> >(d_m, Temps[1], d_pivot, RSize - 1);
			if (COL_enforced)
				checkCudaErrors(hipMemcpy(d_COL_OldRange, &(d_Temps[1]), sizeof(uint32_t), hipMemcpyDeviceToDevice));
			break;
		case 2:

#ifdef _DEBUG
			BWD_ints++;
#endif
			BWDTerminateKernel_OWCTY1 << <grid, threads >> >(d_m, d_Fc, d_Fr, Temps[2], RSize - 1);
			BWDSynchKernel_FWDOWCTYpivot1 << <grid, threads >> >(d_m, Temps[2], &(d_Temps[2]), d_pivot, &(d_Temps[0]), RSize - 1);
			FWDOWCTY_pivot2 << <grid, threads >> >(d_m, &(d_Temps[2]), Temps[2], d_pivot, &(d_Temps[0]), RSize - 1);
			if (COL_enforced)
				checkCudaErrors(hipMemcpy(d_COL_OldRange, &(d_Temps[2]), sizeof(uint32_t), hipMemcpyDeviceToDevice));
			break;
		}
		if (COL_enforced) {

#ifdef _DEBUG
			hipDeviceSynchronize();
			(sdkStartTimer(&COLTime));
			(sdkStopTimer(&IntTime));
#endif

			checkCudaErrors(hipMemset(d_temp_COL, 0, (RSize - 2) * sizeof(uint32_t)));
			OBFCOLcompute << <grid, threads >> >(d_m, d_temp_COL, d_COL_OldRange, RSize - 1);
			cuReduce << <grid1, threads1 >> >(d_temp_COL, d_temp_COL2, RSize - 2);
			cuReduce1 << <grid2, threads2 >> >(d_temp_COL2, d_pivot);
			checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
			if (terminate < COL_limit && terminate != 0) {
				COL_used = true;
				SetCOL << <grid, threads >> >(d_m, d_COL_OldRange, RSize - 1);
			}

#ifdef _DEBUG
			hipDeviceSynchronize();
			(sdkStopTimer(&COLTime));
			(sdkStartTimer(&IntTime));
#endif

		}
		//<----------Interrupt handling-----------------------------

		//-----------Termination detection------------------------->
		checkCudaErrors(hipMemset(d_pivot, 1, sizeof(uint32_t)));
		CheckTerminateKernel << <grid, threads >> >(d_m, d_pivot, RSize - 1);
		checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
		//<----------Termination detection--------------------------

#ifdef _DEBUG
		hipDeviceSynchronize();
		(sdkStopTimer(&IntTime));
#endif

	}
	//<----------Main algorithm---------------------------------

	//-----------COL finish------------------------------------>
	if (COL_enforced && COL_used) {

#ifdef _DEBUG
		hipDeviceSynchronize();
		(sdkStartTimer(&COLTime));
#endif

		checkCudaErrors(hipMalloc((void**)&d_cm, (RSize - 1) * sizeof(COL_vertex)));
		OBFtoCOLKernel << <grid, threads >> >(d_m, d_cm, RSize - 1);

		int c_its = 0;
		do {
			c_its++;

			//time_limit
		/*	if (!(c_its & 0x7)) {
				temp = sdkGetTimerValue(&SCCTime);
				if (temp > Time_limit)
					break;
			}*/

			do {
				checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
				p_COL_MAP << <grid, threads >> >(d_Fc, d_Fr, d_cm, RSize - 1, MIN, d_pivot);
				checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
			} while (terminate != 0);

			setBWDseed << <grid, threads >> >(d_cm, RSize - 1, MIN);
			do {
				checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
				COL_FWD << <grid, threads >> >(d_Fc, d_Fr, d_cm, RSize - 1, d_pivot);
				checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
			} while (terminate != 0);

			checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
			checkTerminateAndSetOldMap << <grid, threads >> >(d_cm, RSize - 1, d_pivot);
			checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
		} while (terminate != 0);

#ifdef _DEBUG
		hipDeviceSynchronize();
		(sdkStopTimer(&COLTime));
#endif

	}
	//<----------COL finish-------------------------------------

	//-----------Scc extraction-------------------------------->


	uint32_t* Fr_F = new uint32_t[RSize];
	memset(Fr_F, 0, RSize*sizeof(uint32_t));
	checkCudaErrors(hipMemset(d_Fr, 0, (RSize - 2) * sizeof(uint32_t)));
	if (COL_enforced && COL_used)
		COLcomputeSCCs << <grid, threads >> >(d_cm, d_Fr, RSize - 1, MIN);
	else
	{
		checkCudaErrors(hipMemcpy(Fr, d_Fr, RSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(m, d_m, (RSize - 1) * sizeof(OBF_vertex), hipMemcpyDeviceToHost));

		//OBFcomputeSCCs << <grid, threads >> >(d_m, d_Fr, RSize - 1);
		OBFcomputeSCCs_c(m,o_m,d, Fr, RSize - 1);

	}
	//checkCudaErrors(hipMemcpy(Fr, d_Fr, RSize * sizeof(uint32_t), hipMemcpyDeviceToHost));

	int cc = 0;
	for (int jj = 0; jj < RSize-2; jj++)
	{
		if (Fr[jj] == 0)
			cc++;// printf("%d ", jj);
	}
	/*cuReduce << <grid1, threads1 >> >(d_Fr, d_Br, RSize - 2);
	checkCudaErrors(hipMemcpy(Br, d_Br, RSize * sizeof(uint32_t), hipMemcpyDeviceToHost));

	cuReduce1 << <grid2, threads2 >> >(d_Br, d_pivot);
	checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));*/
	//<----------Scc extraction---------------------------------
	(sdkStopTimer(&SCCTime));
	float f;
	//if (!quiet)
		//		printf("%u components found.\n", terminate);
		printf("Components: %u\n", cc);

#ifdef _DEBUG
	printf("Kernel iterations = %d\n", its);
	printf("Interruptions =  %d, FWD interruptions = %d, OWCTY interruptions = %d, BWD interruptions = %d \n", interruptions, FWD_ints,
		OWCTY_ints, BWD_ints);
	f = sdkGetTimerValue(&KernelTime);
	printf("Uninterrupted time = %f ms\n", f);
	f = sdkGetTimerValue(&IntTime);
	printf("Interruption time = %f ms\n", f);
	if (COL_enforced) {
		f = sdkGetTimerValue(&COLTime);
		printf("Colouring time = %f ms\n", f);
	}
#endif

	f = sdkGetTimerValue(&SCCTime);
	int min = (int)(f / 60000.0f);
	int sec = (int)(f / 1000.0f) % 60;
	//if (!quiet)
		//		printf("CUDA SCC decomposition time: %d minutes %d seconds (%f ms).\n", min, sec, f);
		printf("Time: %f ms\n", f);

	//CUT_CHECK_ERROR("Kernel execution failed");

	checkCudaErrors(hipFree(d_Fc));
	checkCudaErrors(hipFree(d_Fr));
	checkCudaErrors(hipFree(d_Bc));
	checkCudaErrors(hipFree(d_Br));
	checkCudaErrors(hipFree(d_m));
	checkCudaErrors(hipFree(d_Temps));
	checkCudaErrors(hipFree(d_pivot));
	if (COL_used)
		checkCudaErrors(hipFree(d_cm));
	checkCudaErrors(hipFree(d_temp_COL));
	checkCudaErrors(hipFree(d_temp_COL2));
	checkCudaErrors(hipFree(d_COL_OldRange));

	(sdkDeleteTimer(&SCCTime));

#ifdef _DEBUG
	(sdkDeleteTimer(&KernelTime));
	(sdkDeleteTimer(&IntTime));
	if (COL_enforced)
		(sdkDeleteTimer(&COLTime));
#endif

	return make_pair(terminate, f);
}




//
//pair <uint32_t, float> OBF_Decomposition(uint32_t CSize, uint32_t RSize, Edge *Fc, uint32_t *Fr, Edge * Bc, uint32_t * Br, OBF_vertex * m,
//	bool COL_enforced, uint32_t COL_limit, bool trimm, bool quiet)
//{
//	//-----------GPU initialization---------------------------->
//	uint32_t  * d_Fr, *d_Br;
//	Edge * d_Fc, *d_Bc;
//	OBF_vertex * d_m;
//	float temp;
//
//	uint32_t * d_pivot;
//	uint32_t * d_Temps;//[0]-d_FWD, [1]-d_OWCTY, [2]-d_BWD
//	uint32_t Temps[Temp_count];
//	uint32_t terminate = 1;
//	int interruptions = 0;
//	pair <uint32_t, float> result;
//
//
//#ifdef _DEBUG
//int FWD_ints = 0;
//int OWCTY_ints = 0;
//int BWD_ints = 0;
//StopWatchInterface* KernelTime = NULL;
//StopWatchInterface* IntTime = NULL;
//sdkCreateTimer(&KernelTime);
//sdkCreateTimer(&IntTime);
//#endif
//
//	if( !_DeviceSet ) {
//		_DeviceSet = true;
//		checkCudaErrors(hipSetDevice(1));
//	}
//
//	if ( COL_enforced )
//		COL_limit = RSize / 100;
//
//	hipError_t e1, e2, e3, e4, e5, e6, e7;
//	checkCudaErrors(e1 = hipMalloc((void**)&d_Fc, CSize * sizeof(Edge)));
//	checkCudaErrors(e2 = hipMalloc((void**)&d_Fr, RSize * sizeof(uint32_t)));
//	checkCudaErrors(e3 = hipMalloc((void**)&d_Bc, CSize * sizeof(Edge)));
//	checkCudaErrors(e4 = hipMalloc((void**)&d_Br, RSize * sizeof(uint32_t)));
//	checkCudaErrors(e5 = hipMalloc((void**)&d_m, (RSize - 1) * sizeof(OBF_vertex)));
//	checkCudaErrors(e6 = hipMalloc((void**)&d_Temps, Temp_count * sizeof(uint32_t)));
//	checkCudaErrors(e7 = hipMalloc((void**)&d_pivot, sizeof(uint32_t)));
//
//	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory ||
//		e3 == hipErrorOutOfMemory || e4 == hipErrorOutOfMemory ||
//		e5 == hipErrorOutOfMemory || e6 == hipErrorOutOfMemory ||
//		e7 == hipErrorOutOfMemory ) {
//		throw "Error: Not enough memory on GPU\n";
//	}
//
////col
//	StopWatchInterface* COLTime = 0;
//	sdkCreateTimer(&COLTime);
//	uint32_t * d_temp_COL;
//	uint32_t * d_temp_COL2;
//	uint32_t * d_COL_OldRange;
//	bool COL_used = false;
//	COL_vertex * d_cm;
//
//	checkCudaErrors(e1 = hipMalloc((void**)&d_temp_COL, (RSize - 2) * sizeof(uint32_t)));
//	checkCudaErrors(e2 = hipMalloc((void**)&d_temp_COL2, 64 * sizeof(uint32_t)));
//	checkCudaErrors(e3 = hipMalloc((void**)&d_COL_OldRange, sizeof(uint32_t)));
//	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory || e3 == hipErrorOutOfMemory ) {
//		throw "Error: Not enough memory on GPU\n";
//	}
//
//	checkCudaErrors(hipMemcpy(d_Fc, Fc, CSize * sizeof(Edge), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_Fr, Fr, RSize * sizeof(uint32_t), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_Bc, Bc, CSize * sizeof(Edge), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_Br, Br, RSize * sizeof(uint32_t), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(d_m, m, (RSize - 1) * sizeof(OBF_vertex), hipMemcpyHostToDevice));
//
//	//unsigned int SCCTime = 0;
//	StopWatchInterface* SCCTime = NULL;
//	sdkCreateTimer(&SCCTime);
//	sdkStartTimer(&SCCTime);
//
//	dim3 grid( ((RSize + 510) / 512), 1, 1 );  // (RSize-2) valid vertecis the first one is undefined
//	dim3 threads(512, 1, 1);
//	dim3 grid1(64, 1, 1);
//	dim3 threads1(blockSize, 1, 1);
//	dim3 grid2(1, 1, 1);
//	dim3 threads2(32, 1, 1);
////<----------GPU initialization-----------------------------
//	if ( !quiet ) {
// 		printf("Computing OBF decomposition \n");
// 		printf("Size: %u, blockdim.x: %u, threaddim.x: %u\n", RSize - 1, grid.x, threads.x);
//		printf( "Vertices: %u\n", RSize - 2 );
//		printf( "Edges: %u\n", CSize );
//	}
////-----------Main algorithm-------------------------------->
//		OBF_vertex initial;
//		initial.setInSCC();
//		checkCudaErrors(hipMemcpy(&(d_m[0]), &initial, sizeof(OBF_vertex), hipMemcpyHostToDevice));
////-----------Trimming-------------------------------------->
//	if ( trimm ) {
//		uint32_t tits = 1;
//		do {
//			if ( !(tits++ & 0x7FF) ) {
//				temp = sdkGetTimerValue( &SCCTime );
//				if ( temp > Time_limit )
//					break;
//			}
//			checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
//		//	_Trimming_c(m,Bc,Br,RSize - 1,&terminate);
//			_Trimming<<<grid, threads>>>( d_m, d_Bc, d_Br, RSize - 1, d_pivot );
//			checkCudaErrors(hipMemcpy(&terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost));
//		} while ( terminate );
//		checkCudaErrors(hipMemset(d_pivot, 0, sizeof(uint32_t)));
//		CheckTerminateSetKernel << <grid, threads >> >(d_m, d_pivot, RSize - 1);
//		//CheckTerminateSetKernel_c (m, &terminate, RSize - 1);
//		checkCudaErrors( hipMemcpy( &terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost ));
//	}
////<----------Trimming---------------------------------------
//
//
//
////-----------Initial FWD pivot setup----------------------->
//	if ( terminate ) {
//		initial.setInFWD();
//		initial.setRange( terminate );
//		initial.setFWDVisited();
//		initial.setDone2();
//		checkCudaErrors( hipMemcpy( &(d_m[ terminate ]), &initial, sizeof(OBF_vertex), hipMemcpyHostToDevice ));
//		terminate = 0;
//	}
//	else
//		terminate = 1;
////<----------Initial FWD pivot setup------------------------
//
//#ifdef _DEBUG
//long int its = 0;
//#endif
//
//checkCudaErrors(hipMemcpy(Fc, d_Fc, CSize * sizeof(Edge), hipMemcpyDeviceToHost));
//checkCudaErrors(hipMemcpy(Fr, d_Fr, RSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
//checkCudaErrors(hipMemcpy(Bc, d_Bc, CSize * sizeof(Edge), hipMemcpyDeviceToHost));
//checkCudaErrors(hipMemcpy(Br, d_Br, RSize * sizeof(uint32_t), hipMemcpyDeviceToHost));
//checkCudaErrors(hipMemcpy(m, d_m, (RSize - 1) * sizeof(OBF_vertex), hipMemcpyDeviceToHost));
//
//	int i;
//	while ( !terminate ) {
//
//#ifdef _DEBUG
//its++;
////printf("its: %d\n", its);
//hipDeviceSynchronize();
//(sdkStartTimer(&KernelTime));
//#endif
//
//
//		checkCudaErrors( hipMemset( d_Temps, 0, Temp_count * sizeof(uint32_t) ));
//		//OBFKernel_c( Fc, Fr, Bc, Br, m, Temps, RSize - 1 );
//		OBFKernel<<<grid, threads>>>( d_Fc, d_Fr, d_Bc, d_Br, d_m, d_Temps, RSize - 1 );
//		checkCudaErrors(hipMemcpy(Temps, d_Temps, Temp_count * sizeof(uint32_t), hipMemcpyDeviceToHost));
//
//#ifdef _DEBUG
//hipDeviceSynchronize();
//( sdkStopTimer( &KernelTime ));
//( sdkStartTimer( &IntTime ));
//#endif
//
////-----------Interrupt handling---------------------------->
//		for ( i = 0; i < Temp_count; i++ )
//			if ( Temps[ i ] )
//				break;
//
//		if ( i == Temp_count ) {
//			continue;
//		}
//
//interruptions++;
////printf("inters: %d\n", i);
//
////time_limit
//		if ( !(interruptions & 0x7FF) ) {
//			temp = sdkGetTimerValue( &SCCTime );
//			if ( temp > Time_limit )
//				break;
//		}
//
//		checkCudaErrors( hipMemset( d_pivot, 0, sizeof(uint32_t) ));
//		switch ( i ) {
//			case 0:
//
//#ifdef _DEBUG
//FWD_ints++;
//#endif
//
//				Temps[ 0 ]--;
//
//				if ( COL_enforced )
//					FWDSynchKernel_FWDpivot1<<<grid, threads>>>( d_m, Temps[ 0 ], d_Bc, d_Br, d_pivot, RSize - 1, d_COL_OldRange );
//				else
//					FWDSynchKernel_FWDpivot1<<<grid, threads>>>( d_m, Temps[ 0 ], d_Bc, d_Br, d_pivot, RSize - 1 );
//				FWD_pivot2<<<grid, threads>>>( d_m, Temps[ 0 ], d_pivot, RSize - 1 );
//				break;
//			case 1:
//
//#ifdef _DEBUG
//OWCTY_ints++;
//#endif
//				OWCTYSynchKernel_BWDpivot1<<<grid, threads>>>( d_m, d_Bc, d_Br, Temps[ 1 ], d_pivot, RSize - 1 );
//				BWD_pivot2<<<grid, threads>>>( d_m, Temps[ 1 ], d_pivot, RSize - 1 );
//				if ( COL_enforced )
//					checkCudaErrors( hipMemcpy( d_COL_OldRange, &(d_Temps[ 1 ]), sizeof(uint32_t), hipMemcpyDeviceToDevice ));
//				break;
//			case 2:
//
//#ifdef _DEBUG
//BWD_ints++;
//#endif
//				BWDTerminateKernel_OWCTY1<<<grid, threads>>>( d_m, d_Fc, d_Fr, Temps[ 2 ], RSize - 1 );
//				BWDSynchKernel_FWDOWCTYpivot1<<<grid, threads>>>( d_m, Temps[ 2 ], &(d_Temps[ 2 ]), d_pivot, &(d_Temps[ 0 ]), RSize - 1 );
//				FWDOWCTY_pivot2<<<grid, threads>>>( d_m, &(d_Temps[ 2 ]), Temps[ 2 ], d_pivot, &(d_Temps[ 0 ]), RSize - 1 );
//				if ( COL_enforced )
//					checkCudaErrors( hipMemcpy( d_COL_OldRange, &(d_Temps[ 2 ]), sizeof(uint32_t), hipMemcpyDeviceToDevice ));
//				break;
//		}
//		if ( COL_enforced ) {
//
//#ifdef _DEBUG
//hipDeviceSynchronize();
//( sdkStartTimer( &COLTime ));
//( sdkStopTimer( &IntTime ));
//#endif
//
//			checkCudaErrors( hipMemset( d_temp_COL, 0, (RSize - 2) * sizeof(uint32_t) ));
//			OBFCOLcompute<<<grid, threads>>>( d_m, d_temp_COL, d_COL_OldRange, RSize - 1 );
//			cuReduce<<<grid1, threads1>>>( d_temp_COL, d_temp_COL2, RSize - 2 );
//			cuReduce1<<<grid2, threads2>>>( d_temp_COL2, d_pivot );
//			checkCudaErrors( hipMemcpy( &terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost ));
//			if ( terminate < COL_limit && terminate != 0 ) {
//				COL_used = true;
//				SetCOL<<<grid, threads>>>( d_m, d_COL_OldRange, RSize - 1 );
//			}
//
//#ifdef _DEBUG
//hipDeviceSynchronize();
//checkCudaErrors( sdkStopTimer( &COLTime ));
//checkCudaErrors( sdkStartTimer( &IntTime ));
//#endif
//
//		}
////<----------Interrupt handling-----------------------------
//
////-----------Termination detection------------------------->
//		checkCudaErrors( hipMemset( d_pivot, 1, sizeof(uint32_t) ));
//		CheckTerminateKernel<<<grid, threads>>>( d_m, d_pivot, RSize - 1 );
//		checkCudaErrors( hipMemcpy( &terminate, d_pivot, sizeof(uint32_t), hipMemcpyDeviceToHost ));
////<----------Termination detection--------------------------
//
//#ifdef _DEBUG
//hipDeviceSynchronize();
//( sdkStopTimer( &IntTime ));
//#endif
//
//	}
//	return result;
//}
