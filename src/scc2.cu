#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_timer.h>
#include<stack>
#include<set>
#include<map>
#include<queue>
#include<math.h>

#include "SCC2.h"
#include "scc_kernel.h"
#define BLOCKSIZE 512
#define sample_seeds 50
//#include "graph_generator.h"
//#include "parallel_fwd.h"
//#include "hash_table.h"

/******************/
/* PLAN STRUCTURE */
/******************/
// --- Async
template<class T>
struct plan {
	T *d_data;
	Boundary* b;
	Dimension*d;
	Point* s;
	Point* v;
	fEdge* eg;
	fFace* fc;
	uint32_t* Fe_Edge;
	uint32_t* Fr_Edge;
	uint32_t* Fe_Face;
	uint32_t* Fr_Face;
	//Stream for asynchronous command execution
	hipStream_t stream;
};

bool _DeviceSet1;

void
runTest(uint32_t* d_In, uint32_t* d_Out, uint32_t* h_out, int num_elements);
#ifdef _DEBUG
void bbin_printf(uint32_t elem, int N = 32, int end = 0)
{
	for ( int i = N - 1; i >= end; i-- )
	printf("%d", (bool)(elem & ((uint32_t)1 << i)));
}
#endif

int iDivUp1(int a, int b) // Round a / b to nearest higher integer value

		{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

/*********************/
/* SVD PLAN CREATION */
/*********************/
template<class T>
void createPlan(plan<T>& plan, unsigned int NperGPU, unsigned int NEdgeperGPU,
		unsigned int N, unsigned int gpuID) {

	// --- Device allocation
	(hipSetDevice(gpuID));
	checkCudaErrors(hipStreamCreate(&plan.stream));
	checkCudaErrors(
			hipMalloc(&(plan.d_data), NperGPU * sample_seeds* sizeof(T)));
	checkCudaErrors(hipMalloc(&(plan.v), N * sizeof(Point)));
	checkCudaErrors(
			hipMalloc(&(plan.Fe_Edge), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
	checkCudaErrors(
			hipMalloc(&(plan.Fr_Edge), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));

	checkCudaErrors(
			hipMalloc(&(plan.Fe_Face), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
	checkCudaErrors(
			hipMalloc(&(plan.Fr_Face), NEdgeperGPU * sample_seeds * sizeof(uint32_t)));

	checkCudaErrors(
			hipMalloc(&(plan.eg), NEdgeperGPU * sample_seeds * sizeof(fEdge)));
	checkCudaErrors(
			hipMalloc(&(plan.fc), NEdgeperGPU * sample_seeds * sizeof(fFace)));
	checkCudaErrors(hipMalloc(&(plan.b), 1 * sizeof(Boundary)));
	checkCudaErrors(hipMalloc(&(plan.d), 1 * sizeof(Dimension)));
	checkCudaErrors(hipMalloc(&(plan.s), 1 * sizeof(Point)));

}

//=======================================================================================

__device__ __host__ void error(int errorcode) {

	int temp = errorcode;
	printf("error!!\n");
}

__device__ __host__ void trilinearInterpolation(float p1[3], int idex,
		Boundary*b, Dimension* d, Point* step, float* m_x1, float* m_y1,
		float* m_z1, int primaryXDIMENSION, float& vx, float&vy, float& vz) {
	int xDim;
	int yDim;
	int zDim;

//	float vvx, vvy, vvz;
	if (d->x > primaryXDIMENSION) {
		int coef = primaryXDIMENSION / d->x;
		xDim = d->x / coef;
		yDim = d->y / coef;
		zDim = d->z / coef;

	} else if (d->x < primaryXDIMENSION) {
		int coef = primaryXDIMENSION / d->x;
		xDim = d->x * coef;
		yDim = d->y * coef;
		zDim = d->z * coef;

	} else {
		xDim = d->x;
		yDim = d->y;
		zDim = d->z;
	}

	float step_x = (b->high.x - b->low.x) / (xDim - 1);
	float step_y = (b->high.y - b->low.y) / (yDim - 1);
	float step_z = (b->high.z - b->low.z) / (zDim - 1);

	int kk = (p1[2] /*+ exp(-4.0)*/- b->low.z) / step_z;
	int jj = (p1[1] /*+ exp(-4.0)*/- b->low.y) / step_y;
	int ii = (p1[0] /*+ exp(-4.0)*/- b->low.x) / step_x;

	float p0[3];
	p0[0] = b->low.x + ii * step_x;
	p0[1] = b->low.y + jj * step_y;
	p0[2] = b->low.z + kk * step_z;

	double xd = (p1[0] - p0[0]) / step_x; //(p1[0]-step_x*int(p1[0]/step_x))/step_x;
	double yd = (p1[1] - p0[1]) / step_y; //(p1[1]-step_y*int(p1[1]/step_y))/step_y;
	double zd = (p1[2] - p0[2]) / step_z; //(p1[2]-step_z*int(p1[2]/step_z))/step_z;
	float v1[3];
	float v2[3];
	float v3[3];
	float v4[3];
	float v5[3];
	float v6[3];
	float v7[3];
	float v8[3];

	if (kk >= zDim - 1 || jj >= yDim - 1 || ii >= xDim - 1) {
		vx = m_x1[((kk * yDim + jj) * xDim + ii)];
		vy = m_y1[((kk * yDim + jj) * xDim + ii)];
		vz = m_z1[((kk * yDim + jj) * xDim + ii)];
		return;

	}
	if (xd == 0 && yd == 0 && zd == 0) {
		vx = m_x1[((kk * yDim + jj) * xDim + ii)];
		vy = m_y1[((kk * yDim + jj) * xDim + ii)];
		vz = m_z1[((kk * yDim + jj) * xDim + ii)];
		return;
	}

	{

		/*	int xDim = XDIMENSION;
		 int yDim = YDIMENSION;
		 int zDim = ZDIMENSION;*/
		v1[0] = m_x1[((kk * yDim + jj) * xDim + ii)];
		v1[1] = m_y1[((kk * yDim + jj) * xDim + ii)];
		v1[2] = m_z1[((kk * yDim + jj) * xDim + ii)];

		v2[0] = m_x1[((kk * yDim + jj) * xDim + ii + 1)];
		v2[1] = m_y1[((kk * yDim + jj) * xDim + ii + 1)];
		v2[2] = m_z1[((kk * yDim + jj) * xDim + ii + 1)];

		v3[0] = m_x1[((kk * yDim + (jj + 1)) * xDim + ii)];
		v3[1] = m_y1[((kk * yDim + (jj + 1)) * xDim + ii)];
		v3[2] = m_z1[((kk * yDim + (jj + 1)) * xDim + ii)];
		//

		v4[0] = m_x1[((kk * yDim + (jj + 1)) * xDim + ii + 1)];
		v4[1] = m_y1[((kk * yDim + (jj + 1)) * xDim + ii + 1)];
		v4[2] = m_z1[((kk * yDim + (jj + 1)) * xDim + ii + 1)];

		//int idx2 = (k + 1)*(xDim*yDim) + (j + 1)*yDim + i + 1;

		v5[0] = m_x1[(((kk + 1) * yDim + jj) * xDim + ii)];
		v5[1] = m_y1[(((kk + 1) * yDim + jj) * xDim + ii)];
		v5[2] = m_z1[(((kk + 1) * yDim + jj) * xDim + ii)];

		v6[0] = m_x1[(((kk + 1) * yDim + jj) * xDim + ii + 1)];
		v6[1] = m_y1[(((kk + 1) * yDim + jj) * xDim + ii + 1)];
		v6[2] = m_z1[(((kk + 1) * yDim + jj) * xDim + ii + 1)];

		v7[0] = m_x1[(((kk + 1) * yDim + (jj + 1)) * xDim + ii)];
		v7[1] = m_y1[(((kk + 1) * yDim + (jj + 1)) * xDim + ii)];
		v7[2] = m_z1[(((kk + 1) * yDim + (jj + 1)) * xDim + ii)];

		v8[0] = m_x1[(((kk + 1) * yDim + (jj + 1)) * xDim + ii + 1)];
		v8[1] = m_y1[(((kk + 1) * yDim + (jj + 1)) * xDim + ii + 1)];
		v8[2] = m_z1[(((kk + 1) * yDim + (jj + 1)) * xDim + ii + 1)];
	}

	double c00 = v1[0] * (1 - xd) + v2[0] * xd;
	double c10 = v3[0] * (1 - xd) + v4[0] * xd;
	double c01 = v5[0] * (1 - xd) + v6[0] * xd;
	double c11 = v7[0] * (1 - xd) + v8[0] * xd;

	double c0 = c00 * (1 - yd) + c10 * yd;
	double c1 = c01 * (1 - yd) + c11 * yd;

	vx = c0 * (1 - zd) + c1 * zd;

	c00 = v1[1] * (1 - xd) + v2[1] * xd;
	c10 = v3[1] * (1 - xd) + v4[1] * xd;
	c01 = v5[1] * (1 - xd) + v6[1] * xd;
	c11 = v7[1] * (1 - xd) + v8[1] * xd;

	c0 = c00 * (1 - yd) + c10 * yd;
	c1 = c01 * (1 - yd) + c11 * yd;

	vy = c0 * (1 - zd) + c1 * zd;

	c00 = v1[2] * (1 - xd) + v2[2] * xd;
	c10 = v3[2] * (1 - xd) + v4[2] * xd;
	c01 = v5[2] * (1 - xd) + v6[2] * xd;
	c11 = v7[2] * (1 - xd) + v8[2] * xd;

	c0 = c00 * (1 - yd) + c10 * yd;
	c1 = c01 * (1 - yd) + c11 * yd;

	vz = c0 * (1 - zd) + c1 * zd;

	int cxz = 0;

//	getLorenzField1(p1, vvx, vvy, vvz);
//	if (vvx != vx || vvy != vy || vvz != vz) {
//		vx = vvx;
//		vy = vvy;
//		vz = vvz;
//	}
	return;

}

__device__ __host__ void generalstreamlineTracing_single(float p[3],
		bool bForward, float e[3], float* m_x1, float* m_y1, float* m_z1,
		Boundary*b, Dimension* d, Point* step, int currentDimX, int tau) {

	int start_pixel_id = 0;

	float i_kk = ((p[2] - b->low.z) / step->z);
	float i_jj = ((p[1] - b->low.y) / step->y);
	float i_ii = ((p[0] - b->low.x) / step->x);

	float vx, vy, vz;
	float next_i, next_j, next_k;
	float ii, jj, kk;
	start_pixel_id = i_kk * (d->x * d->y) + i_jj * d->x + i_ii;

//for (int j = 0;j < ndim*ndim*ndim;j++)

	float p2[3];

	next_i = p[0];					//samples_x[j];
	next_j = p[1];					//samples_y[j];
	next_k = p[2];					//samples_z[j];
	for (int k = 0; k < tau; k++) {

		p2[0] = next_i;
		p2[1] = next_j;
		p2[2] = next_k;
		/*	if (m_bTornadoFieldSelected)
		 trilinearInterpolation2(next_i,next_j,next_k, start_pixel_id, vx, vy, vz);

		 else*/
		trilinearInterpolation(p2, start_pixel_id, b, d, step, m_x1, m_y1, m_z1,
				currentDimX, vx, vy, vz);

		//get_ABC_flow(next_i,next_j,next_k,vx,vy,vz);
		/*	if (WhichType == 0)

		 else if (WhichType == 1)
		 get_ABC_flow(next_i, next_j, next_k, vx, vy, vz);*/
		//	get_Lorenz_Field(next_i, next_j, next_k, vx, vy, vz);
		//getLorenzField1(p2, vx, vy, vz);
		float dist = sqrt(vx * vx + vy * vy + vz * vz);
		if (dist < 1.0e-6 || next_i < b->low.x || next_i > b->high.x
				|| next_j < b->low.y || next_j > b->high.y || next_k < b->low.z
				|| next_k > b->high.z) {
			//cout<<"ddd"<<endl;
			break;
		}

		vx = (vx / dist) * (step->x / 4.0);
		vy = (vy / dist) * (step->y / 4.0);
		vz = (vz / dist) * (step->z / 4.0);

		ii = next_i;
		jj = next_j;
		kk = next_k;

		if (bForward) {
			next_i = ii + vx; //RK4
			next_j = jj + vy;
			next_k = kk + vz;
		} else {
			next_i = ii - vx; //RK4
			next_j = jj - vy;
			next_k = kk - vz;
		}

		i_kk = ((next_k - b->low.z) / step->z);
		i_jj = ((next_j - b->low.y) / step->y);
		i_ii = ((next_i - b->low.x) / step->x);

		if (i_ii >= 0 && i_jj >= 0 && i_kk >= 0) {
			// 			next_i = lowBoundary+((floor(i_ii+0.5))*step) /*+ (0.5*step)*/;
			// 			next_j = lowBoundary+((floor(i_jj+0.5))*step) /*+ (0.5*step)*/;
			// 			next_k = lowBoundary_z+((floor(i_kk+0.5))*step) /*+ (0.5*step)*/;
		} else
			break;

	}

	e[0] = next_i;
	e[1] = next_j;
	e[2] = next_k;

	return;

}

__global__ void Tracing(ASF_vertex * m, float* m_x1, float* m_y1, float* m_z1,
		Dimension* d, Boundary* b, Point* step, bool bForward,
		uint32_t whichData, int currentXDim, uint32_t start_rows,
		uint32_t num_rows, uint32_t level, int currtau)

		{
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row > num_rows)
		return;

	ASF_vertex vertex = m[row];

	if (row == 8701)
		vertex = m[row];
	if (!vertex.checkInBoundary(b) || !vertex.checkInBoundary_StartPoint(b)) {
		vertex.unsetInBoundary();
		m[row] = vertex;
		return;
	}

	float p1[3];
	if (vertex.type == 1) {

		p1[0] = vertex.e.x;
		p1[1] = vertex.e.y;
		p1[2] = vertex.e.z;

		float ep[3];
		generalstreamlineTracing_single(p1, bForward, ep, m_x1, m_y1, m_z1, b,
				d, step, currentXDim, 1);

		vertex.e.x = ep[0];
		vertex.e.y = ep[1];
		vertex.e.z = ep[2];
		//
		//			if (Trace[row][currtau + 10].x != vertex.e.x
		//					|| Trace[row][currtau + 10].y != vertex.e.y
		//					|| Trace[row][currtau + 10].z != vertex.e.z)
		//				vertex.e.x = ep[0];
		//Trace[row][currtau] = vertex.e;
		//vertex.es[currtau] = vertex.e;

	} else if (vertex.type == 2) {
		if (!m[vertex.left].checkInBoundary(b)
				|| !m[vertex.right].checkInBoundary(b)
				|| !m[vertex.left].checkInBoundary_StartPoint(b)
				|| !m[vertex.right].checkInBoundary_StartPoint(b)) {
			vertex.unsetInBoundary();
			m[row] = vertex;
			return;
		}

		p1[0] = vertex.p.x;
		p1[1] = vertex.p.y;
		p1[2] = vertex.p.z;
		float ep[3];
		generalstreamlineTracing_single(p1, bForward, ep, m_x1, m_y1, m_z1, b,
				d, step, currentXDim, currtau + 1);

		Point e1 = m[vertex.left].e;
		Point e2 = m[vertex.right].e;
		vertex.e.x = (e1.x + e2.x) / 2;
		vertex.e.y = (e1.y + e2.y) / 2;
		vertex.e.z = (e1.z + e2.z) / 2;
		//Trace[row][currtau] = vertex.e;

		//vertex.es[currtau] = vertex.e;
		//Trace[row][currtau] = vertex.e;
	} else if (vertex.type == 3) {

		if (!m[vertex.left].checkInBoundary(b)
				|| !m[vertex.right].checkInBoundary(b)
				|| !m[vertex.up].checkInBoundary(b)
				|| !m[vertex.down].checkInBoundary(b)
				|| !m[vertex.left].checkInBoundary_StartPoint(b)
				|| !m[vertex.right].checkInBoundary_StartPoint(b)
				|| !m[vertex.up].checkInBoundary_StartPoint(b)
				|| !m[vertex.down].checkInBoundary_StartPoint(b)) {

			vertex.unsetInBoundary();
			m[row] = vertex;
			return;
		}
		Point e1 = m[vertex.left].e;
		Point e2 = m[vertex.right].e;
		Point e3 = m[vertex.up].e;
		Point e4 = m[vertex.down].e;
		vertex.e.x = (e1.x + e2.x + e3.x + e4.x) / 4;
		vertex.e.y = (e1.y + e2.y + e3.y + e4.y) / 4;
		vertex.e.z = (e1.z + e2.z + e3.z + e4.z) / 4;
		//	Trace[row][currtau] = vertex.e;
		//vertex.es[currtau] = vertex.e;
		//Trace[row][currtau] = vertex.e;
	}

	{
		if (vertex.checkInBoundary(b)) {		//}(vertex.e, b, currtau)) {
			int io, jo, ko;
			vertex.getIndex(b, step, d, io, jo, ko);
			uint32_t range = io + jo * d->x + ko * (d->x * d->y);//_tv.getRange(&_b, &step, &_d);

			vertex.setRange(range);
			//				uint32_t range = vertex.getRange(b, step, d);
			//
			//				vertex.setRange(range);

		} else {
			vertex.unsetInBoundary();
		}
	}

	m[row] = vertex;
	return;
}

__device__ __host__ bool checkEdge(ASF_vertex vertex1, ASF_vertex vertex2,
		Boundary*b, Dimension* d, Point* step, bool bForward, int tau, int v1i,
		int v2i) {
//	int range1 = .getRange_tau(&Trace[v1i][tau], b, step, d, tau);
//	int range2 = vertex2.getRange_tau(&Trace[v2i][tau], b, step, d, tau);

	int range1 = vertex1.getRange(b, step, d);
	int range2 = vertex2.getRange(b, step, d);
	/*if (!bForward)
	 {
	 range1 = vertex1.getRangeBackward();
	 range2 = vertex2.getRangeBackward();
	 }*/
	int iz1 = range1 / (d->x * d->y);
	int iy1 = (range1 - iz1 * (d->x * d->y)) / d->x;
	int ix1 = (range1 - iz1 * (d->x * d->y)) % d->x;

	int iz2 = range2 / (d->x * d->y);
	int iy2 = (range2 - iz2 * (d->x * d->y)) / d->x;
	int ix2 = (range2 - iz2 * (d->x * d->y)) % d->x;

//	vertex1.getIndex(b, step, d, ix1, iy1, iz1);
//	vertex2.getIndex(b, step, d, ix2, iy2, iz2);
	float dist = 0.;
	dist = sqrt(
			(float) ((iz2 - iz1) * (iz2 - iz1) + (iy2 - iy1) * (iy2 - iy1)
					+ (ix2 - ix1) * (ix2 - ix1)));

	double distance = vertex1.e.dist(vertex2.e);
//	if (distance >= 2 * step->x)
//		return false;
//	return true;
	/*if (iz1 == d->z || iz1 == d->z - 1 || iy1 == d->y || iy1 == d->y - 1 || ix1 == d->x || ix1 == d->x - 1)
	 printf("");*/

	/*if (iz2 == d->z || iz2 == d->z - 1 || iy2 == d->y || iy2 == d->y - 1 || ix2 == d->x || ix2 == d->x - 1)
	 printf("");*/
	float realdist = sqrt(
			(vertex1.e.x - vertex2.e.x) * (vertex1.e.x - vertex2.e.x)
					+ (vertex1.e.y - vertex2.e.y) * (vertex1.e.y - vertex2.e.y)
					+ (vertex1.e.z - vertex2.e.z)
							* (vertex1.e.z - vertex2.e.z));
	if (realdist > 4)
		printf("");
	if (realdist <= step->x
			|| (abs(iz1 - iz2) <= 1 && abs(iy1 - iy2) <= 1 && ix1 == ix2)
			|| (abs(iy1 - iy2) <= 1 && abs(ix1 - ix2) <= 1 && iz1 == iz2)
			|| (abs(iz1 - iz2) <= 1 && abs(ix1 - ix2) <= 1 && iy1 == iy2)) //
		//if(dist <= 2.0 || abs(range1 - range2) == (2 * d->x + 1) || abs(range1 - range2) == (2 * d->x*d->y + 1) || abs(range1 - range2) == (2 * d->x*d->y + d->x) || abs(range1 - range2) == (2 * d->x*d->y - d->x) || abs(range1 - range2) == (1 * d->x*d->y + d->x) || abs(range1 - range2) == (1 * d->x*d->y - d->x))//&& realdist <= 2.0)//|| (abs(iz1 - iz2) <= 2 && abs(iy2 - iy1) <= 2 && abs(ix2 - ix1) <= 2))

		return true;

	return false;

}

__device__ __host__ bool DivideEdges(ASF_vertex* v1, ASF_vertex* v2,
		uint32_t range1, Boundary* _b, Dimension* _d, Point* step, float*m_x1,
		float* m_y1, float*m_z1, int currentXDim, uint32_t faceNum,
		bool bForward, int divider, int tau, ASF_vertex* otv, int curvertexid,
		int v1i, int v2i) {

	Point p_left_c;
	int n = divider;
	//ASF_vertex* _tv = new ASF_vertex[n];			// = new ASF_vertex[n-1];
	ASF_vertex vcenter;
	vcenter.type = 1;
	double s = 1.0 / n;
	for (int i = 1; i < n; i++) {

		double t = i * s;
		vcenter.p.x = (t) * v1->p.x + (1 - t) * v2->p.x;
		vcenter.p.y = (t) * v1->p.y + (1 - t) * v2->p.y;
		vcenter.p.z = (t) * v1->p.z + (1 - t) * v2->p.z;

		vcenter.e.x = (t) * v1->e.x + (1 - t) * v2->e.x;
		vcenter.e.y = (t) * v1->e.y + (1 - t) * v2->e.y;
		vcenter.e.z = (t) * v1->e.z + (1 - t) * v2->e.z;

		//if (vcenter.type == 2)
		{
//			for (int j = 0; j < tau + 1; j++) {
//				Trace[curvertexid][j].x = (t * Trace[v1i][j].x)
//						+ (1 - t) * Trace[v2i][j].x;
//				Trace[curvertexid][j].y = (t * Trace[v1i][j].y)
//						+ (1 - t) * Trace[v2i][j].y;
//				Trace[curvertexid][j].z = (t * Trace[v1i][j].z)
//						+ (1 - t) * Trace[v2i][j].z;
//
//			}
		}
		//	vcenter.e = Trace[curvertexid][tau];

		//Trace[curvertexid][tau] = vcenter.e;
		//		AdvectParticle_estimated(&vcenter, m_x1, m_y1, m_z1, _b, _d, step,
		//				currentXDim, tau + 1, bForward);

		if (!vcenter.checkInBoundary(_b)) {
			*otv = vcenter;
			//return false;
			error(0);
			return false;
		}
		uint32_t oldrange = v1->getOldRange();// _tv.getOldRange(&_b, &step, &_d);
		vcenter.setOldRange(oldrange);
		vcenter.setInBoundary();

		int io, jo, ko;
		io = (vcenter.e.x - _b->low.x) / (step->x);
		io = (vcenter.e.y - _b->low.y) / (step->y);
		io = (vcenter.e.z - _b->low.z) / (step->z);
		vcenter.getIndex(_b, step, _d, io, jo, ko);
		uint32_t range = io + jo * _d->x + ko * (_d->x * _d->y);//_tv.getRange(&_b, &step, &_d);

		//vcenter.setRange(range);
		vcenter.range = range;
		//printf("vcenter: %d range:%d",vcenter.range,range);
		//if(v1->getRange() == 4430 )
		if (checkEdge(vcenter, *v1, _b, _d, step, bForward, tau, v1i, v2i)
				&& checkEdge(vcenter, *v2, _b, _d, step, bForward, tau, v1i,
						v2i)) {
			*otv = vcenter;

			return true;
		}

	}

	if (n == 5) {
		*otv = vcenter;
	}
	*otv = vcenter;
	return false;

}

__device__ __host__ void Split_One_Edge(ASF_vertex*m, fFace* fc, fEdge* eg,
		uint32_t* Fe_Edge, uint32_t* Fr_Edge, Dimension* d, Boundary* b,
		Point* step, float*m_x1, float* m_y1, float*m_z1, int currentXDim,
		int tau, bool bForward, uint32_t num_vertex, uint32_t num_edges,
		uint32_t row) {

	int curVertexId = num_vertex + Fr_Edge[row];
	int curEdgeId = num_edges + Fr_Edge[row] * 2;

	fEdge edge1 = eg[row];
	fEdge edge2 = eg[row];

	int v1i = edge1.v1;
	int v2i = edge1.v2;

	if (edge1.bsplit)
		error(2);

	ASF_vertex vertex1 = m[edge1.v1];
	ASF_vertex vertex2 = m[edge1.v2];

//		if(curVertexId == 5846 )
//				error(1);

	if (!vertex1.checkInBoundary_StartPoint(b)
			|| !vertex2.checkInBoundary_StartPoint(b)) {
		eg[row].unsetInBoundary();
		return;
	}

	if ((vertex1.p.x - vertex2.p.x) > step->x
			|| (vertex1.p.y - vertex2.p.y) > step->y
			|| (vertex1.p.z - vertex2.p.z) > step->z)
		error(0);
	if (curVertexId == 61996)
		Fe_Edge[row] = 1;

	ASF_vertex allseeds[5];
	ASF_vertex vedge;

	bool bFound = false;
	int n = 2;
	//for (n = 2; n <= 5; n++)
	{
		//int n = 10;
		//ASF_vertex* vedge_array = new ASF_vertex[n];
		DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), b, d, step, m_x1,
				m_y1, m_z1, currentXDim, 0, bForward, n, tau, &vedge,
				curVertexId, v1i, v2i);
		{
			bFound = true;
			//allseeds[n - 2] = vedge;
			//			if (tau == 50)
			//				error(1);
//			ASF_vertex* tempp = new ASF_vertex[3];
//			tempp[0] = vertex1;
//			tempp[1] = vertex2;
//			tempp[2] = vedge;
//			Point temppoint[3];
//			temppoint[0] = vertex1.p;
//			temppoint[1] = vertex2.p;
//			temppoint[2] = vedge.p;
//
//			int ii = (vertex1.e.x - b->low.x) / step->x;
//			int jj = (vertex1.e.y - b->low.y) / step->y;
//			int kk = (vertex1.e.z - b->low.z) / step->z;
//
//			int ii1 = (vertex2.e.x - b->low.x) / step->x;
//			int jj1 = (vertex2.e.y - b->low.y) / step->y;
//			int kk1 = (vertex2.e.z - b->low.z) / step->z;
//			int intarray[3];
////			intarray[0] = vertex1.getRange_tau(&Trace[v1i][tau], b, step, d,
////					tau);
////			intarray[1] = vertex2.getRange_tau(&Trace[v2i][tau], b, step, d,
////					tau);
////			intarray[2] = vedge.getRange_tau(&Trace[curVertexId][tau], b, step,
////					d, tau);
//
//			intarray[0] = vertex1.getRange(b, step, d);
//			intarray[1] = vertex2.getRange(b, step, d);
//			intarray[2] = vedge.getRange(b, step, d);
//			//
//			m[curVertexId] = vedge;
//			if (false && tau == 90) {
//
//				ASF_vertex tempm[3];
//				tempm[0] = vertex1;
//				tempm[1] = vertex2;
//				tempm[2] = vedge;
//				int indexarray[3];
//				indexarray[0] = v1i;
//				indexarray[1] = v2i;
//				indexarray[2] = curVertexId;
////				fs->Save_Streamlines_estimated(m, Trace, indexarray, bForward,
////						"st1", m_x1, m_y1, m_z1, b, d, step, currentXDim,
////						tau + 1, 3);
//
//				fs->Save_Streamlines_EndAdvection(m, indexarray, bForward,
//						"st1", m_x1, m_y1, m_z1, b, d, step, currentXDim,
//						tau + 1, 3);
//				fs->save_Quad_FaceWithSeedPoits(m, fc, tempp, eg, Trace, *d,
//						"face85", num_vertex, 1, 3, tau, 1);
//				if (tau == 99)
//					fs->save_Voxel(m, fc, intarray, eg, *d, "face85",
//							num_vertex, 3, tau, 1, 10);
//			}

			//break;
		}

		//allseeds[n - 2] = vedge;
	}

	//	string dataname = "edge";
	//	fs->start_save_Quad_One_Face(m, fc[edge1.E2F[0]], eg, *d, dataname,
	//			num_vertex, 1, tau, 1);
	//
	//	generate_streamlines_sparsely(allseeds, bForward, 1, m_x1, m_y1, m_z1, b, d,
	//			step, currentXDim, tau + 2, n - 1);
	//
	//	fs->save_Quad_FaceWithSeedPoits(m, fc, allseeds, eg, *d, dataname,
	//			num_vertex, 1, 1, tau, 1);
	//
	//	fs->start_save_Quad_FaceWithSeedPoits(m, fc, allseeds, eg, *d, dataname,
	//				num_vertex, 1, 1, tau, 1);

	//	if (!bFound) {
	//		DivideEdges(&vertex1, &vertex2, vertex1.getOldRange(), b, d, step, m_x1,
	//				m_y1, m_z1, currentXDim, 0, bForward, 2, tau, &vedge);
	//
	//	}
	vedge.type = 2;
	vedge.oldrange = vertex1.oldrange;
	vedge.left = edge1.v1;
	vedge.right = edge1.v2;

	edge1.v2 = curVertexId;
	edge2.v1 = curVertexId;
	//edge1.bsplit = true;
	edge1.next = curEdgeId;
	edge1.level = edge1.level + 1;
	edge2.level = edge1.level;
	edge2.Prev = row;
	vedge.level = edge2.level;

	//vedge.fxy = 0;
	//vedge.fyz = 0;
	//vedge.fxz = 0;

	float rgb[3];
	rgb[0] = 0.;
	rgb[1] = 0.;
	rgb[2] = 0.;

	eg[row].bsplit = true;
	eg[row].subedge[0] = curEdgeId;
	eg[row].subedge[1] = curEdgeId + 1;

	edge1.bsplit = false;
	edge2.bsplit = false;
	edge1.parent = row;
	edge2.parent = row;

	float dist1 = vertex1.e.dist(vertex2.e);
	float dist1_1 = vertex1.e.dist(vedge.e);
	float dist1_2 = vertex2.e.dist(vedge.e);

	if (dist1_2 >= dist1 || dist1_2 >= dist1) {
		error(1);
		ASF_vertex oa[3];
		oa[0] = vertex1;
		oa[1] = vertex2;
		oa[2] = vedge;

//		generate_streamlines_sparsely(oa, bForward, 1, m_x1, m_y1, m_z1, b, d,
//				step, currentXDim, 100 + 2, 3);
//
//		fs->save_Quad_FaceWithSeedPoits(m, fc, oa, eg, Trace, *d, "face85",
//				num_vertex, 1, 3, tau, 1);
		//error(1);
	}

	eg[curEdgeId] = edge1;

	eg[curEdgeId + 1] = edge2;

	m[curVertexId] = vedge;

}

__global__ void CheckNeighborhood(ASF_vertex*m, fEdge* eg, uint32_t*Fr,
		Dimension* d, Boundary* b, Point* step, bool bForward,
		uint32_t original_num_rows, uint32_t num_rows, uint32_t level,
		int tau) {
	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

//uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	for (uint32_t row = 0; row < num_rows; row++)

	if (row >= num_rows)
		return;
	Fr[row] = 1;
	return;
	int co = 0;
	if (row == 11705)
		co = 0;

	fEdge edge = eg[row];
	ASF_vertex vertex1 = m[edge.v1];
	ASF_vertex vertex2 = m[edge.v2];
	//printf("---- %f%f%f \n", vertex1.p.x, vertex1.p.y, vertex1.p.z);

//	if (vertex1.e.dist(vertex2.e) > step->x) {
//		co = 0;
//	}
	if (!edge.isInBoundary())
		return;
	if (edge.bsplit)
		return;
	//	edge.bsplit = false;

	if (!vertex1.isInBoundary() || !vertex2.isInBoundary()) {
		//eg[row].unsetInBoundary();
		edge.unsetInBoundary();
		eg[row] = edge;
		return;
	}

	if (!vertex1.checkInBoundary_StartPoint(b)
			|| !vertex2.checkInBoundary_StartPoint(b)) {
		//	printf("%f%f%f \n", vertex1.p.x, vertex1.p.y, vertex1.p.z);
		//eg[row].unsetInBoundary();
		edge.unsetInBoundary();
		eg[row] = edge;

		return;
	}

	if (!vertex1.checkInBoundary(b) || !vertex2.checkInBoundary(b)) {
		edge.unsetInBoundary();
		eg[row] = edge;

		return;

	}

	if (abs((int) vertex1.getOldRange() - (int) vertex2.getOldRange()) > 1
			&& abs((int) vertex1.getOldRange() - (int) vertex2.getOldRange())
					!= d->x
			&& abs((int) vertex1.getOldRange() - (int) vertex2.getOldRange())
					!= (d->x * d->y))
		error(0);

	/*	if (vertex1.getOldRange() % (d->x - 1) == 0 && (vertex2.getOldRange() % d->x) == 0)
	 continue;
	 if (vertex1.getOldRange() % (d->x*d->y - 1 )==0 && (vertex2.getOldRange() % d->x*d->y) == 0)
	 continue;*/

	float dist = sqrt(
			(vertex1.e.x - vertex2.e.x) * (vertex1.e.x - vertex2.e.x)
					+ (vertex1.e.y - vertex2.e.y) * (vertex1.e.y - vertex2.e.y)
					+ (vertex1.e.z - vertex2.e.z)
							* (vertex1.e.z - vertex2.e.z));

	if (fabs(vertex1.p.x - vertex2.p.x) > step->x + exp(-6.0)
			|| fabs(vertex1.p.y - vertex2.p.y) > step->y + exp(-6.0)
			|| fabs(vertex1.p.z - vertex2.p.z) > step->z + exp(-6.0))
		error(1);
	if ((row == 493 || row == 518 || row == 685 || row == 494))
		Fr[row] = 0;
	//if (dist >  step->x)// || !checkEdge(vertex1, vertex2, d, bForward))

	if (!checkEdge(vertex1, vertex2, b, d, step, bForward, tau, edge.v1,
			edge.v2)) {
		//vertex1.setInNextLevel_xy();
		//	edge.bsplit = true;

		Fr[row] = 1;
	}

	//			continue;

	eg[row] = edge;

}

__global__ void CheckFace(ASF_vertex*m, fEdge* eg, fFace* fc, uint32_t*Fe_Edge,
		uint32_t* Fe_Face, Boundary* b, Dimension* d, Point* step,
		bool bForward, uint32_t num_face, int tau) {

	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= num_face)
		return;
	fFace face = fc[row];
	if (row == 442)
		face = fc[row];

	if (face.bsplit) {
		face = fc[row];
		return;
	}

	if (!face.isInBoundary()) {
		return;
	}

	if (!m[face.vertexes[0]].isInBoundary()
			|| !m[face.vertexes[1]].isInBoundary()
			|| !m[face.vertexes[2]].isInBoundary()
			|| !m[face.vertexes[3]].isInBoundary()) {
		fc[row].unsetInBoundary();
		return;

	}

	if (!m[face.vertexes[0]].checkInBoundary_StartPoint(b)
			|| !m[face.vertexes[1]].checkInBoundary_StartPoint(b)
			|| !m[face.vertexes[2]].checkInBoundary_StartPoint(b)
			|| !m[face.vertexes[3]].checkInBoundary_StartPoint(b)) {
		fc[row].unsetInBoundary();
		return;

	}

	int v1i = face.vertexes[0];
	int v2i = face.vertexes[1];
	int v3i = face.vertexes[2];
	int v4i = face.vertexes[3];

	if (!m[face.vertexes[0]].checkInBoundary(b)
			|| !m[face.vertexes[1]].checkInBoundary(b)
			|| !m[face.vertexes[2]].checkInBoundary(b)
			|| !m[face.vertexes[3]].checkInBoundary(b)) {
		fc[row].unsetInBoundary();
		return;

	}

	ASF_vertex v1 = m[face.vertexes[0]];
	ASF_vertex v2 = m[face.vertexes[1]];
	ASF_vertex v3 = m[face.vertexes[2]];
	ASF_vertex v4 = m[face.vertexes[3]];

	if (!checkEdge(v1, v2, b, d, step, bForward, tau, v1i, v2i)
			|| !checkEdge(v2, v3, b, d, step, bForward, tau, v2i, v3i)
			|| !checkEdge(v3, v4, b, d, step, bForward, tau, v3i, v4i)
			|| !checkEdge(v1, v4, b, d, step, bForward, tau, v1i, v4i))

		Fe_Face[row] = 1;

	fc[row] = face;

}

__global__ void EdgeReduction(fEdge*eg, ASF_vertex*m, fFace* Ff,
		uint32_t* Fe_Face, uint32_t* Fe_Edge, uint32_t num_edges) {

	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;

	if (row >= num_edges)
		return;
}

__global__ void SplitEdge(ASF_vertex*m, fFace* fc, fEdge* eg, uint32_t* Fe_Edge,
		uint32_t* Fr_Edge, Dimension* d, Boundary* b, Point* step, float*m_x1,
		float* m_y1, float*m_z1, int currentXDim, bool bForward,
		uint32_t num_vertex, uint32_t num_edges, uint32_t level, int tau) {

	uint32_t row = blockIdx.x * blockDim.x + threadIdx.x;
//	for (uint32_t row = 0; row < num_rows; row++)

	if (row > num_edges)
		return;

	if (Fe_Edge[row] == 1) {

//			if (row == 3272)
//				error(0);
//				Split_One_Edge(m, fc, eg, Fe_Edge, Fr_Edge, d, b, step, m_x1, m_y1,
//						m_z1, currentXDim, tau, bForward, num_vertex, num_edges,
//						row);

		Split_One_Edge(m, fc, eg, Fe_Edge, Fr_Edge, d, b, step, m_x1, m_y1,
				m_z1, currentXDim, tau, bForward, num_vertex, num_edges, row);
	}

}

pair<uint32_t, float> OBF_Decomposition2(ASF_vertex* _a, fFace *fc, fEdge*eg,
		Boundary* _b, Dimension* _d, Point* step, int _tau, float* m_x1,
		float* m_y1, float* m_z1, ASF_vertex** oVertex, Edge ** oFc,
		uint32_t ** oFr, fFace** oFace, fEdge** oEdge, uint32_t * oRSize,
		uint32_t * oFaceSize, uint32_t * oEdgeSize, uint32_t whichData,
		int CSize, uint32_t _curEdgeSize, uint32_t _curFaceSize,
		int curXDimension, int sampleSeeds, int MorseLevel, bool bForward) {

//-----------CPU initialization---------------------------->

	uint32_t * To, *From;
	uint32_t * To2;

//	Trace = new Point*[CSize * sampleSeeds];

//	for (int j = 0; j < CSize * sampleSeeds; j++) {
//		//if(_a[j].isInBoundary())
//		Trace[j] = new Point[_tau + 1];
//	}

	printf("Vertices: %u\n", CSize - 2);

	uint32_t currTau = _tau;

	uint32_t currSize = CSize;
	uint32_t icycle = currTau;

	uint32_t curEdgeSize = _curEdgeSize;
	uint32_t curFaceSize = _curFaceSize;

	uint32_t* Fe_Edge = new uint32_t[curEdgeSize * sampleSeeds];
	uint32_t* Fr_Edge = new uint32_t[curEdgeSize * sampleSeeds];

	uint32_t* Fe_Edge2 = new uint32_t[curEdgeSize * sampleSeeds];
	uint32_t* Fr_Edge2 = new uint32_t[curEdgeSize * sampleSeeds];

	uint32_t* Fe_Face = new uint32_t[curFaceSize * sampleSeeds];
	uint32_t* Fr_Face = new uint32_t[curFaceSize * sampleSeeds];

	ASF_vertex* currA = new ASF_vertex[currSize * sampleSeeds];

	memcpy(currA, _a, currSize * sizeof(ASF_vertex));

	uint32_t oldsize = CSize;
	int level = 0;

	memset(Fe_Edge, 0, curEdgeSize * sampleSeeds * sizeof(uint32_t));
	memset(Fr_Edge, 0, curEdgeSize * sampleSeeds * sizeof(uint32_t));

	memset(Fe_Edge2, 0, curEdgeSize * sampleSeeds * sizeof(uint32_t));
	memset(Fr_Edge2, 0, curEdgeSize * sampleSeeds * sizeof(uint32_t));

	memset(Fe_Face, 0, curFaceSize * sampleSeeds * sizeof(uint32_t));
	memset(Fr_Face, 0, curFaceSize * sampleSeeds * sizeof(uint32_t));

	int interval = 1;
	int oldFaceSize = curFaceSize;
	int co = 0;

	int c = currSize;
	int nf = curFaceSize;

//================================================================

//-----------GPU initialization---------------------------->

	if (!_DeviceSet1) {
		_DeviceSet1 = true;
		checkCudaErrors(hipSetDevice(2));
	}
	Dimension* d_d;
	Boundary* d_b;
	Point* d_step;
	float* d_vx;
	float* d_vy;
	float* d_vz;
	ASF_vertex* d_currA;
	uint32_t* d_Fe_Edge;
	uint32_t* d_Fe_Face;
	uint32_t* d_Fr_Edge;
	uint32_t* d_Fr_Face;
	fEdge* d_eg;
	fFace* d_fc;

	hipError_t e1, e2, e3, e4, e5, e6, e7;
	checkCudaErrors(
			e1 = hipMalloc((void** )&d_fc,
					curFaceSize * sampleSeeds * sizeof(fFace)));
	checkCudaErrors(
			e2 = hipMalloc((void** )&d_eg,
					curEdgeSize * sampleSeeds * sizeof(fEdge)));

	checkCudaErrors(
			e5 = hipMalloc((void** )&d_currA,
					currSize * sampleSeeds * sizeof(ASF_vertex)));
	checkCudaErrors(e3 = hipMalloc((void** )&d_d, sizeof(Dimension)));
	checkCudaErrors(e4 = hipMalloc((void** )&d_b, sizeof(Boundary)));
	checkCudaErrors(e4 = hipMalloc((void** )&d_step, sizeof(Point)));

	checkCudaErrors(
			e6 = hipMalloc((void** )&d_Fe_Edge,
					curEdgeSize * sampleSeeds * sizeof(uint32_t)));
	checkCudaErrors(
			e6 = hipMalloc((void** )&d_Fr_Edge,
					curEdgeSize * sampleSeeds * sizeof(uint32_t)));

	checkCudaErrors(
			e7 = hipMalloc((void** )&d_Fe_Face,
					curFaceSize * sampleSeeds * sizeof(uint32_t)));
	checkCudaErrors(
			e7 = hipMalloc((void** )&d_Fr_Face,
					curFaceSize * sampleSeeds * sizeof(uint32_t)));

	checkCudaErrors(
			e6 = hipMalloc((void** )&d_vx,
					currSize  * sizeof(float)));
	checkCudaErrors(
			e6 = hipMalloc((void** )&d_vy,
					curEdgeSize  * sizeof(float)));

	checkCudaErrors(
			e7 = hipMalloc((void** )&d_vz,
					curFaceSize * sizeof(float)));


	if (e1 == hipErrorOutOfMemory || e2 == hipErrorOutOfMemory
			|| e3 == hipErrorOutOfMemory
			|| e4 == hipErrorOutOfMemory
			|| e5 == hipErrorOutOfMemory
			|| e6 == hipErrorOutOfMemory
			|| e7 == hipErrorOutOfMemory) {
		throw "Error: Not enough memory on GPU\n";
	}
//===============================================================
//	(hipHostMalloc(&inputMatrices, N * sample_seeds * sizeof(ASF_vertex)));
//	fEdge *inputEdges;
//	(hipHostMalloc(&inputEdges, NEdgeperGPU * sample_seeds * sizeof(fEdge)));
//	uint32_t *inputFe_Edge;
//	(hipHostMalloc(&inputFe_Edge,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
//	uint32_t *inputFr_Edge;
//	(hipHostMalloc(&inputFr_Edge,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
//
//	uint32_t *inputFe_Face;
//	(hipHostMalloc(&inputFe_Face,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
//	uint32_t *inputFr_Face;
//	(hipHostMalloc(&inputFr_Face,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));

//================================================================

	dim3 grid(((CSize + 510) / 512), 1, 1); // (RSize-2) valid vertecis the first one is undefined
	dim3 threads(512, 1, 1);
	dim3 grid1(((CSize + 255) / 256), 1, 1);
	dim3 threads1(blockSize, 1, 1);
	dim3 grid2(1, 1, 1);
	dim3 threads2(32, 1, 1);

//===========================================MGPU===============================================
	int GPU_N;
	checkCudaErrors(hipGetDeviceCount(&GPU_N));
	//hipError_t e1, e2, e3, e4, e5, e6, e7;

	GPU_N = 1;
	const int numGPUs = 1;
	const int NperGPU = CSize / numGPUs;
	const int NEdgeperGPU = (NperGPU * 3) / numGPUs;
	const int NFaceperGPU = (NperGPU * 3) / numGPUs;
	int tempEdgeNum = 0;
	const int N = NperGPU * numGPUs;
	uint nextpower = 1;

//	plan<ASF_vertex> plan[numGPUs];
//	for (int k = 0; k < numGPUs; k++)
//		createPlan(plan[k], NperGPU, NEdgeperGPU, N * sample_seeds, k);
// --- "Breadth-first" approach - async
//	ASF_vertex *inputMatrices;
//	(hipHostMalloc(&inputMatrices, N * sample_seeds * sizeof(ASF_vertex)));
//	fEdge *inputEdges;
//	(hipHostMalloc(&inputEdges, NEdgeperGPU * sample_seeds * sizeof(fEdge)));
//	uint32_t *inputFe_Edge;
//	(hipHostMalloc(&inputFe_Edge,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
//	uint32_t *inputFr_Edge;
//	(hipHostMalloc(&inputFr_Edge,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
//
//	uint32_t *inputFe_Face;
//	(hipHostMalloc(&inputFe_Face,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));
//	uint32_t *inputFr_Face;
//	(hipHostMalloc(&inputFr_Face,
//			NEdgeperGPU * sample_seeds * sizeof(uint32_t)));

//uint32_t* d_Fi;
//	uint32_t* d_Fr;

	//ASF_vertex* h_Out = new ASF_vertex[N * sample_seeds];

	(hipMemcpyAsync(d_currA, _a, currSize * sizeof(ASF_vertex),
			hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_fc, fc, curFaceSize * sizeof(fFace),
			hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_vx, m_x1, CSize * sizeof(float), hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_vy, m_y1, CSize * sizeof(float), hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_vz, m_z1, CSize * sizeof(float), hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_eg, eg, curEdgeSize * sizeof(fEdge),
			hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_step, step, sizeof(Point), hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_b, _b, sizeof(Boundary), hipMemcpyHostToDevice));
	(hipMemcpyAsync(d_d, _d, sizeof(Dimension), hipMemcpyHostToDevice));

	hipDeviceSynchronize();

	for (int ii = 0; ii < icycle; ii++) {

		Tracing<< <iDivUp1(currSize, BLOCKSIZE), BLOCKSIZE >> >(d_currA, d_vx, d_vy, d_vz, d_d, d_b, d_step, bForward, whichData,
				curXDimension, 0, currSize, level, ii);
		/*CheckNeighborhood_c3 << <iDivUp1(NEdgeperGPU, BLOCKSIZE), BLOCKSIZE >> >(plan[k].d_data, plan[k].eg, plan[k].Fe_Edge, plan[k].d, plan[k].b, plan[k].s, bForward, currSize);
		 if (k < numGPUs)
		 runTest(plan[k].Fe_Edge, plan[k].Fr_Edge, inputFr_Edge, NEdgeperGPU);*/

//	hipDeviceSynchronize();
		CheckNeighborhood<< <iDivUp1(curEdgeSize, BLOCKSIZE), BLOCKSIZE >> >(d_currA, d_eg, d_Fe_Edge, d_d, d_b, d_step, bForward, currSize,
				curEdgeSize, level, ii);

		runTest(d_Fe_Edge, d_Fr_Edge, Fr_Edge,
				curEdgeSize);

		memset(Fe_Face, 0, curFaceSize * sizeof(uint32_t));
		memset(Fr_Face, 0, curFaceSize * sizeof(uint32_t));
		CheckFace<< <iDivUp1(curFaceSize, BLOCKSIZE), BLOCKSIZE >> >(currA, eg, fc, Fe_Edge, Fe_Face, _b, _d, step,
				bForward, curFaceSize, ii);

		runTest(d_Fe_Face, d_Fr_Face, Fr_Face,
				curFaceSize);
		printf(" Faces = %d \n", Fr_Face[curFaceSize]);

		//======================================================================

		//while (Fr_Face[curFaceSize] > 0) {
		while (Fr_Edge[curEdgeSize] > 0) {
			//ct++;
			//printf("Edges = %d \n", Fr_Edge[curEdgeSize]);
			printf("Edges = %d \n", Fr_Face[curFaceSize]);
			int oldEdgeSize = curEdgeSize;

			SplitEdge << <iDivUp1(curEdgeSize, BLOCKSIZE), BLOCKSIZE >> >(d_currA, d_fc, d_eg, d_Fe_Edge, d_Fr_Edge, d_d, d_b, d_step, d_vx,
					d_vy, d_vz, curXDimension, bForward, currSize, curEdgeSize,
					level, ii);
			hipDeviceSynchronize();

			currSize = currSize + Fr_Edge[curEdgeSize];
			curEdgeSize = curEdgeSize + Fr_Edge[curEdgeSize] * 2;
//
//			memset(Fr_Edge2, 0, (oldEdgeSize) * sizeof(uint32_t));
//			memset(Fe_Edge2, 0, (oldEdgeSize) * sizeof(uint32_t));
//
//			CheckRemainingEdge(currA, fc, eg, Fe_Edge, Fe_Face, Fe_Edge2, _d,
//					_b, step, bForward, curFaceSize);
//
//			Fr_Edge2[0] = 0;
//			for (int i = 0; i <= oldEdgeSize; i++) {
//				Fr_Edge2[i + 1] = Fr_Edge2[i] + Fe_Edge2[i];
//			}
//
//			printf(" Edges = %d \n", Fr_Edge2[oldEdgeSize]);
//
//			EdgeReduction << <iDivUp1(curEdgeSize, BLOCKSIZE), BLOCKSIZE >> >(plan[k].eg, plan[k].d_data, plan[k].fc, plan[k].Fe_Face, plan[k].Fe_Edge, curEdgeSize);
//			hipDeviceSynchronize();

			memset(Fr_Edge, 0, curEdgeSize*sizeof(uint32_t));
			hipMemset(d_Fr_Edge, 0, curEdgeSize*sizeof(uint32_t));

			CheckNeighborhood<< <iDivUp1(curEdgeSize, BLOCKSIZE), BLOCKSIZE >> >(d_currA, d_eg, d_Fe_Edge, d_d, d_b, d_step, bForward, currSize,
					curEdgeSize, level, ii);

			runTest(d_Fe_Edge, d_Fr_Edge, Fr_Edge,
					curEdgeSize);

			//printf("%d \n", inputFr_Edge[curEdgeSize - 1]);

//			SplitEdge << <iDivUp1(curEdgeSize, BLOCKSIZE), BLOCKSIZE >> >(plan[k].eg, plan[k].d_data, plan[k].fc, plan[k].Fe_Edge, plan[k].Fr_Edge, plan[k].d, plan[k].b, plan[k].s, bForward, currSize, curEdgeSize);
//			hipDeviceSynchronize();
//
//			currSize = currSize + inputFr_Edge[curEdgeSize - 1] + 1;
//
//			curEdgeSize = curEdgeSize + inputFr_Edge[curEdgeSize - 1] + 1;
//
//			CheckNeighborhood << <iDivUp1(curEdgeSize, BLOCKSIZE), BLOCKSIZE >> >(plan[k].eg, plan[k].d_data, plan[k].Fe_Edge, plan[k].d, plan[k].b, plan[k].s, i, bForward, curEdgeSize);
//			hipMemset(plan[k].Fr_Edge, 0, curEdgeSize*sizeof(uint32_t));
//			//hipMemset(inputFr_Edge, 0, curEdgeSize*sizeof(uint32_t));
//			memset(inputFr_Edge, 0, curEdgeSize*sizeof(uint32_t));
//
//			runTest(plan[k].Fe_Edge, plan[k].Fr_Edge, inputFr_Edge, curEdgeSize);

		}

		hipDeviceSynchronize();

		(hipMemcpyAsync(currA, d_currA, currSize * sizeof(ASF_vertex),
						hipMemcpyDeviceToHost));
	}

	printf("current size = %d \n", currSize);

	printf("current size = %d \n", currSize);

	Edge* Fc = new Edge[curEdgeSize];

	memset(Fe_Edge, 0, currSize * sizeof(uint32_t));

	for (int i = 0; i < currSize; i++) {
		ASF_vertex vertex = currA[i];
		if (vertex.checkInBoundary(_b)) {

			Fe_Edge[vertex.getOldRange()]++;

		}
	}

//CheckRangeSetKernel << <grid_2, threads >> >(d_m, d_To, currSize - 1);
	To = new uint32_t[CSize];
	memset(Fr_Edge, 0, CSize * sizeof(uint32_t));
	memset(To, 0, CSize * sizeof(uint32_t));

	uint32_t * Fi = new uint32_t[currSize];
	Fr_Edge[0] = 0;
	for (int i = 1; i <= CSize; i++)
		Fr_Edge[i] = Fr_Edge[i - 1] + Fe_Edge[i];

	memset(Fi, 0, currSize * sizeof(uint32_t));
//checkCudaErrors(hipMemcpy(d_Fr, Fr, currSize * sizeof(uint32_t), hipMemcpyHostToDevice));
	for (int row = 0; row < currSize; row++) {
		ASF_vertex vertex = currA[row];
		if (vertex.checkInBoundary(_b)
				&& vertex.getOldRange() > 0/*&& vertex.getOldRange() == row*/) {
			uint32_t i = Fr_Edge[vertex.getOldRange() - 1]
					+ Fi[vertex.getOldRange()];
			//	if (bForward)
			Fc[i].setValue(vertex.getRange());
			/*else
			 Fc[i].setValue(vertex.getRangeBackward());
			 */
			//Fc[i].setValidBit();
			Fi[vertex.getOldRange()]++;
		}

	}

	float rgb[3];
	rgb[0] = 0.8;
	rgb[1] = 0.0;
	rgb[2] = 0.0;

//display_voxel(1551, rgb, _d->x, _d->y, _d->z);

//////////////////////////////////////////////////////

	rgb[0] = 0.0;
	rgb[1] = 0.0;
	rgb[2] = 0.8;

	float p[3], ep[3];
//glNewList(index_StreamLine_Lorenz, GL_COMPILE);
	{
		for (int row = 0; row < currSize; row++) {
			uint32_t _index = 0;
			if (currA[row].getOldRange() == 1551) {

				p[0] = currA[row].p.x;
				p[1] = currA[row].p.y;
				p[2] = currA[row].p.z;
				_index = currA[row].getRange();

				//	display_voxel(_index,rgb , _d->x, _d->y, _d->z);
				//	drawPoint(p, rgb);
				//	generalstreamlineTracing_single(p, bForward, ep, false);

			}
		}
	}
//	glEndList();
	uint32_t r = currA[14731].getRange();

	/*if (!bForward)
	 r = currA[14731].getRangeBackward();*/

//currA = 0;
//delete[] _aa;
	*oVertex = currA;
//	*oFace = fc;
//	*oEdge = eg;
	*oFc = Fc;
	*oFr = Fr_Edge;
	*oFaceSize = curFaceSize;
	*oEdgeSize = curEdgeSize;
	*oRSize = currSize;			// Fr_Edge[CSize - 1];

}

