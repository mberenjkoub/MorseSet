#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifdef _WIN32
#  define NOMINMAX 
#endif

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <helper_timer.h>
#include "stdint.h"
// includes, kernels
//#include <scan.cu>  // defines prescanArray()

////////////////////////////////////////////////////////////////////////////////


// 16 banks on G80
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// http://www.cs.unc.edu/~prins/Classes/203/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// excellent paper "Prefix sums and their applications".
// http://www-2.cs.cmu.edu/afs/cs.cmu.edu/project/scandal/public/papers/CMU-CS-90-190.html
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//

template <bool isNP2>
__device__ void loadSharedChunkFromMem(uint32_t *s_data,
	const uint32_t *g_idata,
	int n, int baseIndex,
	int& ai, int& bi,
	int& mem_ai, int& mem_bi,
	int& bankOffsetA, int& bankOffsetB)
{
	int thid = threadIdx.x;
	mem_ai = baseIndex + threadIdx.x;
	mem_bi = mem_ai + blockDim.x;

	ai = thid;
	bi = thid + blockDim.x;

	// compute spacing to avoid bank conflicts
	bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	// Cache the computational window in shared memory
	// pad values beyond n with zeros
	s_data[ai + bankOffsetA] = g_idata[mem_ai];

	if (isNP2) // compile-time decision
	{
		s_data[bi + bankOffsetB] = (bi < n) ? g_idata[mem_bi] : 0;
	}
	else
	{
		s_data[bi + bankOffsetB] = g_idata[mem_bi];
	}
}

template <bool isNP2>
__device__ void storeSharedChunkToMem(uint32_t* g_odata,
	const uint32_t* s_data,
	int n,
	int ai, int bi,
	int mem_ai, int mem_bi,
	int bankOffsetA, int bankOffsetB)
{
	__syncthreads();

	// write results to global memory
	g_odata[mem_ai] = s_data[ai + bankOffsetA];
	if (isNP2) // compile-time decision
	{
		if (bi < n)
			g_odata[mem_bi] = s_data[bi + bankOffsetB];
	}
	else
	{
		g_odata[mem_bi] = s_data[bi + bankOffsetB];
	}
}

template <bool storeSum>
__device__ void clearLastElement(uint32_t* s_data,
	uint32_t *g_blockSums,
	int blockIndex)
{
	if (threadIdx.x == 0)
	{
		int index = (blockDim.x << 1) - 1;
		index += CONFLICT_FREE_OFFSET(index);

		if (storeSum) // compile-time decision
		{
			// write this block's total sum to the corresponding index in the blockSums array
			g_blockSums[blockIndex] = s_data[index];
		}

		// zero the last element in the scan so it will propagate back to the front
		s_data[index] = 0;
	}
}



__device__ unsigned int buildSum(uint32_t *s_data)
{
	unsigned int thid = threadIdx.x;
	unsigned int stride = 1;

	// build the sum in place up the tree
	for (int d = blockDim.x; d > 0; d >>= 1)
	{
		__syncthreads();

		if (thid < d)
		{
			int i = __mul24(__mul24(2, stride), thid);
			int ai = i + stride - 1;
			int bi = ai + stride;

			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			s_data[bi] += s_data[ai];
		}

		stride *= 2;
	}

	return stride;
}

__device__ void scanRootToLeaves(uint32_t *s_data, unsigned int stride)
{
	unsigned int thid = threadIdx.x;

	// traverse down the tree building the scan in place
	for (int d = 1; d <= blockDim.x; d *= 2)
	{
		stride >>= 1;

		__syncthreads();

		if (thid < d)
		{
			int i = __mul24(__mul24(2, stride), thid);
			int ai = i + stride - 1;
			int bi = ai + stride;

			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);

			uint32_t t = s_data[ai];
			s_data[ai] = s_data[bi];
			s_data[bi] += t;
		}
	}
}

template <bool storeSum>
__device__ void prescanBlock(uint32_t *data, int blockIndex, uint32_t *blockSums)
{
	int stride = buildSum(data);               // build the sum in place up the tree
	clearLastElement<storeSum>(data, blockSums,
		(blockIndex == 0) ? blockIdx.x : blockIndex);
	scanRootToLeaves(data, stride);            // traverse down tree to build the scan 
}

template <bool storeSum, bool isNP2>
__global__ void prescan(uint32_t *g_odata,
	const uint32_t *g_idata,
	uint32_t *g_blockSums,
	int n,
	int blockIndex,
	int baseIndex)
{
	int ai, bi, mem_ai, mem_bi, bankOffsetA, bankOffsetB;
	extern __shared__ uint32_t s_data[];

	// load data into shared memory
	loadSharedChunkFromMem<isNP2>(s_data, g_idata, n,
		(baseIndex == 0) ?
		__mul24(blockIdx.x, (blockDim.x << 1)) : baseIndex,
		ai, bi, mem_ai, mem_bi,
		bankOffsetA, bankOffsetB);
	// scan the data in each block
	prescanBlock<storeSum>(s_data, blockIndex, g_blockSums);
	// write results to device memory
	storeSharedChunkToMem<isNP2>(g_odata, s_data, n,
		ai, bi, mem_ai, mem_bi,
		bankOffsetA, bankOffsetB);
}


__global__ void uniformAdd(uint32_t *g_data,
	uint32_t *uniforms,
	int n,
	int blockOffset,
	int baseIndex)
{
	__shared__ uint32_t uni;
	if (threadIdx.x == 0)
		uni = uniforms[blockIdx.x + blockOffset];

	unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

	__syncthreads();

	// note two adds per thread
	g_data[address] += uni;
	g_data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}



//======================================================================================
// includes, kernels



inline bool
isPowerOfTwo(int n)
{
	return ((n&(n - 1)) == 0);
}

inline int
floorPow2(int n)
{
#ifdef WIN32
	// method 2
	return 1 << (int)logb((float)n);
#else
	// method 1
	// float nf = (float)n;
	// return 1 << (((*(int*)&nf) >> 23) - 127); 
	int exp;
	frexp((float)n, &exp);
	return 1 << (exp - 1);
#endif
}

#define BLOCK_SIZE 256

uint32_t** g_scanBlockSums;
unsigned int g_numEltsAllocated = 0;
unsigned int g_numLevelsAllocated = 0;

void preallocBlockSums(unsigned int maxNumElements)
{
	assert(g_numEltsAllocated == 0); // shouldn't be called 

	g_numEltsAllocated = maxNumElements;

	unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
	unsigned int numElts = maxNumElements;

	int level = 0;

	do
	{
		unsigned int numBlocks =
			max(1, (int)ceil((float)numElts / (2.f * blockSize)));
		if (numBlocks > 1)
		{
			level++;
		}
		numElts = numBlocks;
	} while (numElts > 1);

	g_scanBlockSums = (uint32_t**)malloc(level * sizeof(uint32_t*));
	g_numLevelsAllocated = level;

	numElts = maxNumElements;
	level = 0;

	do
	{
		unsigned int numBlocks =
			max(1, (int)ceil((float)numElts / (2.f * blockSize)));
		if (numBlocks > 1)
		{
			(hipMalloc((void**)&g_scanBlockSums[level++],
				numBlocks * sizeof(uint32_t)));
		}
		numElts = numBlocks;
	} while (numElts > 1);

	("preallocBlockSums");
}

void deallocBlockSums()
{
	for (int i = 0; i < g_numLevelsAllocated; i++)
	{
		hipFree(g_scanBlockSums[i]);
	}

	("deallocBlockSums");

	free((void**)g_scanBlockSums);

	g_scanBlockSums = 0;
	g_numEltsAllocated = 0;
	g_numLevelsAllocated = 0;
}


void prescanArrayRecursive(uint32_t *outArray,
	const uint32_t *inArray,
	int numElements,
	int level)
{
	unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
	unsigned int numBlocks =
		max(1, (int)ceil((float)numElements / (2.f * blockSize)));
	unsigned int numThreads;

	if (numBlocks > 1)
		numThreads = blockSize;
	else if (isPowerOfTwo(numElements))
		numThreads = numElements / 2;
	else
		numThreads = floorPow2(numElements);

	unsigned int numEltsPerBlock = numThreads * 2;

	// if this is a non-power-of-2 array, the last block will be non-full
	// compute the smallest power of 2 able to compute its scan.
	unsigned int numEltsLastBlock =
		numElements - (numBlocks - 1) * numEltsPerBlock;
	unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
	unsigned int np2LastBlock = 0;
	unsigned int sharedMemLastBlock = 0;

	if (numEltsLastBlock != numEltsPerBlock)
	{
		np2LastBlock = 1;

		if (!isPowerOfTwo(numEltsLastBlock))
			numThreadsLastBlock = floorPow2(numEltsLastBlock);

		unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
		sharedMemLastBlock =
			sizeof(uint32_t)* (2 * numThreadsLastBlock + extraSpace);
	}

	// padding space is used to avoid shared memory bank conflicts
	unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
	unsigned int sharedMemSize =
		sizeof(uint32_t)* (numEltsPerBlock + extraSpace);

#ifdef DEBUG
	if (numBlocks > 1)
	{
		assert(g_numEltsAllocated >= numElements);
	}
#endif

	// setup execution parameters
	// if NP2, we process the last block separately
	dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1);
	dim3  threads(numThreads, 1, 1);

	// make sure there are no CUDA errors before we start
	("prescanArrayRecursive before kernels");

	// execute the scan
	if (numBlocks > 1)
	{
		prescan<true, false> << < grid, threads, sharedMemSize >> >(outArray,
			inArray,
			g_scanBlockSums[level],
			numThreads * 2, 0, 0);
		getLastCudaError("prescanWithBlockSums");
		if (np2LastBlock)
		{
			prescan<true, true> << < 1, numThreadsLastBlock, sharedMemLastBlock >> >
				(outArray, inArray, g_scanBlockSums[level], numEltsLastBlock,
				numBlocks - 1, numElements - numEltsLastBlock);
			getLastCudaError("prescanNP2WithBlockSums");
		}

		// After scanning all the sub-blocks, we are mostly done.  But now we 
		// need to take all of the last values of the sub-blocks and scan those.  
		// This will give us a new value that must be sdded to each block to 
		// get the final results.
		// recursive (CPU) call
		prescanArrayRecursive(g_scanBlockSums[level],
			g_scanBlockSums[level],
			numBlocks,
			level + 1);

		uniformAdd << < grid, threads >> >(outArray,
			g_scanBlockSums[level],
			numElements - numEltsLastBlock,
			0, 0);
		getLastCudaError("uniformAdd");
		if (np2LastBlock)
		{
			uniformAdd << < 1, numThreadsLastBlock >> >(outArray,
				g_scanBlockSums[level],
				numEltsLastBlock,
				numBlocks - 1,
				numElements - numEltsLastBlock);
			getLastCudaError("uniformAdd");
		}
	}
	else if (isPowerOfTwo(numElements))
	{
		prescan<false, false> << < grid, threads, sharedMemSize >> >(outArray, inArray,
			0, numThreads * 2, 0, 0);
		getLastCudaError("prescan");
	}
	else
	{
		prescan<false, true> << < grid, threads, sharedMemSize >> >(outArray, inArray,
			0, numElements, 0, 0);
		getLastCudaError("prescanNP2");
	}
}

void prescanArray(uint32_t *outArray, uint32_t *inArray, int numElements)
{
	prescanArrayRecursive(outArray, inArray, numElements, 0);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
//int 
//main( int argc, char** argv) 
//{
//    runTest( argc, argv);
//    CUT_EXIT(argc, argv);
//}

////////////////////////////////////////////////////////////////////////////////
//! Run a scan test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(uint32_t* d_In, uint32_t* d_Out,uint32_t* h_Out, int num_elements)
{
   


    int num_test_iterations = 100;
  //  int num_elements = 1000000; // can support large, non-power-of-2 arrays!


	unsigned int mem_size = sizeof(uint32_t)* num_elements;
    
    unsigned int timerGPU, timerCPU;
  /*  CUT_SAFE_CALL(cutCreateTimer(&timerCPU));
    CUT_SAFE_CALL(cutCreateTimer(&timerGPU));*/

    // allocate host memory to store the input data
    //uint32_t* h_data = (uint32_t*) malloc( mem_size);
    //  
    //// initialize the input data on the host
    //for( unsigned int i = 0; i < num_elements; ++i) 
    //{
    //    h_data[i] = 1.0f;//(int)(10 * rand()/32768.f);
    //}

    // compute reference solution
	uint32_t* reference = (uint32_t*)malloc(mem_size);
  //  cutStartTimer(timerCPU);
  //  for (int i = 0; i < num_test_iterations; i++)
  //  {
  ////      computeGold( reference, h_data, num_elements);
  //  }
  //  cutStopTimer(timerCPU);

    // allocate device memory input and output arrays
    uint32_t* d_idata = NULL;
	uint32_t* d_odata = NULL;

   /* ( hipMalloc( (void**) &d_idata, mem_size));
    ( hipMalloc( (void**) &d_odata, mem_size));*/
    
    // copy host memory to device input array
  //  ( hipMemcpy( d_idata, h_data, mem_size, hipMemcpyHostToDevice) );
    // initialize all the other device arrays to be safe
  //  ( hipMemcpy( d_odata, h_data, mem_size, hipMemcpyHostToDevice) );

    //printf("Running parallel prefix sum (prescan) of %d elements\n", num_elements);
    //printf("This version is work efficient (O(n) adds)\n");
    //printf("and has very few shared memory bank conflicts\n\n");

    preallocBlockSums(num_elements);

    // run once to remove startup overhead
    prescanArray(d_Out, d_In, num_elements);


    // Run the prescan
//    cutStartTimer(timerGPU);
    for (int i = 0; i < num_test_iterations; i++)
    {
        //printf("prescanArray\n");
        prescanArray(d_Out, d_In, num_elements);
    }
   // cutStopTimer(timerGPU);

    deallocBlockSums();    

    // copy result from device to host
	(hipMemcpy(h_Out, d_Out, sizeof(uint32_t)* num_elements,
                               hipMemcpyDeviceToHost));

    //// If this is a regression test write the results to a file
    //if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    //{
    //    // write file for regression test 
    //    cutWriteFilef( "./data/result.dat", h_data, num_elements, 0.0);
    //}
    //else 
    //{
    //    // custom output handling when no regression test running
    //    // in this case check if the result is equivalent to the expected soluion
    //    unsigned int result_regtest = cutComparef( reference, h_data, num_elements);
    //    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    //    printf( "Average GPU execution time: %f ms\n", cutGetTimerValue(timerGPU) / num_test_iterations);
    //    printf( "CPU execution time:         %f ms\n", cutGetTimerValue(timerCPU) / num_test_iterations);
    //}

    // cleanup memory
 //   cutDeleteTimer(timerCPU);
 //   cutDeleteTimer(timerGPU);
    //free( h_data);
    //free( reference);
    //hipFree( d_odata);
    //hipFree( d_idata);
}
